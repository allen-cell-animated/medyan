#include "hip/hip_runtime.h"

//------------------------------------------------------------------
//  **MEDYAN** - Simulation Package for the Mechanochemical
//               Dynamics of Active Networks, v3.1
//
//  Copyright (2015-2016)  Papoian Lab, University of Maryland
//
//                 ALL RIGHTS RESERVED
//
//  See the MEDYAN web page for more information:
//  http://www.medyan.org
//------------------------------------------------------------------

#include "CGMethod.h"

#include "ForceFieldManager.h"

#include "CGMethodCUDA.h"
#include "Bead.h"
#ifdef CUDAACCL
#ifdef __HIPCC__
#define CUDA_HOSTDEV __host__ __device__
#else
#define CUDA_HOSTDEV
#include "nvToolsExt.h"
#endif
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "CUDAcommon.h"
#endif
#define ARRAY_SIZE 128
//
#include <vector>
#include <cmath>
#include <ctime>
#include "Bead.h"
#include <ctime>
#include <cstdlib>
#include "cross_check.h"
//
long CGMethod::N = 0;
long CGMethod::Ncyl = 0;
#ifdef CUDAACCL
void CGMethod::CUDAresetlambda(hipStream_t stream) {
    resetlambdaCUDA<<<1,1,0, stream>>>(CUDAcommon::getCUDAvars().gpu_lambda);
            CUDAcommon::handleerror(hipGetLastError(), "resetlambdaCUDA", "CGMethod.cu");
}
void CGMethod::CUDAinitializeLambda(hipStream_t stream, bool *check_in, bool *check_out, bool *Polaksafestate, int
                                    *gpu_state){

//    hipStream_t  s;
//    CUDAcommon::handleerror(hipStreamCreate(&s));
//    hipEvent_t  e;
//    CUDAcommon::handleerror(hipEventCreate(&e));


//    maxFCUDA<<<1,1, 0, s>>>(CUDAcommon::getCUDAvars().gpu_forceAux, gpu_nint, gpu_fmax);
//    hipStreamSynchronize(s);

//    maxFCUDAred<<<1,3, 3*sizeof(floatingpoint), s>>>(CUDAcommon::getCUDAvars().gpu_forceAux, gpu_nint, gpu_fmax);
//    hipStreamSynchronize(s);


//    CUDAcommon::handleerror(hipDeviceSynchronize());
//    std::cout<<"======"<<endl;
//    CUDAcommon::handleerror(hipEventRecord(e,s));
//    CUDAcommon::handleerror(hipGetLastError(), "maxFCUDA", "CGMethod.cu");

////    CUDAcommon::handleerror(hipStreamWaitEvent(stream,e,0));

////    CUDAcommon::handleerror(hipEventDestroy(e));

//    CUDAcommon::handleerror(hipStreamDestroy(s));

//    auto gpu_lambda = CUDAcommon::getCUDAvars().gpu_lambda;
    auto gpu_energy = CUDAcommon::getCUDAvars().gpu_energy;
    initializeLambdaCUDA<<<1,1,0, stream>>>(check_in, check_out, g_currentenergy, gpu_energy, gpu_initlambdalocal, gpu_fmax,
            gpu_params, Polaksafestate, gpu_state);

//    CUDAcommon::handleerror(hipStreamSynchronize (stream));

    CUDAcommon::handleerror(hipGetLastError(), "initializeLambdaCUDA", "CGMethod.cu");
}

//void CGMethod::getmaxFCUDA(floatingpoint *gpu_forceAux, int *gpu_nint, floatingpoint *gpu_fmax) {
//    maxFCUDA<<<1,1>>>(CUDAcommon::getCUDAvars().gpu_forceAux, gpu_nint, gpu_fmax);
//    CUDAcommon::handleerror(hipGetLastError(), "getmaxFCUDA", "CGMethod.cu");
//}
void CGMethod::CUDAfindLambda(hipStream_t  stream1, hipStream_t stream2, hipEvent_t  event, bool *checkin, bool
        *checkout, bool *gpu_safestate, int *gpu_state) {
//ToDo remove stream2 from the list of args.
    auto gpu_energy = CUDAcommon::getCUDAvars().gpu_energy;
    auto gpu_lambda = CUDAcommon::getCUDAvars().gpu_lambda;
    findLambdaCUDA << < 1, 1, 0, stream1 >> > (gpu_energy, g_currentenergy, gpu_FDotFA, gpu_fmax, gpu_lambda,
            gpu_params, checkin, checkout, gpu_safestate, gpu_state);
    CUDAcommon::handleerror(hipEventRecord(event, stream1));
#ifdef ALLSYNC
    hipDeviceSynchronize();
#endif
/*    findLambdaCUDA2 << < 1, 1, 0, stream1 >> > (gpu_fmax, gpu_lambda, gpu_params,
            checkin, checkout, gpu_safestate,
            gpu_state);
    CUDAcommon::handleerror(hipGetLastError(), "findLambdaCUDA", "CGMethod.cu")*/;
}
//void CGMethod::CUDAprepforbacktracking(hipStream_t stream, bool *check_in, bool *check_out){

//    hipStream_t  s;
//    CUDAcommon::handleerror(hipStreamCreate(&s));
//    hipEvent_t  e;
//    CUDAcommon::handleerror(hipEventCreate(&e));

//    maxFCUDA<<<1,1, 0, s>>>(CUDAcommon::getCUDAvars().gpu_forceAux, gpu_nint, gpu_fmax);
//    CUDAcommon::handleerror(hipEventRecord(e,s));
//    CUDAcommon::handleerror(hipGetLastError());


//    CUDAcommon::handleerror(hipStreamWaitEvent(stream,e,0));


//    CUDAcommon::handleerror(hipEventDestroy(e));
//    CUDAcommon::handleerror(hipStreamDestroy(s));

////    CUDAcommon::handleerror(hipStreamSynchronize (stream));
//    auto gpu_lambda = CUDAcommon::getCUDAvars().gpu_lambda;
//    auto gpu_energy = CUDAcommon::getCUDAvars().gpu_energy;
//    prepbacktracking<<<1,1,0, stream>>>(check_in, check_out, g_currentenergy, gpu_energy, gpu_lambda, gpu_fmax,
//            gpu_params);
//    CUDAcommon::handleerror(hipStreamSynchronize (stream));
//    CUDAcommon::handleerror(hipGetLastError());
//}
//void CGMethod::CUDAprepforsafebacktracking(hipStream_t stream, bool *check_in, bool *check_out){
//    auto gpu_lambda = CUDAcommon::getCUDAvars().gpu_lambda;
//    auto gpu_energy = CUDAcommon::getCUDAvars().gpu_energy;
//    prepsafebacktracking<<<1,1,0,stream>>>(check_in, check_out, g_currentenergy, gpu_energy, gpu_lambda, gpu_params);
//    CUDAcommon::handleerror(hipStreamSynchronize (stream));
//    CUDAcommon::handleerror(hipGetLastError());
//}

void CGMethod::CUDAallFDotF(hipStream_t stream){

    allFADotFCUDA<<<blocksnthreads[0], blocksnthreads[1],0,stream>>>(CUDAcommon::getCUDAvars().gpu_force,
            CUDAcommon::getCUDAvars().gpu_force ,gpu_g, gpu_nint);
    CUDAcommon::handleerror(hipGetLastError(), "allFADotFCUDA", "CGMethod.cu");
//    addvector<<<1,1,0,stream>>>(gpu_g, gpu_nint, gpu_FDotF);
//    hipStreamSynchronize(stream);
//    addvectorred<<<1,200,200 * sizeof(floatingpoint),stream>>>(gpu_g, gpu_nint, gpu_FDotF);
//    floatingpoint Sum[1];
//        CUDAcommon::handleerror(hipMemcpy(Sum, gpu_FDotF, sizeof(floatingpoint), hipMemcpyDeviceToHost));
    resetfloatingpointvariableCUDA<<<1,1,0,stream>>>(gpu_FDotF);
    addvectorredcgm<<<bntaddvector.at(2),bntaddvector.at(3), bntaddvector.at(3) * sizeof(floatingpoint),stream>>>(gpu_g,
            gpu_nint, gpu_FDotF);
//    floatingpoint Sum2[1];
//    CUDAcommon::handleerror(hipMemcpy(Sum2, gpu_FDotF, sizeof(floatingpoint), hipMemcpyDeviceToHost));
//    std::cout<<Sum[0]<<" "<<Sum2[0]<<endl;
//    hipStreamSynchronize(stream);
    CUDAcommon::handleerror(hipGetLastError(), "allFADotFCUDA", "CGMethod.cu");

}
void CGMethod::CUDAallFADotFA(hipStream_t stream){

    allFADotFCUDA<<<blocksnthreads[0], blocksnthreads[1],0,stream>>>(CUDAcommon::getCUDAvars().gpu_forceAux,
            CUDAcommon::getCUDAvars().gpu_forceAux ,gpu_g, gpu_nint);
    CUDAcommon::handleerror(hipGetLastError(), "allFADotFCUDA", "CGMethod.cu");
//    addvector<<<1,1,0,stream>>>(gpu_g, gpu_nint, gpu_FADotFA);
//    hipStreamSynchronize(stream);
//    addvectorred<<<1,200,200* sizeof(floatingpoint),stream>>>(gpu_g, gpu_nint, gpu_FADotFA);
//    hipStreamSynchronize(stream);
    resetfloatingpointvariableCUDA<<<1,1,0,stream>>>(gpu_FADotFA);
    addvectorredcgm<<<bntaddvector.at(2),bntaddvector.at(3), bntaddvector.at(3) * sizeof(floatingpoint),stream>>>(gpu_g,
            gpu_nint, gpu_FADotFA);
    CUDAcommon::handleerror(hipGetLastError(), "allFADotFCUDA", "CGMethod.cu");

}
void CGMethod::CUDAallFADotFAP(hipStream_t stream){

    allFADotFCUDA<<<blocksnthreads[0], blocksnthreads[1],0,stream>>>(CUDAcommon::getCUDAvars().gpu_forceAux,
            CUDAcommon::getCUDAvars().gpu_forceAuxP ,gpu_g, gpu_nint);
    CUDAcommon::handleerror(hipGetLastError(), "allFADotFCUDA", "CGMethod.cu");
//    addvector<<<1,1,0,stream>>>(gpu_g, gpu_nint, gpu_FADotFAP);
//    hipStreamSynchronize(stream);
//    addvectorred<<<1,200,200 * sizeof(floatingpoint),stream>>>(gpu_g, gpu_nint, gpu_FADotFAP);
    resetfloatingpointvariableCUDA<<<1,1,0,stream>>>(gpu_FADotFAP);
    addvectorredcgm<<<bntaddvector.at(2),bntaddvector.at(3), bntaddvector.at(3) * sizeof(floatingpoint),stream>>>(gpu_g,
            gpu_nint, gpu_FADotFAP);
//    hipStreamSynchronize(stream);
    CUDAcommon::handleerror(hipGetLastError(), "allFADotFCUDA", "CGMethod.cu");

}
void CGMethod::CUDAallFDotFA(hipStream_t stream){

    allFADotFCUDA<<<blocksnthreads[0], blocksnthreads[1],0,stream>>>(CUDAcommon::getCUDAvars().gpu_force,
            CUDAcommon::getCUDAvars().gpu_forceAux ,gpu_g, gpu_nint);
    CUDAcommon::handleerror(hipGetLastError(), "allFADotFCUDA", "CGMethod.cu");
//    addvector<<<1,1,0,stream>>>(gpu_g, gpu_nint, gpu_FDotFA);
//    hipStreamSynchronize(stream);
//    addvectorred<<<1,200,200* sizeof(floatingpoint),stream>>>(gpu_g, gpu_nint, gpu_FDotFA);
//    hipStreamSynchronize(stream);
    resetfloatingpointvariableCUDA<<<1,1,0,stream>>>(gpu_FDotFA);
    addvectorredcgm<<<bntaddvector.at(2),bntaddvector.at(3), bntaddvector.at(3) * sizeof(floatingpoint),stream>>>(gpu_g,
            gpu_nint, gpu_FDotFA);
    CUDAcommon::handleerror(hipGetLastError(), "allFADotFCUDA", "CGMethod.cu");

}

void CGMethod::CUDAshiftGradient(hipStream_t stream, bool *Mcheckin) {
    shiftGradientCUDA<<<blocksnthreads[0], blocksnthreads[1],0, stream>>>(CUDAcommon::getCUDAvars().gpu_force,
            CUDAcommon::getCUDAvars().gpu_forceAux, gpu_nint, gpu_FADotFA, gpu_FADotFAP, gpu_FDotF, Mcheckin);
}

void CGMethod::CUDAshiftGradientifSafe(hipStream_t stream, bool *Mcheckin, bool *Scheckin){
    shiftGradientCUDAifsafe<<<blocksnthreads[0], blocksnthreads[1],0, stream>>>(CUDAcommon::getCUDAvars().gpu_force, CUDAcommon::getCUDAvars().gpu_forceAux, gpu_nint,
                            Mcheckin, Scheckin);
    CUDAcommon::handleerror(hipGetLastError(),"CUDAshiftGradientifSafe", "CGMethod.cu");
}

//void CGMethod::CUDAgetPolakvars(bool calc_safestate,hipStream_t streamcalc, floatingpoint* gpu_GRADTOL, bool *gminstatein,
//                                    bool *gminstateout, bool *gsafestateout, volatile bool *cminstate){
////    state[0] = false;
////    state[1] = false;
//    if(cminstate[0] == true) {

////        maxFCUDA << < 1, 1, 0, streamcalc >> > (CUDAcommon::getCUDAvars().gpu_forceAux, gpu_nint, gpu_fmax);
//        maxFCUDAred<<<1,3, 3*sizeof(floatingpoint), streamcalc>>>(CUDAcommon::getCUDAvars().gpu_forceAux, gpu_nint, gpu_fmax);
////        CUDAcommon::handleerror(hipDeviceSynchronize());
////        std::cout<<"======"<<endl;
//        CUDAcommon::handleerror(hipGetLastError(), "maxFCUDA", "CGMethod.cu");

//        getminimizestateCUDA << < 1, 1, 0, streamcalc >> > (gpu_fmax, gpu_GRADTOL, gminstatein, gminstateout);
//        CUDAcommon::handleerror(hipGetLastError(), "getminimizestateCUDA", "CGMethod.cu");
////        CUDAcommon::handleerror(hipStreamSynchronize(streamcalc));
//    }
//    if(calc_safestate){
//        CUDAallFDotFA(streamcalc);
//        getsafestateCUDA<<<1,1,0,streamcalc>>>(gpu_FDotFA, gpu_FDotF, gpu_FADotFA, gsafestateout);
//        CUDAcommon::handleerror(hipGetLastError(), "getsafestateCUDA", "CGMethod.cu");
//    }
//}

void CGMethod::CUDAgetPolakvars(hipStream_t streamcalc, floatingpoint* gpu_GRADTOL, bool *gminstatein,
                                bool *gminstateout, volatile bool *cminstate){
//    state[0] = false;
//    state[1] = false;
    if(cminstate[0] == true) {

//        maxFCUDA << < 1, 1, 0, streamcalc >> > (CUDAcommon::getCUDAvars().gpu_forceAux, gpu_nint, gpu_fmax);
//        maxFCUDAred<<<1,3, 3*sizeof(floatingpoint), streamcalc>>>(CUDAcommon::getCUDAvars().gpu_forceAux, gpu_nint, gpu_fmax);
//        hipStreamSynchronize(streamcalc);

        //@{ V2
//        allFADotFCUDA<<<blocksnthreads[0], blocksnthreads[1],0,streamcalc>>>(CUDAcommon::getCUDAvars().gpu_forceAux,
//                CUDAcommon::getCUDAvars().gpu_forceAux ,gpu_maxF, gpu_nint);
//        CUDAcommon::handleerror(hipGetLastError(), "allFADotFACUDA", "CGMethod.cu");
//        maxFCUDAredv2<<<1,512,512*sizeof(floatingpoint), streamcalc>>>(gpu_maxF, gpu_nint,
//                gpu_fmax);
        //@}
        //Test
//        CUDAcommon::handleerror(hipDeviceSynchronize());
//        floatingpoint maxFv1[1];
//        hipMemcpy(maxFv1, gpu_fmax,  sizeof(floatingpoint), hipMemcpyDeviceToHost);
//        std::cout<<"v1 maxF "<<maxFv1[0]<<endl;
//        floatingpoint *gpu_fmax2;
//        CUDAcommon::handleerror(hipMalloc((void **)&gpu_fmax2, sizeof(floatingpoint)));
/*#ifdef CUDATIMETRACK
        hipStream_t  streamcalc2;
        hipStreamCreate(&streamcalc2);
        streamcalc = streamcalc2;
        chrono::high_resolution_clock::time_point tbegin, tend;
        tbegin = chrono::high_resolution_clock::now();
#endif*/
        //@{ V3
        //TODO combine with FADOTFA calculation by making it write it to gpu_maxF before
        // adding.
        allFADotFCUDA<<<blocksnthreads[0], blocksnthreads[1],0,streamcalc>>>(CUDAcommon::getCUDAvars().gpu_forceAux,
                CUDAcommon::getCUDAvars().gpu_forceAux ,gpu_maxF, gpu_nint);
        resetfloatingpointvariableCUDA<<<1,1,0,streamcalc>>>(gpu_fmax);
        resetintvariableCUDA<<<1,1,0,streamcalc>>>(gpu_mutexlock);
        maxFCUDAredv3<<<bntaddvector.at(2),bntaddvector.at(3), bntaddvector.at(3) *
                sizeof(floatingpoint),streamcalc>>>(gpu_maxF, gpu_nint, gpu_fmax, gpu_mutexlock);
        CUDAcommon::handleerror(hipGetLastError(), "maxFCUDA", "CGMethod.cu");
        getminimizestateCUDA << < 1, 1, 0, streamcalc >> > (gpu_fmax, gpu_GRADTOL, gminstatein, gminstateout);
        CUDAcommon::handleerror(hipGetLastError(), "getminimizestateCUDA", "CGMethod.cu");
        //@}
/*#ifdef CUDATIMETRACK
        hipStreamSynchronize(streamcalc);
        tend = chrono::high_resolution_clock::now();
        chrono::duration<floatingpoint> elapsed_runs1(tend - tbegin);
        std::cout<<"CUDA maxF "<<elapsed_runs1.count()<<endl;
#endif
#ifdef CUDATIMETRACK
        tbegin = chrono::high_resolution_clock::now();
#endif
        maxF();
        auto x = maxF()>maxF();
#ifdef CUDATIMETRACK
        hipStreamSynchronize(streamcalc);
        tend = chrono::high_resolution_clock::now();
        chrono::duration<floatingpoint> elapsed_runs2(tend - tbegin);
        std::cout<<"SERL maxF "<<elapsed_runs2.count()<<endl;
#endif*/


//        CUDAcommon::handleerror(hipDeviceSynchronize());
//        hipMemcpy(maxFv1, gpu_fmax2,  sizeof(floatingpoint), hipMemcpyDeviceToHost);
//        std::cout<<"v2 maxF "<<maxFv1[0]<<endl;
//        hipFree(gpu_fmax2);
        //Test ends

//                cout<<"MaxF algorithm is not accurate. Redo algorithm. Exiting"<<endl;
//                exit(EXIT_FAILURE);

//        hipStreamSynchronize(streamcalc);
//        CUDAcommon::handleerror(hipDeviceSynchronize());
//        std::cout<<"======"<<endl;
/*        CUDAcommon::handleerror(hipGetLastError(), "maxFCUDA", "CGMethod.cu");
        getminimizestateCUDA << < 1, 1, 0, streamcalc >> > (gpu_fmax, gpu_GRADTOL, gminstatein, gminstateout);
        CUDAcommon::handleerror(hipGetLastError(), "getminimizestateCUDA", "CGMethod.cu");*/
//        CUDAcommon::handleerror(hipStreamSynchronize(streamcalc));
    }
//    if(calc_safestate){
//        CUDAallFDotFA(streamcalc);
//        getsafestateCUDA<<<1,1,0,streamcalc>>>(gpu_FDotFA, gpu_FDotF, gpu_FADotFA, gsafestateout);
//        CUDAcommon::handleerror(hipGetLastError(), "getsafestateCUDA", "CGMethod.cu");
//    }
    CUDAcommon::handleerror(hipGetLastError(),"CUDAgetPolakvars", "CGMethod.cu");
}

void CGMethod::CUDAgetPolakvars2(hipStream_t streamcalc, bool *gsafestateout){
        CUDAallFDotFA(streamcalc);
        getsafestateCUDA<<<1,1,0,streamcalc>>>(gpu_FDotFA, gpu_FDotF, gpu_FADotFA, gsafestateout);
        CUDAcommon::handleerror(hipGetLastError(), "getsafestateCUDA", "CGMethod.cu");
}

void CGMethod::CUDAmoveBeads(hipStream_t stream, bool *gpu_checkin){
    floatingpoint *gpu_lambda = CUDAcommon::getCUDAvars().gpu_lambda;
    floatingpoint *gpu_coord = CUDAcommon::getCUDAvars().gpu_coord;
    floatingpoint *gpu_force = CUDAcommon::getCUDAvars().gpu_force;

    moveBeadsCUDA<<<blocksnthreads[0], blocksnthreads[1],0, stream>>>(gpu_coord, gpu_force, gpu_lambda, gpu_nint,
            gpu_checkin);

    CUDAcommon::handleerror(hipGetLastError(),"moveBeadsCUDA", "CGMethod.cu");
}

void CGMethod::CUDAinitializePolak(hipStream_t stream, bool *minstatein, bool *minstateout, bool *safestatein, bool
        *safestateout){
    CUDAallFDotFA(stream);
    initializePolak<<<1,1,0,stream>>>(minstatein, minstateout, safestatein, safestateout);
    CUDAcommon::handleerror(hipGetLastError(),"CUDAinitializePolak", "CGPolakRibiereMethod.cu");
}

//floatingpoint CGMethod::gpuFDotF(floatingpoint *f1,floatingpoint *f2){
//
//    allFADotFCUDA<<<blocksnthreads[0], blocksnthreads[1]>>>(f1, f2 ,gpu_g, gpu_nint);
//    CUDAcommon::handleerror(hipGetLastError(),"allFADotFCUDA", "CGMethod.cu");
////    addvector<<<1,1>>>(gpu_g, gpu_nint, gSum);
//    addvectorred<<<1,200,200* sizeof(floatingpoint)>>>(gpu_g, gpu_nint, gSum);
//    CUDAcommon::handleerror(hipGetLastError(),"allFADotFCUDA", "CGMethod.cu");
//
////    CUDAcommon::handleerror( hipPeekAtLastError() );
////    CUDAcommon::handleerror(hipDeviceSynchronize());
//
//    floatingpoint g[1];
//    CUDAcommon::handleerror(hipMemcpy(g, gSum, sizeof(floatingpoint),
//                                       hipMemcpyDeviceToHost));
//
//
////    floatingpoint g[N/3];
////    CUDAcommon::handleerror(hipMemcpy(g, gpu_g, N/3 * sizeof(floatingpoint),
////                                       hipMemcpyDeviceToHost));
////    CUDAcommon::handleerror(hipFree(gpu_g));
////    floatingpoint sum=0.0;
////    for(auto i=0;i<N/3;i++)
////        sum+=g[i];
//    return g[0];
//}
#endif
totalforcefloatingpoint CGMethod::allFDotF()
{

	totalforcefloatingpoint g = 0;
    for(int i = 0; i < N; i++)
        g += force[i] * force[i];

    return g;
}

totalforcefloatingpoint CGMethod::allFADotFA()
{

	totalforcefloatingpoint g = 0;
    for(int i = 0; i < N; i++)
        g += forceAux[i] * forceAux[i];
//#ifdef CUDAACCL

//    auto g_cuda = gpuFDotF(CUDAcommon::getCUDAvars().gpu_forceAux,CUDAcommon::getCUDAvars().gpu_forceAux);

//#endif
//    if(g>1000000.0){
//        if(abs(g-g_cuda)/abs(g) > 0.001){
//            std::cout << g << " " << g_cuda << endl;
//            std::cout << "Precison mismatch FADotFA " << abs(g - g_cuda) << endl;
//        }
//
//    }
//    else if(abs(g-g_cuda)>1/100000000.0) {
//        std::cout << g << " " << g_cuda << endl;
//        std::cout << "Precison mismatch FADotFA " << abs(g - g_cuda) << endl;
//    }
    return g;
}

totalforcefloatingpoint CGMethod::allFADotFAP()
{
	totalforcefloatingpoint g = 0;
    for(int i = 0; i < N; i++)
        g += forceAux[i] * forceAuxPrev[i];

    return g;
}

totalforcefloatingpoint CGMethod::allFDotFA()
{
	totalforcefloatingpoint g = 0;
    for(int i = 0; i < N; i++) {
        g += force[i] * forceAux[i];
    }
//#ifdef CUDAACCL
//    auto g_cuda = gpuFDotF(CUDAcommon::getCUDAvars().gpu_force,CUDAcommon::getCUDAvars().gpu_forceAux);
//#endif
//    if(g>1000000.0){
//        if(abs(g-g_cuda)/abs(g) > 0.001){
//            std::cout << g << " " << g_cuda << endl;
//            std::cout << "Precison mismatch FDotFA " << abs(g - g_cuda) << endl;
//        }
//
//    }
//    else if(abs(g-g_cuda)>1/100000000.0) {
//        std::cout << "Precison mismatch FDotFA " << abs(g - g_cuda) << endl;
//        std::cout << g << " " << g_cuda << endl;
//
//    }
    return g;
}

floatingpoint CGMethod::maxF() {

    floatingpoint maxF = 0.0;
    floatingpoint mag = 0.0;
    for(int i = 0; i < N/3; i++) {
        mag = 0.0;
        for(int j = 0; j < 3; j++)
            mag += forceAux[3 * i + j]*forceAux[3 * i + j];
        mag = sqrt(mag);
//        std::cout<<"SL "<<i<<" "<<mag*mag<<" "<<forceAux[3 * i]<<" "<<forceAux[3 * i + 1]<<" "<<forceAux[3 * i +
//                2]<<endl;
        if(mag > maxF) maxF = mag;
    }

//    for(int i = 0; i < N; i++) {
//        mag = sqrt(forceAux[i]*forceAux[i]);
//        if(mag > maxF) maxF = mag;
//    }

    return maxF;
}

Bead* CGMethod::maxBead() {

    floatingpoint maxF = 0.0;
    floatingpoint currentF;
    long index = 0;
#ifdef SERIAL
    for (int i = 0; i < N/3; i++) {
        for (int j = 0 ;j< 3; j++) {
            currentF = forceAux[3*i+j] * forceAux[3*i+j];
        }
        if(currentF > maxF) {
            index = i;
            maxF = currentF;
        }
    }
#endif
#ifdef CUDAACCL
    floatingpoint F_i[N];
    floatingpoint gmaxF = 0.0;
    CUDAcommon::handleerror(hipDeviceSynchronize());
    CUDAcommon::handleerror(hipMemcpy(F_i, CUDAcommon::getCUDAvars().gpu_forceAux, N *
                                                                                 sizeof(floatingpoint), hipMemcpyDeviceToHost));
    floatingpoint gcurrentF;
//    long gindex = 0;

    for (int i = 0; i < N; i++) {

        gcurrentF = F_i[i] * F_i[i];
        if(gcurrentF > gmaxF) {
            index = (i - i%3)/3;
            gmaxF = gcurrentF;
//            std::cout<<gcurrentF<<" "<<forceAux[i] * forceAux[i]<<endl;
        }
    }
//    if(gindex!=index)
//        std::cout<<N<<endl;
//        std::cout<<"CPU and GPU codes do not point to same bead with maxF."<<endl;
#endif
    return Bead::getBeads()[index];
}

void CGMethod::moveBeads(totalenergyfloatingpoint d)
{
    ///<NOTE: Ignores static beads for now.
    //if(!b->getstaticstate())

//    std::cout<<"3N "<<N<<endl;
	totalenergyfloatingpoint temp;
    for (int i = 0; i < N; i++) {
    	temp = coord[i] + d * force[i];
        coord[i] = temp;
        cout<<"C&F "<<coord[i]<<" "<<force[i]<<" lambda "<<d<<endl;
    }
    cout<<"---"<<endl;
}

void CGMethod::shiftGradient(totalforcefloatingpoint d)
{
    for (int i = 0; i < N; i ++)
        force[i] = forceAux[i] + d * force[i];
}

void CGMethod::printForces()
{
    cout << "Print Forces" << endl;
    for(auto b: Bead::getBeads()) {

        for (int i = 0; i<3; i++)
            cout << b->coordinate[i] << "  "<<
                 b->force[i] <<"  "<<b->forceAux[i]<<endl;
    }
    cout << "End of Print Forces" << endl;
}

void CGMethod::startMinimization() {
    //COPY BEAD DATA
#ifdef CUDATIMETRACK
    chrono::high_resolution_clock::time_point tbegin, tend;
    tbegin = chrono::high_resolution_clock::now();
#endif
    coord = CUDAcommon::serlvars.coord;
//    N = 3 * Bead::getBeads().size();
        N = 3 * Bead::getmaxbindex();
    Ncyl = Cylinder::getCylinders().size();
    deallocate();
    allocate(N, Ncyl);


    //coord management
    long i = 0;
    long index = 0;
    for(auto b: Bead::getBeads()) {

        //set bead index
        b->_dbIndex = i;

        //flatten indices
        index = 3 * i;
        coord[index] = b->coordinate[0];
        coord[index + 1] = b->coordinate[1];
        coord[index + 2] = b->coordinate[2];
        b->coordinateP = b->coordinate;
        i++;
    }
    CUDAcommon::serlvars.coord = coord;
#ifdef CUDATIMETRACK
    tend= chrono::high_resolution_clock::now();
    chrono::duration<floatingpoint> elapsed_runst(tend - tbegin);
    CUDAcommon::cudatime.Tstartmin = elapsed_runst.count();
    std::cout<<"Start conv to vec time taken (s) "<<elapsed_runst.count()<<endl;
#endif
#ifdef CUDAACCL
#ifdef CUDATIMETRACK
    tbegin = chrono::high_resolution_clock::now();
#endif
    //Start stream
    if(stream_startmin == NULL || !(CUDAcommon::getCUDAvars().conservestreams))
        CUDAcommon::handleerror(hipStreamCreate(&stream_startmin));
    int nDevices;
//    hipDeviceProp_t prop;
    hipGetDeviceCount(&nDevices);
    if(nDevices>1){
        cout<<"Code not configured for multiple devices. Exiting..."<<endl;
        exit(EXIT_FAILURE);
    }

    floatingpoint f[N];
    for(auto iter=0;i<N;i++)
        f[iter]=0.0;
    floatingpoint* gpu_coord;
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_coord, N*sizeof(floatingpoint)));
    floatingpoint* gpu_lambda;
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_lambda, sizeof(floatingpoint)));
    floatingpoint* gpu_force;
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_force, N*sizeof(floatingpoint)));
    floatingpoint* gpu_forceAux;
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_forceAux, N*sizeof(floatingpoint)));
    floatingpoint* gpu_forceAuxP;
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_forceAuxP, N*sizeof(floatingpoint)));
    floatingpoint* gpu_energy;
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_energy, sizeof(floatingpoint)));
    bool* gpu_btstate;
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_btstate, sizeof(bool)));
    cylinder* gpu_cylindervec;
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_cylindervec, Ncyl*sizeof(cylinder)));

    CUDAcommon::handleerror(hipMalloc((void **) &gpu_initlambdalocal, sizeof(floatingpoint)));

    CUDAcommon::handleerror(hipMalloc((void **)&gpu_fmax, sizeof(floatingpoint)));
    CUDAcommon::handleerror(hipMalloc((void **)&g_currentenergy, sizeof(floatingpoint)));
    CUDAcommon::handleerror(hipMalloc((void **)&gpu_FDotF, sizeof(floatingpoint)));
    CUDAcommon::handleerror(hipMalloc((void **)&gpu_FADotFA, sizeof(floatingpoint)));
    CUDAcommon::handleerror(hipMalloc((void **)&gpu_FADotFAP, sizeof(floatingpoint)));
    CUDAcommon::handleerror(hipMalloc((void **)&gpu_FDotFA, sizeof(floatingpoint)));

    CUDAcommon::handleerror(hipHostAlloc((void**)&convergencecheck, 3 * sizeof(bool), hipHostMallocMapped));
    CUDAcommon::handleerror(hipHostGetDevicePointer(&gpu_convergencecheck, convergencecheck, 0));

    //PING PONG
    CUDAcommon::handleerror(hipMalloc(&g_stop1, sizeof(bool)));
    CUDAcommon::handleerror(hipMalloc(&g_stop2, sizeof(bool)));
    CUDAcommon::handleerror(hipHostAlloc(&h_stop, sizeof(bool), hipHostMallocDefault));

    //@
    //Store the pointers so they can be tracked while calculating energies.
    CUDAcommon::cudavars.backtrackbools.clear();
    CUDAcommon::cudavars.backtrackbools.push_back(g_stop1);
    CUDAcommon::cudavars.backtrackbools.push_back(g_stop2);

//    CUDAcommon::handleerror(hipHostAlloc((void**)&convergencecheck, 3 * sizeof(bool), hipHostMallocDefault));
//    CUDAcommon::handleerror(hipMalloc((void **) &gpu_convergencecheck, 3 * sizeof(bool)));

//    CUDAcommon::handleerror(hipMalloc((void **) &gpu_lambda, sizeof(floatingpoint))); REPEAT.
    CUDAcommon::handleerror(hipMemcpyAsync(gpu_coord, coord, N*sizeof(floatingpoint),
                                        hipMemcpyHostToDevice, stream_startmin));
    CUDAcommon::handleerror(hipMemcpyAsync(gpu_force, f, N*sizeof(floatingpoint),
                                        hipMemcpyHostToDevice, stream_startmin));
    CUDAcommon::handleerror(hipMemcpyAsync(gpu_forceAux, f, N*sizeof(floatingpoint),
                                        hipMemcpyHostToDevice, stream_startmin));
    CUDAcommon::handleerror(hipMemcpyAsync(gpu_forceAuxP, f, N*sizeof(floatingpoint),
                                        hipMemcpyHostToDevice, stream_startmin));
    bool dummy[1];dummy[0] = true;
    CUDAcommon::handleerror(hipMemcpyAsync(gpu_btstate, dummy, sizeof(bool),
                                        hipMemcpyHostToDevice, stream_startmin));

    CUDAcommon::handleerror(hipMemcpyAsync(gpu_cylindervec, cylindervec, Ncyl*sizeof
                                                    (cylinder),
                                            hipMemcpyHostToDevice, stream_startmin));
    int *gculpritID;
    char *gculpritFF;
    char *gculpritinteraction;
    int *culpritID;
    char *culpritFF;
    char *culpritinteraction;
    CUDAcommon::handleerror(hipHostAlloc((void**)&culpritID, 4 * sizeof(int), hipHostMallocMapped));
    CUDAcommon::handleerror(hipHostAlloc((void**)&culpritFF, 100*sizeof(char), hipHostMallocMapped));
    CUDAcommon::handleerror(hipHostAlloc((void**)&culpritinteraction, 100*sizeof(char), hipHostMallocMapped));
    CUDAcommon::handleerror(hipHostGetDevicePointer(&gculpritID, culpritID, 0));
    CUDAcommon::handleerror(hipHostGetDevicePointer(&gculpritFF, culpritFF, 0));
    CUDAcommon::handleerror(hipHostGetDevicePointer(&gculpritinteraction, culpritinteraction, 0));
//    CUDAcommon::handleerror(hipMalloc((void **) &gculpritID, sizeof(int)));
//    CUDAcommon::handleerror(hipMalloc((void **) &gculpritFF, 11*sizeof(char)));
//    char a[] = "FilamentFF";
//    CUDAcommon::handleerror(hipMemcpy(gculpritFF, a, 100 * sizeof(char), hipMemcpyHostToDevice));
//    CUDAcommon::handleerror(hipMalloc((void **) &gculpritinteraction, 100*sizeof(char)));

    CUDAvars cvars=CUDAcommon::getCUDAvars();
    cvars.gpu_coord=gpu_coord;
    cvars.gpu_lambda=gpu_lambda;
    cvars.gpu_forceAux = gpu_forceAux;
    cvars.gpu_force = gpu_force;
    cvars.gpu_forceAuxP = gpu_forceAuxP;
    cvars.gpu_energy = gpu_energy;
    cvars.gculpritID = gculpritID;
    cvars.culpritID = culpritID;
    cvars.gculpritinteraction = gculpritinteraction;
    cvars.gculpritFF = gculpritFF;
    cvars.culpritinteraction = culpritinteraction;
    cvars.culpritFF = culpritFF;
    cvars.gpu_btstate = gpu_btstate;
    cvars.gpu_cylindervec = gpu_cylindervec;
    CUDAcommon::cudavars=cvars;
//SET CERTAIN GPU PARAMETERS SET FOR EASY ACCESS DURING MINIMIZATION._
//    int THREADSPERBLOCK;
//    hipDeviceProp_t prop;
//    hipGetDeviceProperties(&prop, 0);
//    THREADSPERBLOCK = prop.maxThreadsPerBlock;
    //@{ Reduction Add variables
    bntaddvector.clear();
    bntaddvector = getaddred2bnt(N/3);
    int M = bntaddvector.at(0);
    vector<floatingpoint> zerovec(M);
    fill(zerovec.begin(),zerovec.begin()+M,0.0);
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_g, M * sizeof(floatingpoint)));
    CUDAcommon::handleerror(hipMemcpyAsync(gpu_g, zerovec.data(),
                           M * sizeof(floatingpoint), hipMemcpyHostToDevice, stream_startmin));
    /*CUDAcommon::handleerror(hipMemsetAsync(gpu_g, 0, M * sizeof(floatingpoint), stream_startmin));*/
    //MaxF
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_maxF, M * sizeof(floatingpoint)));
    CUDAcommon::handleerror(hipMemcpyAsync(gpu_maxF, zerovec.data(),
                                            M * sizeof(floatingpoint), hipMemcpyHostToDevice, stream_startmin));
    /*CUDAcommon::handleerror(hipMemsetAsync(gpu_maxF, 0, M * sizeof(floatingpoint), stream_startmin));*/
    int THREADSPERBLOCK = bntaddvector.at(1);
    //@}

    int nint[1]; nint[0]=CGMethod::N/3;
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_nint, sizeof(int)));
    CUDAcommon::handleerror(hipMemcpyAsync(gpu_nint, nint, sizeof(int),
                                        hipMemcpyHostToDevice, stream_startmin));
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_state, sizeof(int)));
    blocksnthreads.push_back(CGMethod::N/(3*THREADSPERBLOCK) + 1);
    if(blocksnthreads[0]==1) blocksnthreads.push_back(CGMethod::N/3);
    else blocksnthreads.push_back(THREADSPERBLOCK);
    auto maxthreads = 8 * THREADSPERBLOCK;

    //@{maxFredv3
    int state[1];state[0] = 0;
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_mutexlock, sizeof(int)));
    CUDAcommon::handleerror(hipMemcpyAsync(gpu_mutexlock, state, sizeof(int),
                                        hipMemcpyHostToDevice, stream_startmin));
    //Synchronize
    CUDAcommon::handleerror(hipStreamSynchronize(stream_startmin),"CGMethod.cu",
                            "startMinimization");

#ifdef CUDATIMETRACK
    tend= chrono::high_resolution_clock::now();
    chrono::duration<floatingpoint> elapsed_run(tend - tbegin);
    CUDAcommon::cudatime.Tstartmin = elapsed_run.count();
    std::cout<<"start min time taken (s) "<<elapsed_run.count()<<endl;
#endif

#ifdef CUDATIMETRACK
    CUDAcommon::cudatime.Tlambdap.clear();
    CUDAcommon::cudatime.Tlambdapcount.clear();
    CUDAcommon::cudatime.Tlambdap.push_back(0);
    CUDAcommon::cudatime.Tlambdap.push_back(0);
    CUDAcommon::cudatime.Tlambdap.push_back(0);
    CUDAcommon::cudatime.Tlambdapcount.push_back(0);
    CUDAcommon::cudatime.Tlambdapcount.push_back(0);
    CUDAcommon::cudatime.Tlambdapcount.push_back(0);
    //
    CUDAcommon::serltime.Tlambdap.clear();
    CUDAcommon::serltime.Tlambdapcount.clear();
    CUDAcommon::serltime.Tlambdap.push_back(0);
    CUDAcommon::serltime.Tlambdap.push_back(0);
    CUDAcommon::serltime.Tlambdap.push_back(0);
    CUDAcommon::serltime.Tlambdapcount.push_back(0);
    CUDAcommon::serltime.Tlambdapcount.push_back(0);
    CUDAcommon::serltime.Tlambdapcount.push_back(0);
#endif
    //@}
    //addvectorred2@{

//    int blocks, threads;
//    if(M > THREADSPERBLOCK){
//        if(M > maxthreads) {
//            blocks = 8;
//            threads = THREADSPERBLOCK;
//        }
//        else if(M > THREADSPERBLOCK){
//            blocks = M /(4 * THREADSPERBLOCK) +1;
//            threads = THREADSPERBLOCK;
//        }
//    }
//    else
//    { blocks = 1; threads = M/4;}
//    std::cout<<blocks<<" "<<threads<<" "<<M<<" "<<N/3<<" "<<maxthreads<<" "<<THREADSPERBLOCK<<endl;
//    bntaddvector.clear();
//    bntaddvector.push_back(blocks);
//    bntaddvector.push_back(threads);
//    CUDAcommon::handleerror(hipMalloc((void **) &gSum, sizeof(floatingpoint)));
//    CUDAcommon::handleerror(hipMalloc((void **) &gSum2, sizeof(floatingpoint)));
    //@}
//    CUDAcommon::handleerror(hipMalloc((void **) &gpu_g, N/3 * sizeof(floatingpoint)));

    //Memory alloted
    //@{
//    size_t allocmem = 0;
//    allocmem += (4*N + 9 + M)*sizeof(floatingpoint) + 6 * sizeof(bool) + 6 * sizeof(int) + 200 * sizeof(char);
//    auto c = CUDAcommon::getCUDAvars();
//    c.memincuda += allocmem;
//    CUDAcommon::cudavars = c;
//    std::cout<<"Total allocated memory KB"<<c.memincuda/1024<<endl;
//    std::cout<<"Memory allocated "<< allocmem/1024<<"Memory freed 0"<<endl;
    //@}


//    cvars.gpu_globalMem = prop.totalGlobalMem;
//    cvars.gpu_sharedMem = prop.sharedMemPerBlock;
//    floatingpoint a;
//    std::cout<<cvars.gpu_globalMem<<" "<<cvars.gpu_sharedMem<<" "<<sizeof(a)<<endl;
//
//    floatingpoint ccoord[N];
//    hipMemcpy(ccoord, gpu_coord, N*sizeof(floatingpoint), hipMemcpyDeviceToHost);
//    for(auto i=0;i<N;i++)
//        std::cout<<ccoord[i]<<" "<<coord[i]<<endl;

//    vector<floatingpoint> c2;c2.push_back(273.14);c2.push_back(273.14);
//    floatingpoint c2[2];
//    c2[0]=10.234;c2[1]=20.234;
//    floatingpoint *gpu_coord2;
//    hipMalloc((void **) &gpu_coord2, 2*sizeof(floatingpoint));
//    hipMemcpy(gpu_coord2, c2, 2*sizeof(floatingpoint), hipMemcpyHostToDevice);
//
//    floatingpoint cc[2];
//    hipMemcpy(cc, gpu_coord2, 2*sizeof(floatingpoint), hipMemcpyDeviceToHost);
//    std::cout<<cc[0]<<" "<<cc[1]<<endl;
//    hipFree(gpu_coord2);
//    hipFree(gpu_coord);
#endif
}

void CGMethod::endMinimization() {
#ifdef CUDATIMETRACK
    chrono::high_resolution_clock::time_point tbegin, tend;
    tbegin = chrono::high_resolution_clock::now();
#endif
#ifdef CUDAACCL

    CUDAcommon::handleerror(hipMemcpy(coord, CUDAcommon::getCUDAvars().gpu_coord, N *
                            sizeof(floatingpoint), hipMemcpyDeviceToHost));
    CUDAcommon::handleerror(hipMemcpy(force, CUDAcommon::getCUDAvars().gpu_force, N *
                            sizeof(floatingpoint), hipMemcpyDeviceToHost));
//    CUDAcommon::handleerror(hipMemcpy(forceAux, CUDAcommon::getCUDAvars().gpu_forceAux, N *
//                            sizeof(floatingpoint), hipMemcpyDeviceToHost));

    #endif
    ///RECOPY BEAD DATA
    //coord management
    long i = 0;
    long index = 0;
    for(auto b: Bead::getBeads()) {

        //flatten indices
        index = 3 * b->_dbIndex;
        b->coordinate[0] = coord[index];
        b->coordinate[1] = coord[index + 1];
        b->coordinate[2] = coord[index + 2];
//        std::cout<<"Bead "<<b->coordinate[0]<<" "<<b->coordinate[1]<<" "<<b->coordinate[2]<<endl;
        b->force[0] = force[index];
        b->force[1] = force[index +1];
        b->force[2] = force[index +2];

        i++;
    }

//    deallocate();
#ifdef CUDAACCL
    bool deletecndn = true;
#ifdef CUDAACCL_NLS
    deletecndn = false;
#endif
    if(deletecndn) {
        CUDAcommon::handleerror(hipFree(CUDAcommon::getCUDAvars().gpu_coord));
        CUDAcommon::handleerror(hipFree(CUDAcommon::getCUDAvars().gpu_cylindervec));
    }
    CUDAcommon::handleerror(hipFree(CUDAcommon::getCUDAvars().gpu_force));
    CUDAcommon::handleerror(hipFree(CUDAcommon::getCUDAvars().gpu_forceAux));
    CUDAcommon::handleerror(hipFree(CUDAcommon::getCUDAvars().gpu_forceAuxP));
    CUDAcommon::handleerror(hipFree(CUDAcommon::getCUDAvars().gpu_lambda));
    CUDAcommon::handleerror(hipFree(CUDAcommon::getCUDAvars().gpu_energy));
//    CUDAcommon::handleerror(hipFree(CUDAcommon::getCUDAvars().gculpritID));

    CUDAcommon::handleerror(hipFree(CUDAcommon::getCUDAvars().gpu_btstate));
    CUDAcommon::handleerror(hipFree(gpu_initlambdalocal));
//    CUDAcommon::handleerror(hipHostFree(CUDAcommon::getCUDAvars().culpritFF));
    CUDAcommon::handleerror(hipHostFree(CUDAcommon::getCUDAvars().culpritID));
    CUDAcommon::handleerror(hipHostFree(CUDAcommon::getCUDAvars().culpritFF));
    CUDAcommon::handleerror(hipHostFree(CUDAcommon::getCUDAvars().culpritinteraction));
    CUDAcommon::handleerror(hipFree(gpu_g));
//    CUDAcommon::handleerror(hipFree(gSum));
//    CUDAcommon::handleerror(hipFree(gSum2));
    CUDAcommon::handleerror(hipFree(gpu_fmax));
    CUDAcommon::handleerror(hipFree(gpu_FDotF));
    CUDAcommon::handleerror(hipFree(gpu_FADotFA));
    CUDAcommon::handleerror(hipFree(gpu_FADotFAP));
    CUDAcommon::handleerror(hipFree(gpu_FDotFA));
    CUDAcommon::handleerror(hipHostFree(convergencecheck));
    CUDAcommon::handleerror(hipFree(g_currentenergy));
    //PING PONG SAFEBACKTRACKING AND BACKTRACKING
    CUDAcommon::handleerror(hipFree(g_stop1));
    CUDAcommon::handleerror(hipFree(g_stop2));
    CUDAcommon::handleerror(hipHostFree(h_stop));
    //@
//    CUDAcommon::handleerror(hipFree(gpu_convergencecheck));


    CUDAcommon::handleerror(hipFree(gpu_nint));
    CUDAcommon::handleerror(hipFree(gpu_state));
    CUDAcommon::handleerror(hipFree(gpu_mutexlock));
    blocksnthreads.clear();
    if(!(CUDAcommon::getCUDAvars().conservestreams))
        CUDAcommon::handleerror(hipStreamDestroy(stream_startmin));

    //TODO cross check later
//    CUDAcommon::handleerror(hipFree(CUDAcommon::getCUDAvars().motorparams));

//    CUDAcommon::getCUDAvars().gpu_coord = NULL;
//    CUDAcommon::getCUDAvars().gpu_force = NULL;
//    CUDAcommon::getCUDAvars().gpu_forceAux = NULL;
//    CUDAcommon::getCUDAvars().gpu_lambda = NULL;

    //Memory alloted
    //@{
//    size_t allocmem = 0;
//    allocmem += (4*N + 9 +  bntaddvector.at(0))*sizeof(floatingpoint) + 6 * sizeof(bool) + 6 * sizeof(int) + 200 * sizeof(char);
//    auto c = CUDAcommon::getCUDAvars();
//    c.memincuda -= allocmem;
//    CUDAcommon::cudavars = c;
//    std::cout<<"Total allocated memory "<<c.memincuda/1024<<endl;
//    std::cout<<"Memory allocated 0 . Memory freed "<<allocmem/1024<<endl;
    //@}

//    size_t free, total;
//    CUDAcommon::handleerror(hipMemGetInfo(&free, &total));
//    fprintf(stdout,"\t### After Min Available VRAM : %g Mo/ %g Mo(total)\n\n",
//            free/1e6, total/1e6);
//
//    hipFree(0);
//
//    CUDAcommon::handleerror(hipMemGetInfo(&free, &total));
//    fprintf(stdout,"\t### Available VRAM : %g Mo/ %g Mo(total)\n\n",
//            free/1e6, total/1e6);
#endif
#ifdef CUDATIMETRACK
    tend= chrono::high_resolution_clock::now();
    chrono::duration<floatingpoint> elapsed_run(tend - tbegin);
    CUDAcommon::cudatime.Tstartmin = elapsed_run.count();
    std::cout<<"end min time taken (s) "<<elapsed_run.count()<<endl;
#endif
}

#ifdef CUDAACCL
floatingpoint CGMethod::backtrackingLineSearchCUDA(ForceFieldManager& FFM, floatingpoint MAXDIST,
                                        floatingpoint LAMBDAMAX, bool *gpu_safestate) {
#ifdef CUDATIMETRACK
    chrono::high_resolution_clock::time_point tbegin, tend;
    CUDAcommon::cudatime.Tlambdapcount.at(0)++;
    tbegin = chrono::high_resolution_clock::now();
#endif
    //@{ Lambda phase 1
    floatingpoint lambda;
    h_stop[0] = false;
    if(s1 == NULL || !(CUDAcommon::getCUDAvars().conservestreams))
        CUDAcommon::handleerror(hipStreamCreate(&s1));
    if(s2 == NULL || !(CUDAcommon::getCUDAvars().conservestreams))
        CUDAcommon::handleerror(hipStreamCreate(&s2));
    if(s3 == NULL || !(CUDAcommon::getCUDAvars().conservestreams))
        CUDAcommon::handleerror(hipStreamCreate(&s3));
    if(e1 == NULL || !(CUDAcommon::getCUDAvars().conservestreams))
        CUDAcommon::handleerror(hipEventCreate(&e1));
    if(e2 == NULL || !(CUDAcommon::getCUDAvars().conservestreams))
        CUDAcommon::handleerror(hipEventCreate(&e2));
    sp1 = &s1;
    sp2 = &s2;
    ep1 = &e1;
    ep2 = &e2;
    g_s1 = g_stop1;
    g_s2 = g_stop2;
    //prep for backtracking.
    if(gpu_params == NULL){
        //TODO move gpu_params copy permanently out of the function.
        floatingpoint params[5];
        params[0] = BACKTRACKSLOPE;
        params[1] = LAMBDAREDUCE;
        params[2] = LAMBDATOL;
        params[3] = LAMBDAMAX;
        params[4] = MAXDIST;
        CUDAcommon::handleerror(hipMalloc((void **) &gpu_params, 5 * sizeof(floatingpoint)));
        CUDAcommon::handleerror(hipMemcpy(gpu_params, params, 5 * sizeof(floatingpoint),
                                           hipMemcpyHostToDevice));
    }
    CUDAresetlambda(*sp1);//set lambda to zero.
    if(e == NULL || !(CUDAcommon::getCUDAvars().conservestreams))  {
        CUDAcommon::handleerror(hipEventCreate(&e));
    }

    CUDAcommon::handleerror(hipEventRecord(e, *sp1));
    auto cvars = CUDAcommon::getCUDAvars();
    cvars.streamvec.clear();
    CUDAcommon::cudavars = cvars;
    //initialize lambda search
    CUDAinitializeLambda(*sp1, g_s1, g_s2, gpu_safestate, gpu_state);
    //@} Lambda phase 1
#ifdef CUDATIMETRACK
    tend= chrono::high_resolution_clock::now();
    chrono::duration<floatingpoint> elapsed_run(tend - tbegin);
    CUDAcommon::cudatime.Tlambdap.at(0) += elapsed_run.count();
#endif
    //Calculate current energy.
    totalenergyfloatingpoint currentEnergy = FFM.computeEnergy(coord, force, 0.0);
    //wait for energies to be calculated
    for(auto strm:CUDAcommon::getCUDAvars().streamvec)
        CUDAcommon::handleerror(hipStreamSynchronize(*strm),"backConvSync","CGMethod.cu");
    if(stream_bt == NULL || !(CUDAcommon::getCUDAvars().conservestreams))
        CUDAcommon::handleerror(hipStreamCreate(&stream_bt),"find lambda", "CGMethod.cu");

#ifdef DETAILEDOUTPUT_ENERGY
//    CUDAcommon::handleerror(hipDeviceSynchronize());
    floatingpoint cuda_energy[1];
    CUDAcommon::handleerror(hipMemcpy(cuda_energy, CUDAcommon::cudavars.gpu_energy,  sizeof(floatingpoint),
                                       hipMemcpyDeviceToHost));
    std::cout<<"Total Energy cE pN.nm CUDA "<<cuda_energy[0]<<" SERL "<<currentEnergy<<endl;
    std::cout<<endl;
#endif

#ifdef CUDATIMETRACK
    tbegin = chrono::high_resolution_clock::now();
#endif
    //@{ Lambda phase 1b
    hipStreamSynchronize(*sp1);
    setcurrentenergy<<<1,1,0,*sp1>>>(CUDAcommon::getCUDAvars().gpu_energy, g_currentenergy, CUDAcommon::getCUDAvars()
            .gpu_lambda, gpu_initlambdalocal);
    CUDAcommon::handleerror(hipGetLastError(),"setcurrentenergy", "CGMethod.cu");
    hipStreamSynchronize(*sp1);

    //check if converged.
    //TODO commented coz this line is not needed
//    CUDAcommon::handleerror(hipStreamWaitEvent(s3, *ep1, 0));
//    CUDAcommon::handleerror(hipEventRecord(*CUDAcommon::getCUDAvars().event, *sp1));
    CUDAcommon::handleerror(hipMemcpyAsync(h_stop, g_s2, sizeof(bool), hipMemcpyDeviceToHost, s3));
//    CUDAcommon::handleerror(hipStreamSynchronize (*sp1)); CHECK IF NEEDED
    cconvergencecheck = h_stop;
    int iter = 0;
    //@} Lambda phase 1b
#ifdef CUDATIMETRACK
    tend= chrono::high_resolution_clock::now();
    chrono::duration<floatingpoint> elapsed_run1b(tend - tbegin);
    CUDAcommon::cudatime.Tlambdap.at(0) += elapsed_run1b.count();
#endif

    while(!(cconvergencecheck[0])) {
#ifdef CUDATIMETRACK
        CUDAcommon::cudatime.Tlambdapcount.at(1)++;
        tbegin = chrono::high_resolution_clock::now();
#endif
        //@{ Lambda phase 2
        iter++;
        CUDAcommon::handleerror(hipStreamWaitEvent(*sp2, *ep1, 0));
        CUDAcommon::handleerror(hipStreamSynchronize(*sp2));
        //ping pong swap
        sps = sp1;
        sp1 = sp2;
        sp2 = sps;
        eps = ep1;
        ep1 = ep2;
        ep2 = eps;
//        g_bs = g_b1;
//        g_b1 = g_b2;
//        g_b2 = g_bs;
        g_ss = g_s1;
        g_s1 = g_s2;
        g_s2 = g_ss;

        auto cvars = CUDAcommon::getCUDAvars();
        cvars.streamvec.clear();
//        cvars.event = ep1;
        CUDAcommon::cudavars = cvars;
        //@} Lambda phase 2
#ifdef CUDATIMETRACK
        tend= chrono::high_resolution_clock::now();
        chrono::duration<floatingpoint> elapsed_run2(tend - tbegin);
        CUDAcommon::cudatime.Tlambdap.at(1) += elapsed_run2.count();
#endif

#ifdef SERIAL_CUDACROSSCHECK
        floatingpoint cuda_lambda[1];
        CUDAcommon::handleerror(hipDeviceSynchronize(),"CGPolakRibiereMethod.cu","CGPolakRibiereMethod.cu");
        CUDAcommon::handleerror(hipMemcpy(cuda_lambda, CUDAcommon::cudavars.gpu_lambda,  sizeof(floatingpoint),
                                           hipMemcpyDeviceToHost));
        lambda = cuda_lambda[0];
#endif

        //TODO let each forcefield calculate energy IFF conv state = false. That will help
        // them avoid unnecessary iterations.
        //let each forcefield also add energies to two different energy variables.
        totalenergyfloatingpoint energyLambda = FFM.computeEnergy(coord, force, lambda);

        //wait for energies to be calculated
         for(auto strm:CUDAcommon::getCUDAvars().streamvec) {
            CUDAcommon::handleerror(hipStreamSynchronize(*strm), "backConvsync", "CGMethod.cu");
        }
#ifdef SERIAL_CUDACROSSCHECK
        for(auto strm:CUDAcommon::getCUDAvars().streamvec) {
            CUDAcommon::handleerror(hipStreamSynchronize(*strm), "backConvsync", "CGMethod.cu");
        }
        CUDAcommon::handleerror(hipDeviceSynchronize());
        floatingpoint cuda_energy[1];
        CUDAcommon::handleerror(hipMemcpy(cuda_energy, CUDAcommon::cudavars.gpu_energy,  sizeof(floatingpoint),
                                           hipMemcpyDeviceToHost));
        std::cout<<"Total Energy EL pN.nm CUDA "<<cuda_energy[0]<<" SERL "
                ""<<energyLambda<<endl;
        std::cout<<endl;
#endif
#ifdef CUDATIMETRACK
        tbegin = chrono::high_resolution_clock::now();
#endif
        //@{ Lambda phase 2
        if(!(cconvergencecheck[0])){
            CUDAcommon::handleerror(hipStreamSynchronize(stream_bt));
            CUDAfindLambda(*sp1, stream_bt, *ep1, g_s1, g_s2, gpu_safestate, gpu_state);
            CUDAcommon::handleerror(hipStreamSynchronize(*sp1));
            CUDAcommon::handleerror(hipStreamSynchronize(stream_bt));
            if(cconvergencecheck[0]  == false){
                CUDAcommon::handleerror(hipStreamWaitEvent(s3, *ep1, 0));
                CUDAcommon::handleerror(hipMemcpyAsync(h_stop, g_s2, sizeof(bool), hipMemcpyDeviceToHost, s3));
            }
        }
        //@Lambda phase 2
#ifdef CUDATIMETRACK
        tend= chrono::high_resolution_clock::now();
        chrono::duration<floatingpoint> elapsed_run2b(tend - tbegin);
        CUDAcommon::cudatime.Tlambdap.at(1) += elapsed_run2b.count();
#endif
    }
    if(!(CUDAcommon::getCUDAvars().conservestreams))
        CUDAcommon::handleerror(hipFree(gpu_params), "CudaFree", "CGMethod.cu");
#ifdef CUDATIMETRACK
    CUDAcommon::cudatime.Tlambdapcount.at(2)++;
    tbegin = chrono::high_resolution_clock::now();
#endif
    //@{ Lambda phase 3
    //commented on 18 Sep 2018.
//    correctlambdaCUDA<<<1,1,0, stream_bt>>>(CUDAcommon::getCUDAvars().gpu_lambda, gpu_state, gpu_params);

/*    correctlambdaCUDA<<<1,1,0, *sp1>>>(CUDAcommon::getCUDAvars().gpu_lambda, gpu_state,
            gpu_params);*/

    CUDAcommon::handleerror(hipStreamSynchronize(stream_bt));
    CUDAcommon::handleerror(hipStreamSynchronize(s1));
    CUDAcommon::handleerror(hipStreamSynchronize(s2));
    CUDAcommon::handleerror(hipStreamSynchronize(s3));
    //@} Lambda phase 3
#ifdef CUDATIMETRACK
    tend= chrono::high_resolution_clock::now();
    chrono::duration<floatingpoint> elapsed_run3(tend - tbegin);
    CUDAcommon::cudatime.Tlambdap.at(2) += elapsed_run3.count();
#endif
    if(!(CUDAcommon::getCUDAvars().conservestreams))  {
        CUDAcommon::handleerror(hipStreamDestroy(s1));
        CUDAcommon::handleerror(hipStreamDestroy(s2));
        CUDAcommon::handleerror(hipStreamDestroy(s3));
        CUDAcommon::handleerror(hipStreamDestroy(stream_bt));
        CUDAcommon::handleerror(hipEventDestroy(e1));
        CUDAcommon::handleerror(hipEventDestroy(e2));
    }
    std::cout<<"CUDA lambda determined in "<<iter<< " iterations "<<endl;

    if(cconvergencecheck[0]||sconvergencecheck)
        return lambda;

}
#endif // CUDAACCL

totalenergyfloatingpoint CGMethod::backtrackingLineSearch(ForceFieldManager& FFM, floatingpoint MAXDIST,
                                        floatingpoint LAMBDAMAX, bool *gpu_safestate) {

    //@{ Lambda phase 1
    totalenergyfloatingpoint lambda;
    sconvergencecheck = true;
#ifdef SERIAL //SERIAL
    sconvergencecheck = false;
    cconvergencecheck = new bool[1];
    cconvergencecheck[0] = true;
#endif
#ifdef SERIAL
    floatingpoint f = maxF();
    //return zero if no forces
    if(f == 0.0){
        lambda = 0.0;
#ifdef DETAILEDOUTPUT_LAMBDA
        std::cout<<"initial_lambda_serial "<<lambda<<endl;
#endif
        sconvergencecheck = true;}
    //calculate first lambda
    lambda = min(LAMBDAMAX, MAXDIST / f);

    //@} Lambda phase 1
#ifdef DETAILEDOUTPUT_LAMBDA
    std::cout<<"SL lambdamax "<<LAMBDAMAX<<" serial_lambda "<<lambda<<" fmax "<<f<<" state "<<sconvergencecheck<<endl;
#endif
#endif
    totalenergyfloatingpoint currentEnergy = FFM.computeEnergy(coord, force, 0.0);
#ifdef DETAILEDOUTPUT_ENERGY
    CUDAcommon::handleerror(hipDeviceSynchronize());
    floatingpoint cuda_energy[1];
    CUDAcommon::handleerror(hipMemcpy(cuda_energy, CUDAcommon::cudavars.gpu_energy,  sizeof(floatingpoint),
                                       hipMemcpyDeviceToHost));
    std::cout<<"Total Energy CE pN.nm CUDA "<<cuda_energy[0]<<" SERL "<<currentEnergy<<endl;
    std::cout<<endl;
#endif

    int iter = 0;
    while(!(cconvergencecheck[0])||!(sconvergencecheck)) {
        iter++;
        //TODO let each forcefield calculate energy IFF conv state = false. That will help
        // them avoid unnecessary iterations.
        //let each forcefield also add energies to two different energy variables.
        totalenergyfloatingpoint energyLambda = FFM.computeEnergy(coord, force, lambda);
#ifdef DETAILEDOUTPUT_ENERGY
        CUDAcommon::handleerror(hipDeviceSynchronize());
        floatingpoint cuda_energy[1];
        CUDAcommon::handleerror(hipMemcpy(cuda_energy, CUDAcommon::cudavars.gpu_energy,  sizeof(floatingpoint),
                                           hipMemcpyDeviceToHost));
        std::cout<<"Total Energy EL pN.nm CUDA "<<cuda_energy[0]<<" SERL "
                ""<<energyLambda<<endl;
        std::cout<<endl;
#endif

#ifdef SERIAL
        //@{ Lambda phase 2
        if(!(sconvergencecheck)){
            totalenergyfloatingpoint idealEnergyChange = -BACKTRACKSLOPE * lambda *
                    allFDotFA();
            totalenergyfloatingpoint energyChange = energyLambda - currentEnergy;
#ifdef DETAILEDOUTPUT_LAMBDA
            std::cout<<"BACKTRACKSLOPE "<<BACKTRACKSLOPE<<" lambda "<<lambda<<" allFDotFA"
                    " "<<allFDotFA()<<endl;
            std::cout<<"SL energyChange "<<energyChange<<" idealEnergyChange "
                    ""<<idealEnergyChange<<endl;
#endif
            //return if ok
            if(energyChange <= idealEnergyChange) {
                sconvergencecheck = true;}
            else
                //reduce lambda
                lambda *= LAMBDAREDUCE;

            if(lambda <= 0.0 || lambda <= LAMBDATOL) {
                sconvergencecheck = true;
                lambda = 0.0;

            }
#ifdef DETAILEDOUTPUT_LAMBDA
            std::cout<<"SL2 BACKTRACKSLOPE "<<BACKTRACKSLOPE<<" lambda "<<lambda<<" allFDotFA "
                                                                                <<allFDotFA()<<endl;
            std::cout<<"SL2 energyChange "<<energyChange<<" idealEnergyChange "
                    ""<<idealEnergyChange
                     <<" lambda "<<lambda<<" state "<<sconvergencecheck<<endl;
#endif
            cout<<" lambda "<<lambda<<endl;
            std::cout<<"SL2 BACKTRACKSLOPE "<<BACKTRACKSLOPE<<" allFDotFA "
                     <<allFDotFA()<<endl;
            std::cout<<"SL2 energyChange "<<energyChange<<" idealEnergyChange "
                                                          ""<<idealEnergyChange
                     <<" energylambda "<<energyLambda<<" state "<<sconvergencecheck<<endl;
        }
        //@{ Lambda phase 2

#endif
    }
    std::cout<<"lambda determined in "<<iter<< " iterations. FL "<<lambda<<endl;
//synchronize streams
    if(cconvergencecheck[0]||sconvergencecheck) {
#ifdef SERIAL
        delete [] cconvergencecheck;
#endif
        return lambda;
    }

}

totalenergyfloatingpoint CGMethod::safeBacktrackingLineSearch(ForceFieldManager& FFM, floatingpoint MAXDIST,
                                            floatingpoint LAMBDAMAX, bool *gpu_safestate) {
    //reset safe mode
    _safeMode = false;
    sconvergencecheck = true;
    //calculate first lambda
    totalenergyfloatingpoint lambda = LAMBDAMAX;
//    std::cout<<"safe 0"<<endl;
#ifdef SERIAL //SERIAL
    sconvergencecheck = false;
    cconvergencecheck = new bool[1];
    cconvergencecheck[0] = true;
#endif
//prepare for ping pong optimization
    totalenergyfloatingpoint currentEnergy = FFM.computeEnergy(coord, force, 0.0);
#ifdef DETAILEDOUTPUT_ENERGY
    CUDAcommon::handleerror(hipDeviceSynchronize());
    floatingpoint cuda_energy[1];
    CUDAcommon::handleerror(hipMemcpy(cuda_energy, CUDAcommon::cudavars.gpu_energy,  sizeof(floatingpoint),
                                       hipMemcpyDeviceToHost));
    std::cout<<"Total Energy CE pN.nm CUDA "<<cuda_energy[0]<<" SERL "<<currentEnergy<<endl;
    std::cout<<endl;
#endif
    int iter =0;
    //safe backtracking loop
    std::cout<<"safe z"<<endl;
    while(!(cconvergencecheck[0])||!(sconvergencecheck)) {
        //new energy when moved by lambda
        iter++;
        totalenergyfloatingpoint energyLambda = FFM.computeEnergy(coord, force, lambda);
#ifdef DETAILEDOUTPUT_ENERGY
        CUDAcommon::handleerror(hipDeviceSynchronize());
        floatingpoint cuda_energy[1];
        CUDAcommon::handleerror(hipMemcpy(cuda_energy, CUDAcommon::cudavars.gpu_energy,  sizeof(floatingpoint),
                                           hipMemcpyDeviceToHost));
        std::cout<<"Total Energy EL pN.nm CUDA "<<cuda_energy[0]<<" SERL "
                ""<<energyLambda<<endl;
        std::cout<<endl;
#endif

#ifdef SERIAL
        if(!(sconvergencecheck)){
            totalenergyfloatingpoint energyChange = energyLambda - currentEnergy;

            //return if ok
            if(energyChange <= 0.0) sconvergencecheck = true;
            else
                //reduce lambda
                lambda *= LAMBDAREDUCE;

            //just shake if we cant find an energy min,
            //so we dont get stuck
            if(lambda <= 0.0 || lambda <= LAMBDATOL) {
                lambda = MAXDIST / maxF();
                sconvergencecheck = true;
            }
            cout<<"Safe energyChange "<<energyChange<<" maxF"<<maxF()<<" MAXDIST "
                                                                       ""<<MAXDIST<<endl;
        }

#endif
    }
    std::cout<<"lambda determined in "<<iter<< " iterations. FL "<<lambda<<endl;
    if(cconvergencecheck[0]||sconvergencecheck) {
#ifdef SERIAL
        delete [] cconvergencecheck;
#endif
        return lambda;
    }
}
