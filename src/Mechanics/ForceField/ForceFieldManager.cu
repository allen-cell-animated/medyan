#include "hip/hip_runtime.h"

//------------------------------------------------------------------
//  **MEDYAN** - Simulation Package for the Mechanochemical
//               Dynamics of Active Networks, v3.1
//
//  Copyright (2015-2016)  Papoian Lab, University of Maryland
//
//                 ALL RIGHTS RESERVED
//
//  See the MEDYAN web page for more information:
//  http://www.medyan.org
//------------------------------------------------------------------

#include "ForceFieldManager.h"
#include "ForceFieldManagerCUDA.h"

#include "CGMethod.h"
#include "cross_check.h"

void ForceFieldManager::vectorizeAllForceFields() {

    for(auto &ff : _forceFields)
        ff->vectorize();
#ifdef CUDAACCL
    CUDAcommon::handleerror(hipStreamCreate(&stream));
    int nint[1]; nint[0]=CGMethod::N/3;
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_nint, sizeof(int)));
    CUDAcommon::handleerror(hipMemcpy(gpu_nint, nint, sizeof(int), hipMemcpyHostToDevice));
    blocksnthreads.push_back(CGMethod::N/(3*THREADSPERBLOCK) + 1);
    if(blocksnthreads[0]==1) blocksnthreads.push_back(CGMethod::N/3);
    else blocksnthreads.push_back(THREADSPERBLOCK);
#endif
}

void ForceFieldManager::cleanupAllForceFields() {

    for(auto &ff : _forceFields)
        ff->cleanup();
#ifdef CUDAACCL
    CUDAcommon::handleerror(hipStreamDestroy(stream));
    if(CGMethod::N/3 > 0) {
        CUDAcommon::handleerror(hipFree(gpu_nint));
        blocksnthreads.clear();
    }
#endif
}

double ForceFieldManager::computeEnergy(double *coord, double *f, double d, bool verbose) {

    double energy = 0;
#ifdef CUDAACCL
    auto gU_tot = CUDAcommon::getCUDAvars().gpu_energy;
    setenergytozero<<<1,1,0,stream>>>(gU_tot);
    CUDAcommon::handleerror(hipStreamSynchronize(stream));
#endif
//    std::cout<<"-------"<<endl;
    for(auto &ff : _forceFields) {
//        std::cout<<ff->getName()<<endl;
//        std::cout<<"ForceField "<<ff->getName()<<endl;
        auto tempEnergy = ff->computeEnergy(coord, f, d);
//        CUDAcommon::handleerror(hipDeviceSynchronize());

        if(verbose) cout << ff->getName() << " energy = " << tempEnergy << endl;

        //if energy is infinity, exit with infinity.
        if(tempEnergy <= -1) {

            //if this is the current energy, exit ungracefully
            if(d == 0.0) {

                cout << "Energy = " << tempEnergy << endl;

                cout << "Energy of system became infinite. Try adjusting minimization parameters." << endl;
                cout << "The culprit was ... " << ff->getName() << endl;

                //get the culprit in output
                ff->whoIsCulprit();

                exit(EXIT_FAILURE);
            }
                //if this is a minimization try, just return infinity
            else return numeric_limits<double>::infinity();
        }
        else energy += tempEnergy;

    }
//    std::cout<<"-------"<<endl;
    return energy;
}
void ForceFieldManager::computeForces(double *coord, double *f) {
    //reset to zero
    for (int i = 0; i < CGMethod::N; i++)
        f[i] = 0.0;

#ifdef CUDAACCL
    CUDAvars cvars=CUDAcommon::getCUDAvars();
    hipStream_t  stream;
    CUDAcommon::handleerror(hipStreamCreate( &stream));
    if(cross_checkclass::Aux)
        resetForcesCUDA<<<blocksnthreads[0],blocksnthreads[1],0,stream>>>(cvars.gpu_forceAux, gpu_nint);
    else
        resetForcesCUDA<<<blocksnthreads[0],blocksnthreads[1],0,stream>>>(cvars.gpu_force, gpu_nint);
    CUDAcommon::handleerror(hipStreamSynchronize(stream));
    CUDAcommon::handleerror(hipStreamDestroy(stream));

    CUDAcommon::handleerror( hipGetLastError() ,"resetForcesCUDA", "ForceFieldManager.cu");
#endif
    //recompute
    double *F_i = new double[CGMethod::N];
    for(auto &ff : _forceFields) {
        ff->computeForces(coord, f);
//        CUDAcommon::handleerror(hipDeviceSynchronize());
//        std::cout<<"ForceField "<<ff->getName()<<endl;
//        if(cross_checkclass::Aux)
//            CUDAcommon::handleerror(
//                hipMemcpy(F_i, CUDAcommon::getCUDAvars().gpu_forceAux, 3 * Bead::getBeads().size() * sizeof
//                                   (double),
//                           hipMemcpyDeviceToHost));
//        else
//            CUDAcommon::handleerror(
//                    hipMemcpy(F_i, CUDAcommon::getCUDAvars().gpu_force, 3 * Bead::getBeads().size() * sizeof
//                                       (double),
//                               hipMemcpyDeviceToHost));
//        double fmax = 0.0;
//        int id=0;
//        for (auto iter = 0; iter < Bead::getBeads().size(); iter++) {
//            if(abs(F_i[3 *iter])> fmax) {fmax = abs(F_i[3*iter]);id = iter;}
//            if(abs(F_i[3 *iter +1])> fmax) {fmax = abs(F_i[3*iter +1]);id = iter;}
//            if(abs(F_i[3 *iter +2])> fmax) {fmax = abs(F_i[3*iter +2]);id = iter;}
////            std::cout << F_i[3 * iter] << " " << F_i[3 * iter + 1] << " " << F_i[3 * iter + 2] << endl;
//        }
//        std::cout <<"Fmax "<< id<<" "<<fmax<<" "<<F_i[3 * id] << " " << F_i[3 * id + 1] << " " << F_i[3 * id + 2] <<
//                                                                                                                 endl;
    }
    delete F_i;
}

void ForceFieldManager::computeLoadForces() {

    //reset
    for (auto b: Bead::getBeads()) {

        b->loadForcesP.clear();
        b->loadForcesM.clear();
    }

    for(auto &f : _forceFields)
        f->computeLoadForces();

    //reset lfi as well
    for(auto b: Bead::getBeads()) {
        b->lfip = 0;
        b->lfim = 0;
    }
}

void ForceFieldManager::copyForces(double *fprev, double *f) {

    for (int i = 0; i < CGMethod::N; i++)
        fprev[i] = f[i];
}

#ifdef CUDAACCL
void ForceFieldManager::CUDAcopyForces(hipStream_t stream, double *fprev, double *f) {


//    CUDAcommon::handleerror(hipFree(CUDAcommon::getCUDAvars().gpu_forceAux));
//    double* gpu_forceAux;
//    CUDAcommon::handleerror(hipMalloc((void **) &gpu_forceAux, CGMethod::N * sizeof(double)));
//    CUDAvars cvars=CUDAcommon::getCUDAvars();
//    cvars.gpu_forceAux=gpu_forceAux;
//    CUDAcommon::cudavars=cvars;

//    std::cout<<"Copyforces Number of Blocks: "<<blocksnthreads[0]<<endl;
//    std::cout<<"Threads per block: "<<blocksnthreads[1]<<endl;
    copyForcesCUDA<<<blocksnthreads[0],blocksnthreads[1],0,stream>>>(f, fprev, gpu_nint);
    CUDAcommon::handleerror( hipGetLastError(),"copyForcesCUDA", "ForceFieldManager.cu");
}
#endif