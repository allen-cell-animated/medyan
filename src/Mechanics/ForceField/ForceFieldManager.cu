#include "hip/hip_runtime.h"

//------------------------------------------------------------------
//  **MEDYAN** - Simulation Package for the Mechanochemical
//               Dynamics of Active Networks, v3.1
//
//  Copyright (2015-2016)  Papoian Lab, University of Maryland
//
//                 ALL RIGHTS RESERVED
//
//  See the MEDYAN web page for more information:
//  http://www.medyan.org
//------------------------------------------------------------------

#include "ForceFieldManager.h"
#include "ForceFieldManagerCUDA.h"

#include "CGMethod.h"
#include "cross_check.h"
#include <algorithm>

void ForceFieldManager::vectorizeAllForceFields() {
#ifdef CUDATIMETRACK
    chrono::high_resolution_clock::time_point tbegin, tend;
    CUDAcommon::cudatime.TvectorizeFF = 0.0;
    CUDAcommon::cudatime.TvecvectorizeFF.clear();
#endif
#ifdef CUDAACCL
    // PT1 Generate single vector of energies from all FF and add them together.
    //@{
    CUDAcommon::cudavars.offset_E=0.0;
    //@}
#endif

    for (auto &ff : _forceFields)
        ff->vectorize();

#ifdef CUDATIMETRACK
    tbegin = chrono::high_resolution_clock::now();
#endif
#ifdef CUDAACCL
    //reset offset
    if (streamF == NULL || !(CUDAcommon::getCUDAvars().conservestreams))
        CUDAcommon::handleerror(hipStreamCreate(&streamF));
    int nint[1];
    nint[0] = CGMethod::N / 3;
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_nint, sizeof(int)));
    CUDAcommon::handleerror(hipMemcpyAsync(gpu_nint, nint, sizeof(int),
                                        hipMemcpyHostToDevice, streamF));
    CUDAcommon::handleerror(hipMalloc((void **) &(CUDAcommon::cudavars.gpu_energyvec),
                                       CUDAcommon::cudavars.offset_E * sizeof(floatingpoint)));
    int THREADSPERBLOCK;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    THREADSPERBLOCK = prop.maxThreadsPerBlock;

    blocksnthreads.push_back(CGMethod::N / (3 * THREADSPERBLOCK) + 1);
    if (blocksnthreads[0] == 1) blocksnthreads.push_back(CGMethod::N / 3);
    else blocksnthreads.push_back(THREADSPERBLOCK);

    // PT2 Generate single vector of energies from all FF and add them together.
    //@{
//    std::cout<<"CUDA energy total nint "<<CUDAcommon::cudavars.offset_E<<endl;
    bntaddvec2.clear();
    bntaddvec2 = getaddred2bnt(CUDAcommon::cudavars.offset_E);
    CUDAcommon::handleerror(hipMalloc((void **) &(CUDAcommon::cudavars.gpu_energyvec), bntaddvec2.at
            (0)*sizeof (floatingpoint)));
    vector<floatingpoint> zerovec(bntaddvec2.at(0));
    fill(zerovec.begin(),zerovec.begin()+bntaddvec2.at(0),0.0);
    CUDAcommon::handleerror(hipMemcpyAsync(CUDAcommon::cudavars.gpu_energyvec, zerovec.data(),
                            bntaddvec2.at(0) * sizeof(floatingpoint), hipMemcpyHostToDevice,streamF));
/*    CUDAcommon::handleerror(hipMemsetAsync(CUDAcommon::cudavars.gpu_energyvec, 0,
                                            bntaddvec2.at(0) * sizeof(floatingpoint), streamF));*/

    params.clear();
    params.push_back(CUDAcommon::cudavars.offset_E);
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_params, sizeof(int)));
    CUDAcommon::handleerror(hipMemcpyAsync(gpu_params, params.data(), sizeof(int),
                                       hipMemcpyHostToDevice, streamF));
    //@}
#endif
#ifdef CUDATIMETRACK
//    CUDAcommon::handleerror(hipDeviceSynchronize(),"CGPolakRibiereMethod.cu","CGPolakRibiereMethod.cu");
    tend= chrono::high_resolution_clock::now();
    chrono::duration<floatingpoint> elapsed_run(tend - tbegin);
    CUDAcommon::cudatime.TvectorizeFF += elapsed_run.count();
    std::cout<<"Time total vectorizeFF (s) "<<CUDAcommon::cudatime.TvectorizeFF<<endl;
    std::cout<<"Time split vectorizeFF (s) ";
    for(auto x:CUDAcommon::cudatime.TvecvectorizeFF)
        std::cout<<x<<" ";
    std::cout<<endl;
#endif
}

void ForceFieldManager::cleanupAllForceFields() {

    for (auto &ff : _forceFields)
        ff->cleanup();
#ifdef CUDAACCL
    if (!(CUDAcommon::getCUDAvars().conservestreams))
        CUDAcommon::handleerror(hipStreamDestroy(streamF));
    //cleanup energy vector
    // single vector of energies from all FF and add them together.
    //@{
    CUDAcommon::handleerror(hipFree(CUDAcommon::cudavars.gpu_energyvec));
    CUDAcommon::handleerror(hipFree(gpu_params));
    //@}
    if (CGMethod::N / 3 > 0) {
        CUDAcommon::handleerror(hipFree(gpu_nint));
        //Memory alloted
        //@{
//        size_t allocmem = 0;
//        allocmem += sizeof(floatingpoint);
//        auto c = CUDAcommon::getCUDAvars();
//        c.memincuda -= allocmem;
//        CUDAcommon::cudavars = c;
//        std::cout<<"Total allocated memory "<<c.memincuda/1024<<endl;
//        std::cout<<"Memory allocated 0 . Memory freed "<<allocmem/1024<<endl;
        //@}
        blocksnthreads.clear();
    }
#endif
}

floatingpoint ForceFieldManager::computeEnergy(floatingpoint *coord, floatingpoint *f, floatingpoint d, bool verbose) {
#ifdef CUDATIMETRACK
    chrono::high_resolution_clock::time_point tbegin, tend;
//    CUDAcommon::cudatime.TcomputeE = 0.0;
    CUDAcommon::cudatime.TveccomputeE.clear();
    CUDAcommon::cudatime.Ecount++;
//    CUDAcommon::serltime.TcomputeE = 0.0;
    CUDAcommon::serltime.TveccomputeE.clear();
    CUDAcommon::serltime.Ecount++;
#endif
    floatingpoint energy = 0.0;
#ifdef CUDAACCL
#ifdef CUDA_INDIVIDUAL_ESUM
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_Uvec, sizeof (floatingpoint)));
    CUDAcommon::handleerror(hipMemset(gpu_Uvec, 0.0, sizeof (floatingpoint)));
#else
    floatingpoint *gpu_Uvec = CUDAcommon::getCUDAvars().gpu_energy;
    /*floatingpoint *gpu_Uvec;
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_Uvec, sizeof (floatingpoint)));
    CUDAcommon::handleerror(hipMemsetAsync(gpu_Uvec, 0, sizeof (floatingpoint),streamF));*/
#endif
#ifdef SERIAL_CUDACROSSCHECK
    CUDAcommon::handleerror(hipMemset(CUDAcommon::cudavars.gpu_energyvec, 0, bntaddvec2.at(0) * sizeof
            (floatingpoint)));
#endif
#ifdef CUDATIMETRACK
    tbegin = chrono::high_resolution_clock::now();
#endif
/*    auto gU_tot = CUDAcommon::getCUDAvars().gpu_energy;
    setenergytozero << < 1, 1, 0, streamF >> > (gU_tot);*/
    CUDAcommon::handleerror(hipStreamSynchronize(streamF));
#ifdef CUDATIMETRACK
//    CUDAcommon::handleerror(hipDeviceSynchronize(),"ForceFieldManager.cu",
//                            "computeEnergy");
    tend= chrono::high_resolution_clock::now();
    chrono::duration<floatingpoint> elapsed_run(tend - tbegin);
    CUDAcommon::cudatime.TcomputeE += elapsed_run.count();
    CUDAcommon::cudatime.TcomputeEiter += elapsed_run.count();
#endif
#endif
#ifdef SERIAL_CUDACROSSCHECK
    CUDAcommon::handleerror(hipDeviceSynchronize());
    floatingpoint cuda_lambda[1];
    CUDAcommon::handleerror(hipMemcpy(cuda_lambda, CUDAcommon::cudavars.gpu_lambda,  sizeof(floatingpoint),
                                       hipMemcpyDeviceToHost));

    std::cout<<"Lambda used CUDA "<<cuda_lambda[0]<<" SERL "<<d<<endl;
#endif
    for (auto &ff : _forceFields) {

        auto tempEnergy = ff->computeEnergy(coord, f, d);
#ifdef ALLSYNC
        hipDeviceSynchronize();
#endif
//        std::cout<<ff->getName()<<" "<<tempEnergy<<endl;
        if (verbose) cout << ff->getName() << " energy = " << tempEnergy << endl;
        //if energy is infinity, exit with infinity.
        if (tempEnergy <= -1) {

            //if this is the current energy, exit ungracefully
            if (d == 0.0) {

                cout << "Energy = " << tempEnergy << endl;

                cout
                        << "Energy of system became infinite. Try adjusting minimization parameters."
                        << endl;
                cout << "The culprit was ... " << ff->getName() << endl;

                //get the culprit in output
                ff->whoIsCulprit();

                exit(EXIT_FAILURE);
            }
                //if this is a minimization try, just return infinity
            else return numeric_limits<floatingpoint>::infinity();
        } else energy += tempEnergy;
#ifdef SERIAL_CUDACROSSCHECK
        hipDeviceSynchronize();
        resetfloatingpointvariableCUDA<<<1,1,0, streamF>>>(gpu_Uvec);
        addvectorred3<<<bntaddvec2.at(2),bntaddvec2.at(3), bntaddvec2.at(3) * sizeof(floatingpoint)
                                                                   , streamF>>>
        (CUDAcommon::cudavars.gpu_energyvec, gpu_params,
                gpu_Uvec);
        floatingpoint cuda_energyvec[1];
        CUDAcommon::handleerror(hipMemcpy(cuda_energyvec, gpu_Uvec, sizeof(floatingpoint),
                                           hipMemcpyDeviceToHost));
        std::cout<<ff->getName()<<" Energy. CUDA "<<cuda_energyvec[0]<<" SERL "
                ""<<energy<<endl;
#endif
    }
#ifdef CUDATIMETRACK
    tbegin = chrono::high_resolution_clock::now();
#endif
    //Add energies
#ifdef CUDAACCL
//    std::cout<<"Total nint "<<bntaddvec2.at(0)<<" "<<CUDAcommon::cudavars.offset_E<<endl;
    //Synchronize streams
    for(auto strm:CUDAcommon::getCUDAvars().streamvec) {
            CUDAcommon::handleerror(hipStreamSynchronize(*strm), "computeEnergy",
                                    "ForceFieldManager.cu");
        }
#ifdef CUDATIMETRACK
    tend= chrono::high_resolution_clock::now();
    chrono::duration<floatingpoint> elapsed_run2(tend - tbegin);
    CUDAcommon::cudatime.TveccomputeE.push_back(elapsed_run2.count());
    CUDAcommon::cudatime.TcomputeE += elapsed_run2.count();
    CUDAcommon::cudatime.TcomputeEiter += elapsed_run2.count();
    tbegin = chrono::high_resolution_clock::now();
#endif
//    std::cout<<"CUDA energy total nint "<<CUDAcommon::cudavars.offset_E<<endl;
    /*vector<floatingpoint> ones;
    for(int i = 0;i<8192;i++)
        ones.push_back(1.0);
    CUDAcommon::handleerror(hipMemcpyAsync(CUDAcommon::cudavars.gpu_energyvec, ones
                                                        .data() ,
                                                bntaddvec2.at(0) * sizeof
                                                        (floatingpoint),
                                                hipMemcpyHostToDevice,streamF));*/
/*    CUDAcommon::handleerror(hipMemsetAsync(CUDAcommon::cudavars.gpu_energyvec, 1,
                                            bntaddvec2.at(0) * sizeof
            (floatingpoint),streamF));
    hipDeviceSynchronize();*/
    resetfloatingpointvariableCUDA<<<1,1,0, streamF>>>(gpu_Uvec);
    addvectorred3<<<bntaddvec2.at(2),bntaddvec2.at(3), bntaddvec2.at(3) * sizeof(floatingpoint)
                    , streamF>>>(CUDAcommon::cudavars.gpu_energyvec, gpu_params,
                        gpu_Uvec);
    CUDAcommon::handleerror(hipStreamSynchronize(streamF));
#ifdef DETAILEDOUTPUT_ENERGY
    hipDeviceSynchronize();
    floatingpoint cuda_energyvec[1];
    CUDAcommon::handleerror(hipMemcpy(cuda_energyvec, gpu_Uvec, sizeof(floatingpoint),
                                       hipMemcpyDeviceToHost));
    std::cout<<"vector energy addition CUDA "<<cuda_energyvec[0]<<" SERL "<<energy<<endl;
#endif
//    CUDAcommon::handleerror(hipFree(CUDAcommon::cudavars.gpu_energyvec));
#endif
#ifdef ALLSYNC
    hipDeviceSynchronize();
#endif

#ifdef CUDATIMETRACK
    tend= chrono::high_resolution_clock::now();
    chrono::duration<floatingpoint> elapsed_run3(tend - tbegin);
    CUDAcommon::cudatime.TveccomputeE.push_back(elapsed_run3.count());
    CUDAcommon::cudatime.TcomputeE += elapsed_run3.count();
    CUDAcommon::cudatime.TcomputeEiter += elapsed_run3.count();
//    std::cout<<"Time total computeEnergy (s) CUDA "<<CUDAcommon::cudatime
//            .TcomputeE<<" SERL "<<CUDAcommon::serltime.TcomputeE<<" factor "
//                     ""<<CUDAcommon::serltime.TcomputeE/CUDAcommon::cudatime.TcomputeE<<endl;
//    std::cout<<"Time split computeEnergy (s) CUDA ";
//    for(auto x:CUDAcommon::cudatime.TveccomputeE)
//        std::cout<<x<<" ";
//    std::cout<<endl;
//    std::cout<<"Time split computeEnergy (s) SERL ";
//    for(auto x:CUDAcommon::serltime.TveccomputeE)
//        std::cout<<x<<" ";
//    std::cout<<endl;
#endif
    return energy;
}

void ForceFieldManager::computeForces(floatingpoint *coord, floatingpoint *f) {
    //reset to zero
#ifdef CUDATIMETRACK
    chrono::high_resolution_clock::time_point tbegin, tend;
    CUDAcommon::cudatime.TcomputeF = 0.0;
    CUDAcommon::cudatime.TveccomputeF.clear();
    CUDAcommon::serltime.TcomputeF = 0.0;
    CUDAcommon::serltime.TveccomputeF.clear();
    tbegin = chrono::high_resolution_clock::now();
#endif
    //@{
    for (int i = 0; i < CGMethod::N; i++)
        f[i] = 0.0;
    //@}
#ifdef CUDATIMETRACK
    tend= chrono::high_resolution_clock::now();
    chrono::duration<floatingpoint> elapsed_run(tend - tbegin);
    CUDAcommon::serltime.TveccomputeF.push_back(elapsed_run.count());
    CUDAcommon::serltime.TcomputeF += elapsed_run.count();
    tbegin = chrono::high_resolution_clock::now();
#endif
#ifdef CUDAACCL
    CUDAvars cvars = CUDAcommon::getCUDAvars();
    if (cross_checkclass::Aux)
        resetForcesCUDA << < blocksnthreads[0], blocksnthreads[1], 0, streamF >> >
                                                                      (cvars.gpu_forceAux, gpu_nint);
    else
        resetForcesCUDA << < blocksnthreads[0], blocksnthreads[1], 0, streamF >> >
                                                                      (cvars.gpu_force, gpu_nint);
    CUDAcommon::handleerror(hipStreamSynchronize(streamF));

    CUDAcommon::handleerror(hipGetLastError(), "resetForcesCUDA", "ForceFieldManager.cu");
#endif
#ifdef CUDATIMETRACK
    tend= chrono::high_resolution_clock::now();
    chrono::duration<floatingpoint> elapsed_run2(tend - tbegin);
    CUDAcommon::cudatime.TveccomputeF.push_back(elapsed_run2.count());
    CUDAcommon::cudatime.TcomputeF += elapsed_run2.count();
    tbegin = chrono::high_resolution_clock::now();
#endif
    //recompute
//    floatingpoint *F_i = new floatingpoint[CGMethod::N];
    for (auto &ff : _forceFields) {
        ff->computeForces(coord, f);
#ifdef ALLSYNC
        hipDeviceSynchronize();
#endif

//        if(cross_checkclass::Aux)
//            CUDAcommon::handleerror(
//                hipMemcpy(F_i, CUDAcommon::getCUDAvars().gpu_forceAux, 3 * Bead::getBeads().size() * sizeof
//                                   (floatingpoint),
//                           hipMemcpyDeviceToHost));
//        else
//            CUDAcommon::handleerror(
//                    hipMemcpy(F_i, CUDAcommon::getCUDAvars().gpu_force, 3 * Bead::getBeads().size() * sizeof
//                                       (floatingpoint),
//                               hipMemcpyDeviceToHost));
//        floatingpoint fmax = 0.0;
//        int id=0;
//        for (auto iter = 0; iter < Bead::getBeads().size(); iter++) {
//            if(abs(F_i[3 *iter])> fmax) {fmax = abs(F_i[3*iter]);id = iter;}
//            if(abs(F_i[3 *iter +1])> fmax) {fmax = abs(F_i[3*iter +1]);id = iter;}
//            if(abs(F_i[3 *iter +2])> fmax) {fmax = abs(F_i[3*iter +2]);id = iter;}
////            std::cout << F_i[3 * iter] << " " << F_i[3 * iter + 1] << " " << F_i[3 * iter + 2] << endl;
//        }
//        std::cout <<"Fmax "<< id<<" "<<fmax<<" "<<F_i[3 * id] << " " << F_i[3 * id + 1] << " " << F_i[3 * id + 2] <<
//                                                                                                                 endl;
    }
//    delete F_i;
}

void ForceFieldManager::computeLoadForces() {

    //reset
    for (auto b: Bead::getBeads()) {
        std::fill(b->loadForcesM.begin(), b->loadForcesM.end(), 0.0);
        std::fill(b->loadForcesP.begin(), b->loadForcesP.end(), 0.0);
//        b->loadForcesP.clear();
//        b->loadForcesM.clear();
    }

    for (auto &f : _forceFields)
        f->computeLoadForces();

    //reset lfi as well
    for (auto b: Bead::getBeads()) {
        b->lfip = 0;
        b->lfim = 0;
    }
}

void ForceFieldManager::copyForces(floatingpoint *fprev, floatingpoint *f) {

    for (int i = 0; i < CGMethod::N; i++)
        fprev[i] = f[i];
}

#ifdef CUDAACCL

void ForceFieldManager::CUDAcopyForces(hipStream_t stream, floatingpoint *fprev, floatingpoint *f) {


//    CUDAcommon::handleerror(hipFree(CUDAcommon::getCUDAvars().gpu_forceAux));
//    floatingpoint* gpu_forceAux;
//    CUDAcommon::handleerror(hipMalloc((void **) &gpu_forceAux, CGMethod::N * sizeof(floatingpoint)));
//    CUDAvars cvars=CUDAcommon::getCUDAvars();
//    cvars.gpu_forceAux=gpu_forceAux;
//    CUDAcommon::cudavars=cvars;

//    std::cout<<"Copyforces Number of Blocks: "<<blocksnthreads[0]<<endl;
//    std::cout<<"Threads per block: "<<blocksnthreads[1]<<endl;
    copyForcesCUDA << < blocksnthreads[0], blocksnthreads[1], 0, stream >> >
                                                                 (f, fprev, gpu_nint);
    CUDAcommon::handleerror(hipGetLastError(), "copyForcesCUDA", "ForceFieldManager.cu");
}

void ForceFieldManager::assignallforcemags() {

    for (auto &ff : _forceFields)
        ff->assignforcemags();
}

#endif