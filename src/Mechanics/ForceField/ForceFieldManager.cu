
//------------------------------------------------------------------
//  **MEDYAN** - Simulation Package for the Mechanochemical
//               Dynamics of Active Networks, v3.1
//
//  Copyright (2015-2016)  Papoian Lab, University of Maryland
//
//                 ALL RIGHTS RESERVED
//
//  See the MEDYAN web page for more information:
//  http://www.medyan.org
//------------------------------------------------------------------

#include "ForceFieldManager.h"
#include "ForceFieldManagerCUDA.h"

#include "CGMethod.h"
#include "cross_check.h"

void ForceFieldManager::vectorizeAllForceFields() {

    for(auto &ff : _forceFields)
        ff->vectorize();
#ifdef CUDAACCL
    int nint[1]; nint[0]=CGMethod::N/3;
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_nint, sizeof(int)));
    CUDAcommon::handleerror(hipMemcpy(gpu_nint, nint, sizeof(int), hipMemcpyHostToDevice));
    blocksnthreads.push_back(CGMethod::N/(3*THREADSPERBLOCK) + 1);
    if(blocksnthreads[0]==1) blocksnthreads.push_back(CGMethod::N/3);
    else blocksnthreads.push_back(THREADSPERBLOCK);
#endif
}

void ForceFieldManager::cleanupAllForceFields() {

    for(auto &ff : _forceFields)
        ff->cleanup();
#ifdef CUDAACCL
    CUDAcommon::handleerror(hipFree(gpu_nint));
    blocksnthreads.clear();
#endif
}

double ForceFieldManager::computeEnergy(double *coord, double *f, double d, bool verbose) {

    double energy = 0;
#ifdef CUDAACCL
    auto gU_tot = CUDAcommon::getCUDAvars().gpu_energy;
//    hipStream_t  stream;
//    CUDAcommon::handleerror(hipStreamCreate(&stream));

    setenergytozero<<<1,1>>>(gU_tot);
//    CUDAcommon::handleerror(hipStreamSynchronize(stream));
//
//    CUDAcommon::handleerror(hipStreamDestroy(stream));
//    CUDAcommon::handleerror(hipGetLastError());
#endif
    for(auto &ff : _forceFields) {
        auto tempEnergy = ff->computeEnergy(coord, f, d);


        if(verbose) cout << ff->getName() << " energy = " << tempEnergy << endl;

        //if energy is infinity, exit with infinity.
        if(tempEnergy <= -1) {

            //if this is the current energy, exit ungracefully
            if(d == 0.0) {

                cout << "Energy = " << tempEnergy << endl;

                cout << "Energy of system became infinite. Try adjusting minimization parameters." << endl;
                cout << "The culprit was ... " << ff->getName() << endl;

                //get the culprit in output
                ff->whoIsCulprit();

                exit(EXIT_FAILURE);
            }
                //if this is a minimization try, just return infinity
            else return numeric_limits<double>::infinity();
        }
        else energy += tempEnergy;

    }

//    double U[1];
//    CUDAcommon::handleerror(hipMemcpy(U, gU_tot, sizeof(double),
//                                           hipMemcpyDeviceToHost));
//    std::cout<<U[0]<<" "<<energy<<endl;
    return energy;
}

#ifdef CROSSCHECK
void ForceFieldManager::resetForces() {

    for(auto b: Bead::getBeads()) {
        if(cross_checkclass::Aux)
        b->forceAux.assign (3,0);
        else
        b->force.assign (3, 0); //Set force to zero;
        std::memset((void*)(&b->loadForcesP[0]), 0, sizeof(b->loadForcesP));  //Set load force to zero;
        std::memset((void*)(&b->loadForcesM[0]), 0, sizeof(b->loadForcesM));  //Set load force to zero;
    }
}
#endif
void ForceFieldManager::computeForces(double *coord, double *f) {
#ifdef CROSSCHECK
    resetForces();
#endif
    //TODO change so that you don't have to copy a vector every time during minimization.
    //reset to zero
    for (int i = 0; i < CGMethod::N; i++)
        f[i] = 0.0;

#ifdef CUDAACCL
//    ForceField* arrayff;
//    ForceField* gpu_arrayff;
//    CUDAcommon::handleerror(
//            hipHostAlloc((void**) &arrayff, _forceFields.size()*sizeof(ForceField), hipHostMallocMapped));
//    for(auto i =0;i<_forceFields.size();i++)
//        arrayff[i] = *_forceFields.at(i);
//    CUDAcommon::handleerror(hipHostGetDevicePointer(&gpu_arrayff, arrayff, 0));
//    testfunction<<<1,1>>>(gpu_arrayff);

    CUDAvars cvars=CUDAcommon::getCUDAvars();
//    std::cout<<"Blocks "<<blocksnthreads[0]<<" Threads "<<blocksnthreads[1]<<endl;
    hipStream_t  stream;
    CUDAcommon::handleerror(hipStreamCreate( &stream));
    if(cross_checkclass::Aux)
        resetForcesCUDA<<<blocksnthreads[0],blocksnthreads[1],0,stream>>>(cvars.gpu_forceAux, gpu_nint);
    else
        resetForcesCUDA<<<blocksnthreads[0],blocksnthreads[1],0,stream>>>(cvars.gpu_force, gpu_nint);
    CUDAcommon::handleerror(hipStreamSynchronize(stream));
    CUDAcommon::handleerror(hipStreamDestroy(stream));

    CUDAcommon::handleerror( hipGetLastError() );
    //TODO can be removed as you have rewritten the code to prevent hipFree everytime force is calculated.
//    double* gpu_force;
//    CUDAcommon::handleerror(hipMalloc((void **) &gpu_force, CGMethod::N * sizeof(double)));
//    CUDAcommon::handleerror(hipMemcpy(gpu_force, f, CGMethod::N * sizeof(double), hipMemcpyHostToDevice));
//    CUDAvars cvars=CUDAcommon::getCUDAvars();
//    if(cross_checkclass::Aux) {
//        if(cvars.gpu_forceAux != NULL )
//        CUDAcommon::handleerror(hipFree(cvars.gpu_forceAux));
//        cvars.gpu_forceAux = gpu_force;
//    }
//    else{
//        if(cvars.gpu_forceAux != NULL )
//        CUDAcommon::handleerror(hipFree(cvars.gpu_force));
//        cvars.gpu_force = gpu_force;
//    }
//    CUDAcommon::cudavars=cvars;


//    double F_i[CGMethod::N];
//    hipMemcpy(F_i, gpu_force, CGMethod::N * sizeof(double), hipMemcpyDeviceToHost);
//    for(auto i=0;i<CGMethod::N;i++)
//        std::cout<<F_i[i]<<" ";
//    std::cout<<endl;
#endif
    //recompute
    for(auto &ff : _forceFields)
        ff->computeForces(coord, f);

//#ifdef CUDAACCL
//    //TODO Remove later
//    double* gpu_force;
//    hipMalloc((void **) &gpu_force, CGMethod::N * sizeof(double));
//    hipMemcpy(gpu_force, f, CGMethod::N * sizeof(double), hipMemcpyHostToDevice);
//    CUDAvars cvars=CUDAcommon::getCUDAvars();
//    cvars.gpu_force=gpu_force;
//    CUDAcommon::cudavars=cvars;
//#endif
    //WILL HAVE TO COPY AUXS AFTER THIS CALL
}

void ForceFieldManager::computeLoadForces() {

    //reset
    for (auto b: Bead::getBeads()) {

        b->loadForcesP.clear();
        b->loadForcesM.clear();
    }

    for(auto &f : _forceFields)
        f->computeLoadForces();

    //reset lfi as well
    for(auto b: Bead::getBeads()) {
        b->lfip = 0;
        b->lfim = 0;
    }
}

void ForceFieldManager::copyForces(double *fprev, double *f) {

    for (int i = 0; i < CGMethod::N; i++)
        fprev[i] = f[i];
}

void ForceFieldManager::CUDAcopyForces(hipStream_t stream, double *fprev, double *f) {
#ifdef CUDAACCL
    //TODO Change so that the pointers to forceAux and force are exchanged and pointer to force is flushed.

//    CUDAcommon::handleerror(hipFree(CUDAcommon::getCUDAvars().gpu_forceAux));
//    double* gpu_forceAux;
//    CUDAcommon::handleerror(hipMalloc((void **) &gpu_forceAux, CGMethod::N * sizeof(double)));
//    CUDAvars cvars=CUDAcommon::getCUDAvars();
//    cvars.gpu_forceAux=gpu_forceAux;
//    CUDAcommon::cudavars=cvars;

//    std::cout<<"Copyforces Number of Blocks: "<<blocksnthreads[0]<<endl;
//    std::cout<<"Threads per block: "<<blocksnthreads[1]<<endl;
    copyForcesCUDA<<<blocksnthreads[0],blocksnthreads[1],0,stream>>>(f, fprev, gpu_nint);
    CUDAcommon::handleerror( hipGetLastError() );
#endif
}
