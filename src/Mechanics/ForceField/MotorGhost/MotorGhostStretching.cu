#include "hip/hip_runtime.h"

//------------------------------------------------------------------
//  **MEDYAN** - Simulation Package for the Mechanochemical
//               Dynamics of Active Networks, v3.1
//
//  Copyright (2015-2016)  Papoian Lab, University of Maryland
//
//                 ALL RIGHTS RESERVED
//
//  See the MEDYAN web page for more information:
//  http://www.medyan.org
//------------------------------------------------------------------

#include "MotorGhostStretching.h"

#include "MotorGhostStretchingHarmonic.h"

#include "MotorGhost.h"
#include "Cylinder.h"
#include "Bead.h"
#include "cross_check.h"
#include "CGMethod.h"
#ifdef CUDAACCL
#include "nvToolsExt.h"
#endif

template <class MStretchingInteractionType>
void MotorGhostStretching<MStretchingInteractionType>::assignforcemags() {
#ifdef CUDAACCL
    double stretchforce[MotorGhost::getMotorGhosts().size()];
    CUDAcommon::handleerror(hipMemcpy(stretchforce, gpu_Mstretchforce,
                                       MotorGhost::getMotorGhosts().size() * sizeof(double),
                                       hipMemcpyDeviceToHost));
    int id = 0;
    for(auto m:MotorGhost::getMotorGhosts())
    {m->getMMotorGhost()->stretchForce = stretchforce[id];id++;}
#endif
}

template <class MStretchingInteractionType>
void MotorGhostStretching<MStretchingInteractionType>::vectorize() {

    beadSet = new int[n * MotorGhost::getMotorGhosts().size()];
    kstr = new double[MotorGhost::getMotorGhosts().size()];
    eql = new double[MotorGhost::getMotorGhosts().size()];
    pos1 = new double[MotorGhost::getMotorGhosts().size()];
    pos2 = new double[MotorGhost::getMotorGhosts().size()];
    stretchforce = new double[MotorGhost::getMotorGhosts().size()];

    int i = 0;
    
    for (auto m: MotorGhost::getMotorGhosts()) {
        m->_dbIndex = i;
        beadSet[n * i] = m->getFirstCylinder()->getFirstBead()->_dbIndex;
        beadSet[n * i + 1] = m->getFirstCylinder()->getSecondBead()->_dbIndex;
        beadSet[n * i + 2] = m->getSecondCylinder()->getFirstBead()->_dbIndex;
        beadSet[n * i + 3] = m->getSecondCylinder()->getSecondBead()->_dbIndex;

        kstr[i] = m->getMMotorGhost()->getStretchingConstant();
        eql[i] = m->getMMotorGhost()->getEqLength();
        pos1[i] = m->getFirstPosition();
        pos2[i] = m->getSecondPosition();
        stretchforce[i] = 0.0;

        i++;
    }

    //CUDA
#ifdef CUDAACCL
#ifdef CUDATIMETRACK
    chrono::high_resolution_clock::time_point tbegin, tend;
    tbegin = chrono::high_resolution_clock::now();
#endif
    //CUDA stream create
    if(stream == NULL || !(CUDAcommon::getCUDAvars().conservestreams))
        CUDAcommon::handleerror(hipStreamCreate(&stream));
//    F_i = new double[3 * Bead::getBeads().size()];
//    hipEvent_t start, stop;
//    CUDAcommon::handleerror(hipEventCreate( &start));
//    CUDAcommon::handleerror(hipEventCreate( &stop));
//    CUDAcommon::handleerror(hipEventRecord( start, 0));

    int numInteractions = MotorGhost::getMotorGhosts().size();
    _FFType.optimalblocksnthreads(numInteractions, stream);
//    blocksnthreads.clear();
//    blocksnthreads.push_back(numInteractions/THREADSPERBLOCK + 1);
//
//    if(blocksnthreads[0]==1) blocksnthreads.push_back( numInteractions);
////    if(blocksnthreads[0]==1) blocksnthreads.push_back( 32*(int(numInteractions/32 +1)) );
//    else blocksnthreads.push_back(THREADSPERBLOCK);
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_beadSet, n * numInteractions * sizeof(int)));
    CUDAcommon::handleerror(hipMemcpyAsync(gpu_beadSet, beadSet, n * numInteractions *
                                                sizeof(int),
                                       hipMemcpyHostToDevice, stream));

    CUDAcommon::handleerror(hipMalloc((void **) &gpu_kstr, numInteractions * sizeof(double)));
    CUDAcommon::handleerror(hipMemcpyAsync(gpu_kstr, kstr, numInteractions * sizeof
                            (double), hipMemcpyHostToDevice, stream));

    CUDAcommon::handleerror(hipMalloc((void **) &gpu_eql, numInteractions * sizeof(double)));
    CUDAcommon::handleerror(hipMemcpyAsync(gpu_eql, eql, numInteractions * sizeof(double),
                                        hipMemcpyHostToDevice, stream));

    CUDAcommon::handleerror(hipMalloc((void **) &gpu_pos1, numInteractions * sizeof(double)));
    CUDAcommon::handleerror(hipMemcpyAsync(gpu_pos1, pos1, numInteractions * sizeof
                            (double), hipMemcpyHostToDevice, stream));

//    double checkpos1[numInteractions];
//    hipMemcpy(checkpos1, gpu_pos1, numInteractions * sizeof(double), hipMemcpyDeviceToHost);
//    for(auto i=0;i<numInteractions;i++) std::cout<<pos1[i]<<" "<<checkpos1[i]<<endl;

    CUDAcommon::handleerror(hipMalloc((void **) &gpu_pos2, numInteractions * sizeof(double)));
    CUDAcommon::handleerror(hipMemcpyAsync(gpu_pos2, pos2, numInteractions * sizeof
                           (double), hipMemcpyHostToDevice, stream));
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_Mstretchforce, numInteractions *
                                                                     sizeof(double)),"cuda data transfer",
                            "MotorGhostStretching.cu");

    vector<int> params;
    params.push_back(int(n));
    params.push_back(numInteractions);
    params.push_back(CUDAcommon::cudavars.offset_E);
    //set offset
    CUDAcommon::cudavars.offset_E += numInteractions;
//    std::cout<<"offset "<<getName()<<" "<<CUDAcommon::cudavars.offset_E<<endl;

    CUDAcommon::handleerror(hipMalloc((void **) &gpu_params, 3 * sizeof(int)));
    CUDAcommon::handleerror(hipMemcpyAsync(gpu_params, params.data(), 3 * sizeof(int),
                                       hipMemcpyHostToDevice, stream));
//    CUDAcommon::cudavars.motorparams = gpu_params;
#ifdef CUDATIMETRACK
//    CUDAcommon::handleerror(hipDeviceSynchronize(),"MotorGhostStretching.cu",
//                            "vectorizeFF");
    tend= chrono::high_resolution_clock::now();
    chrono::duration<double> elapsed_run(tend - tbegin);
    CUDAcommon::cudatime.TvecvectorizeFF.push_back(elapsed_run.count());
    CUDAcommon::cudatime.TvectorizeFF += elapsed_run.count();
#endif
#endif

    //
}

template<class MStretchingInteractionType>
void MotorGhostStretching<MStretchingInteractionType>::deallocate() {
    for(auto m: MotorGhost::getMotorGhosts()){
        //Using += to ensure that the stretching forces are additive.
        m->getMMotorGhost()->stretchForce += stretchforce[m->_dbIndex];
//        std::cout<<m->getMMotorGhost()->stretchForce<<endl;
    }
    delete [] stretchforce;
    delete [] beadSet;
    delete [] kstr;
    delete [] eql;
    delete [] pos1;
    delete [] pos2;
#ifdef CUDAACCL
    if(!(CUDAcommon::getCUDAvars().conservestreams))
        CUDAcommon::handleerror(hipStreamDestroy(stream));
    _FFType.deallocate();
    CUDAcommon::handleerror(hipFree(gpu_beadSet));
    CUDAcommon::handleerror(hipFree(gpu_kstr));
    CUDAcommon::handleerror(hipFree(gpu_pos1));
    CUDAcommon::handleerror(hipFree(gpu_pos2));
    CUDAcommon::handleerror(hipFree(gpu_eql));
    CUDAcommon::handleerror(hipFree(gpu_params));
    CUDAcommon::handleerror(hipFree(gpu_Mstretchforce));
#endif
}


template <class MStretchingInteractionType>
double MotorGhostStretching<MStretchingInteractionType>::computeEnergy(double* coord, double *f, double d){
    double U_i[1], U_ii=0.0;
    double* gU_i;
    U_ii = 0.0;
#ifdef CUDATIMETRACK
    chrono::high_resolution_clock::time_point tbegin, tend;
#endif
#ifdef CUDAACCL
//    std::cout<<"Motor size "<<MotorGhost::getMotorGhosts().size()<<endl;
#ifdef CUDATIMETRACK
    tbegin = chrono::high_resolution_clock::now();
#endif

    //has to be changed to accomodate aux force
    double * gpu_coord=CUDAcommon::getCUDAvars().gpu_coord;
    double * gpu_force=CUDAcommon::getCUDAvars().gpu_force;
    double * gpu_d = CUDAcommon::getCUDAvars().gpu_lambda;


//    if(d == 0.0){
//        gU_i=_FFType.energy(gpu_coord, gpu_force, gpu_beadSet, gpu_kstr, gpu_eql, gpu_pos1, gpu_pos2, gpu_params);
//
//    }
//    else{
        gU_i=_FFType.energy(gpu_coord, gpu_force, gpu_beadSet, gpu_kstr, gpu_eql, gpu_pos1, gpu_pos2, gpu_d,
                            gpu_params);
//    }


#ifdef CUDATIMETRACK
//    CUDAcommon::handleerror(hipDeviceSynchronize(),"MotorGhostStretching.cu", "computeEnergy");
    tend= chrono::high_resolution_clock::now();
    chrono::duration<double> elapsed_run(tend - tbegin);
    CUDAcommon::cudatime.TveccomputeE.push_back(elapsed_run.count());
    CUDAcommon::cudatime.TcomputeE += elapsed_run.count();
    CUDAcommon::cudatime.TcomputeEiter += elapsed_run.count();
#endif

#endif
#ifdef SERIAL
#ifdef CUDATIMETRACK
    tbegin = chrono::high_resolution_clock::now();
#endif

    if (d == 0.0)
        U_ii = _FFType.energy(coord, f, beadSet, kstr, eql, pos1, pos2);
    else
        U_ii = _FFType.energy(coord, f, beadSet, kstr, eql, pos1, pos2, d);

#ifdef CUDATIMETRACK
    tend= chrono::high_resolution_clock::now();
    chrono::duration<double> elapsed_runs(tend - tbegin);
    CUDAcommon::serltime.TveccomputeE.push_back(elapsed_runs.count());
    CUDAcommon::serltime.TcomputeE += elapsed_runs.count();
    CUDAcommon::serltime.TcomputeEiter += elapsed_runs.count();
#endif
#endif

    return U_ii;
}

template <class MStretchingInteractionType>
void MotorGhostStretching<MStretchingInteractionType>::computeForces(double *coord, double *f) {
#ifdef CUDATIMETRACK
    chrono::high_resolution_clock::time_point tbegin, tend;
    tbegin = chrono::high_resolution_clock::now();
#endif
#ifdef CUDAACCL
    //has to be changed to accomodate aux force
    double * gpu_coord=CUDAcommon::getCUDAvars().gpu_coord;

    double * gpu_force;
    if(cross_checkclass::Aux){
        gpu_force=CUDAcommon::getCUDAvars().gpu_forceAux;
        _FFType.forces(gpu_coord, gpu_force, gpu_beadSet, gpu_kstr, gpu_eql, gpu_pos1,
                       gpu_pos2, gpu_params, gpu_Mstretchforce);
    }
    else {
        gpu_force = CUDAcommon::getCUDAvars().gpu_force;
        _FFType.forces(gpu_coord, gpu_force, gpu_beadSet, gpu_kstr, gpu_eql, gpu_pos1,
                       gpu_pos2, gpu_params, gpu_Mstretchforce);
    }
#endif
#ifdef CUDATIMETRACK
    tend= chrono::high_resolution_clock::now();
    chrono::duration<double> elapsed_run(tend - tbegin);
    CUDAcommon::cudatime.TveccomputeF.push_back(elapsed_run.count());
    CUDAcommon::cudatime.TcomputeF += elapsed_run.count();
    tbegin = chrono::high_resolution_clock::now();
#endif
#ifdef SERIAL
    _FFType.forces(coord, f, beadSet, kstr, eql, pos1, pos2, stretchforce);
#endif
#ifdef CUDATIMETRACK
    tend= chrono::high_resolution_clock::now();
    chrono::duration<double> elapsed_runs(tend - tbegin);
    CUDAcommon::serltime.TveccomputeF.push_back(elapsed_runs.count());
    CUDAcommon::serltime.TcomputeF += elapsed_runs.count();
#endif
#ifdef DETAILEDOUTPUT
    double maxF = 0.0;
    double mag = 0.0;
    for(int i = 0; i < CGMethod::N/3; i++) {
        mag = 0.0;
        for(int j = 0; j < 3; j++)
            mag += f[3 * i + j]*f[3 * i + j];
        mag = sqrt(mag);
//        std::cout<<"SL "<<i<<" "<<mag*mag<<" "<<forceAux[3 * i]<<" "<<forceAux[3 * i + 1]<<" "<<forceAux[3 * i +
//                2]<<endl;
        if(mag > maxF) maxF = mag;
    }
    std::cout<<"max "<<getName()<<" "<<maxF<<endl;
#endif
}


///Temlate specializations
template double MotorGhostStretching<MotorGhostStretchingHarmonic>::computeEnergy(double *coord, double *f, double d);
template void MotorGhostStretching<MotorGhostStretchingHarmonic>::computeForces(double *coord, double *f);
template void MotorGhostStretching<MotorGhostStretchingHarmonic>::vectorize();
template void MotorGhostStretching<MotorGhostStretchingHarmonic>::deallocate();
template void MotorGhostStretching<MotorGhostStretchingHarmonic>::assignforcemags();


