#include "hip/hip_runtime.h"

//------------------------------------------------------------------
//  **MEDYAN** - Simulation Package for the Mechanochemical
//               Dynamics of Active Networks, v3.1
//
//  Copyright (2015-2016)  Papoian Lab, University of Maryland
//
//                 ALL RIGHTS RESERVED
//
//  See the MEDYAN web page for more information:
//  http://www.medyan.org
//------------------------------------------------------------------

#include "MotorGhostStretchingHarmonic.h"
#include "MotorGhostStretching.h"
#include "MotorGhost.h"
#include "MotorGhostStretchingHarmonicCUDA.h"
#include "Bead.h"
#include "MathFunctions.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "nvToolsExt.h"

using namespace mathfunc;
#ifdef CUDAACCL
void MotorGhostStretchingHarmonic::deallocate(){
    CUDAcommon::handleerror(hipStreamDestroy(stream));
    CUDAcommon::handleerror(hipFree(gU_i));
    CUDAcommon::handleerror(hipFree(gU_sum));
    CUDAcommon::handleerror(hipFree(gFF));
    CUDAcommon::handleerror(hipFree(ginteraction));
}
void MotorGhostStretchingHarmonic::checkforculprit() {
    CUDAcommon::printculprit("MotorGhostStretching","MotorGhostStretchingHarmonic");
    MotorGhost* m;
    m = MotorGhost::getMotorGhosts()[CUDAcommon::getCUDAvars().culpritID[0]];
    cout<<"Printing culprit Filament information."<<endl;
    m->printSelf();
    exit(EXIT_FAILURE);
}
void MotorGhostStretchingHarmonic::optimalblocksnthreads( int nint){
    //CUDA stream create
    CUDAcommon::handleerror(hipStreamCreate(&stream));
    blocksnthreadse.clear();
    blocksnthreadsez.clear();
    blocksnthreadsf.clear();
    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the
    // maximum occupancy for a full device launch
//    int gridSize;    // The actual grid size needed, based on input size
//    unaryfn::argument_type blksize;
//    unaryfn::result_type result;
//    unaryfn ufn;
    if(nint>0) {
        hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize,
                                                       MotorGhostStretchingHarmonicenergy, blockToSmem, 0);
//    std::cout<<(nint +blockSize -1) / blockSize<<" "<<blockSize<<endl;
//
//    hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize,
//                                        CUDAExclVolRepulsionenergy, 0, 0);
        blocksnthreadse.push_back((nint + blockSize - 1) / blockSize);
        blocksnthreadse.push_back(blockSize);
//    std::cout<<(nint +blockSize -1) / blockSize<<" "<<blockSize<<endl;
        blockSize = 0;

        hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize,
                                                       MotorGhostStretchingHarmonicenergyz, blockToSmemez, 0);
        blocksnthreadsez.push_back((nint + blockSize - 1) / blockSize);
        blocksnthreadsez.push_back(blockSize);
        blockSize = 0;

        hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize,
                                                       MotorGhostStretchingHarmonicforces, blockToSmem, 0);
        blocksnthreadsf.push_back((nint + blockSize - 1) / blockSize);
        blocksnthreadsf.push_back(blockSize);

        CUDAcommon::handleerror(hipMalloc((void **) &gU_i, nint*sizeof(double)));
	    CUDAcommon::handleerror(hipMalloc((void **) &gU_sum, sizeof(double)));
        char a[] = "MotorGhostFF";
        char b[] = "MotorGhost Stretching Harmonic";
        CUDAcommon::handleerror(hipMalloc((void **) &gFF, 100 * sizeof(char)));
        CUDAcommon::handleerror(hipMalloc((void **) &ginteraction, 100 * sizeof(char)));
        CUDAcommon::handleerror(hipMemcpy(gFF, a, 100 * sizeof(char), hipMemcpyHostToDevice));
        CUDAcommon::handleerror(hipMemcpy(ginteraction, b, 100 * sizeof(char), hipMemcpyHostToDevice));
    }
    else{
        blocksnthreadse.push_back(0);
        blocksnthreadse.push_back(0);
        blocksnthreadsez.push_back(0);
        blocksnthreadsez.push_back(0);
        blocksnthreadsf.push_back(0);
        blocksnthreadsf.push_back(0);
    }

}
#endif
double* MotorGhostStretchingHarmonic::energy(double *coord, double *f, int *beadSet,
                                            double *kstr, double *eql, double *pos1, double *pos2,
                                            int *params) {
    if(blocksnthreadse[1]>0) {
//        nvtxRangePushA("cmsestream");
//        hipStream_t  stream;
//        hipEvent_t event;
//        CUDAcommon::handleerror(hipStreamCreate(&stream));
//        CUDAcommon::handleerror(hipEventCreate(&event));
//        auto cvars = CUDAcommon::getCUDAvars();
//        cvars.eventvec.push_back(event);
//        cvars.streamvec.push_back(stream);
//        CUDAcommon::cudavars = cvars;
//        nvtxRangePop();

//    double *gU_ii;
    //double *gU_i;
//    double *gc1, *gc2, *gcheckU;
//    double U_ii[blocksnthreads[0] * blocksnthreads[1]];
//    double c1[3*blocksnthreads[0]*blocksnthreads[1]], c2[3*blocksnthreads[0]*blocksnthreads[1]];
    //double U_i[blocksnthreadse[0]*blocksnthreadse[1]];
//    double checkU[blocksnthreads[1]];

//    double ccoord[3*Bead::getBeads().size()];
//    hipMemcpy(ccoord, coord, 3*Bead::getBeads().size()*sizeof(double), hipMemcpyDeviceToHost);
//    double cforce[3*Bead::getBeads().size()];
//    hipMemcpy(cforce, f, 3*Bead::getBeads().size()*sizeof(double), hipMemcpyDeviceToHost);
//    for(auto i =0; i < Bead::getBeads().size(); i++)
//        std::cout<<ccoord[3 * i]<<" "<<ccoord[3 * i +1]<<" "<<ccoord[3 * i +2]<<" "<<cforce[3 * i]<<" "<<cforce[3 * i
//                                                                                                                +1]<<" "<<cforce[3 * i +2]<<endl;
//    std::cout<<"C+F---------------------------- "<<endl;

//    std::cout<<"MSE Number of Blocks: "<<blocksnthreads[0]<<endl;
//    std::cout<<"Threads per block: "<<blocksnthreads[1]<<endl;

    //TODO  since the number of threads needed is constant through out the minimization, consider storing the pointer.
    //`CUDAcommon::handleerror(hipMalloc((void **) &gU_i, blocksnthreadse[0]*blocksnthreadse[1]*sizeof(double)));
//    CUDAcommon::handleerror(hipMalloc((void **) &gU_ii, blocksnthreads[0]*blocksnthreads[1]*sizeof(double)));
//    CUDAcommon::handleerror(hipMalloc((void **) &gc1, 3*blocksnthreads[1]*sizeof(double)));
//    CUDAcommon::handleerror(hipMalloc((void **) &gc2, 3*blocksnthreads[1]*sizeof(double)));
//    CUDAcommon::handleerror(hipMalloc((void **) &gcheckU, blocksnthreads[1]*sizeof(double)));
    //
//    std::cout<<"MSE CUDA"<<endl;
        nvtxRangePushA("cmse");

    MotorGhostStretchingHarmonicenergy<<<blocksnthreadse[0], blocksnthreadse[1], (12 * blocksnthreadse[1]) * sizeof
                                                                                                                  (double), stream>>>
            (coord, f, beadSet, kstr, eql, pos1, pos2, params, gU_i);
        auto cvars = CUDAcommon::getCUDAvars();
        cvars.streamvec.push_back(&stream);
        CUDAcommon::cudavars = cvars;
//        hipEventRecord(event, stream);
        nvtxRangePop();
        nvtxRangePushA("cmseError");
                CUDAcommon::handleerror( hipGetLastError(), "MotorGhostStretchingHarmonicenergy",
                                         "MotorGhostStretchingHarmonic.cu");
        nvtxRangePop();
        //    CUDAcommon::handleerror( hipPeekAtLastError() );
//    CUDAcommon::handleerror( hipDeviceSynchronize() );
        nvtxRangePushA("cmseadd");
        double* gpu_Utot = CUDAcommon::getCUDAvars().gpu_energy;
        addvector<<<1,1,0,stream>>>(gU_i,params, gU_sum, gpu_Utot, CUDAcommon::getCUDAvars().gculpritID,
                CUDAcommon::getCUDAvars().gculpritFF,
                CUDAcommon::getCUDAvars().gculpritinteraction, gFF, ginteraction);
        nvtxRangePop();
        nvtxRangePushA("cmseError");
        CUDAcommon::handleerror( hipGetLastError() , "MotorGhostStretchingHarmonicenergy",
                                 "MotorGhostStretchingHarmonic.cu");
        nvtxRangePop();
//        nvtxRangePushA("cmsecopy");
//        double U_i[1];
//        CUDAcommon::handleerror(hipMemcpy(U_i, gU_sum, sizeof(double),
//                                           hipMemcpyDeviceToHost));
//        nvtxRangePop();


//        CUDAcommon::handleerror(hipMemcpy(U_ii, gU_ii, blocksnthreads[0] * blocksnthreads[1] * sizeof(double),
//                                           hipMemcpyDeviceToHost));
//        CUDAcommon::handleerror(hipMemcpy(c1, gc1, 3*blocksnthreads[1]*sizeof(double), hipMemcpyDeviceToHost));
//        CUDAcommon::handleerror(hipMemcpy(c2, gc2, 3*blocksnthreads[1]*sizeof(double), hipMemcpyDeviceToHost));
//        CUDAcommon::handleerror(hipMemcpy(checkU, gcheckU, blocksnthreads[1]*sizeof(double), hipMemcpyDeviceToHost));

//    for(auto j=0;j<blocksnthreads[0]* blocksnthreads[1];j++) {
//        std::cout << U_ii[j] <<" "<<checkU[j]<<endl;
//    }

//    for(auto j=0;j<blocksnthreads[0]* blocksnthreads[1];j++) {
//        std::cout << U_i[j] << endl;
//    }
//    CylinderExclVolume<CylinderExclVolRepulsion>::numInteractions
//    for(auto j=0;j<blocksnthreads[0] * blocksnthreads[1];j++) {
////        std::cout << c1[3 * j] << " " << c1[3 * j + 1] << " " << c1[3 * j + 2] << " " << c2[3 * j] << " "
////                  << c2[3 * j + 1] << " " << c2[3 * j + 2] << " ";
//        std::cout << U_i[j] << endl;
//    }
//    std::cout<<"*************"<<endl;

//    auto j=blocksnthreads[1]; int idx=0;
//    while(j!=1)
//    {std::cout<<checkU[idx]<<endl;idx=idx+1;j=j/2;}
//    for (auto i=0;i<blocksnthreads[1];i++)
//        std::cout<<checkU[i]<<endl;
//    std::cout<<"----------------"<<endl;


//    if(U_i[0]!=-1.0) {
//        for (auto i = 1; i < blocksnthreadse[0] * blocksnthreadse[1]; i++) {
////            std::cout<<U_i[i]<<endl;
//            U_i[0] = U_i[0] + U_i[i];
//            if (U_i[i] == -1.0) {
//                U_i[0] = -1.0;
//                break;
//            }
//        }
//    }
//    std::cout<<"MS Total energy CUDA   "<<U_i[0]<<endl;

    //CUDAcommon::handleerror(hipFree(gU_i));

//        CUDAcommon::handleerror(hipFree(gU_ii));
//        CUDAcommon::handleerror(hipFree(gc1));
//        CUDAcommon::handleerror(hipFree(gc2));
//        CUDAcommon::handleerror(hipFree(gcheckU));
//    gU_i = NULL;
//    gU_ii = NULL;
//    gc1 = NULL;
//    gc2 = NULL;
//    gcheckU = NULL;
//    free(U_i);


//        CUDAcommon::handleerror(hipStreamSynchronize(stream));
    return gU_sum;}
    else
        return NULL;
}


double* MotorGhostStretchingHarmonic::energy(double *coord, double *f, int *beadSet,
                                            double *kstr, double *eql, double *pos1, double *pos2, double *z,
                                            int *params) {

    ///TEST CODE ///
//    double *gcheckvar;
//    double checkvar[blocksnthreads[0]*blocksnthreads[1]];
//    CUDAcommon::handleerror(hipMalloc((void **) &gcheckvar, blocksnthreads[0] * blocksnthreads[1]*sizeof(double)));
//    testifitworks<<<blocksnthreads[0], blocksnthreads[1] >>>(coord,gcheckvar);
//    hipMemcpy(checkvar, gcheckvar, blocksnthreads[0] * blocksnthreads[1]*sizeof(double), hipMemcpyDeviceToHost);
//    for (auto i = 1; i < blocksnthreads[0] * blocksnthreads[1]; i++) {
//        std::cout<<checkvar[i]<<" ";
//    }
//    std::cout<<endl;
//    hipFree(gcheckvar);
//    gcheckvar = NULL;
    ///@@END@@///
    if(blocksnthreadsez[1]>0) {
//        hipStream_t  stream;
//        hipEvent_t event;
//        CUDAcommon::handleerror(hipStreamCreate(&stream));
//        CUDAcommon::handleerror(hipEventCreate(&event));
//        auto cvars = CUDAcommon::getCUDAvars();
//        cvars.eventvec.push_back(event);
//        cvars.streamvec.push_back(stream);
//        CUDAcommon::cudavars = cvars;
//        double dd[1];
//        CUDAcommon::handleerror(hipMemcpy(dd, z, sizeof(double), hipMemcpyDeviceToHost));
//        std::cout << "d = " << dd[0] << endl;

//        double *gU_ii;
        //double *gU_i;
//        double *gc1, *gc2, *gcheckU;
//        double U_ii[blocksnthreads[0] * blocksnthreads[1]];
//        double c1[3 * blocksnthreads[0] * blocksnthreads[1]], c2[3 * blocksnthreads[0] * blocksnthreads[1]];
        //double U_i[blocksnthreadsez[0] * blocksnthreadsez[1]];
//        double checkU[33 * blocksnthreads[0] * blocksnthreads[1]];



//    double ccoord[3*Bead::getBeads().size()];
//    hipMemcpy(ccoord, coord, 3*Bead::getBeads().size()*sizeof(double), hipMemcpyDeviceToHost);
//    double cforce[3*Bead::getBeads().size()];
//    hipMemcpy(cforce, f, 3*Bead::getBeads().size()*sizeof(double), hipMemcpyDeviceToHost);
//        int cparams[2];
//        CUDAcommon::handleerror(hipMemcpy(cparams, params, 2 * sizeof(int), hipMemcpyDeviceToHost));
//    for(auto i =0; i < Bead::getBeads().size(); i++)
//
//        std::cout<<ccoord[3 * i]<<" "<<ccoord[3 * i +1]<<" "<<ccoord[3 * i +2]<<" "<<cforce[3 * i]<<" "<<cforce[3 * i
//                                                                                                                +1]<<" "<<cforce[3 * i +2]<<endl;
//        std::cout << cparams[0] << " " << cparams[1] << endl;

//    std::cout<<"C+F Z---------------------------- "<<endl;

//        std::cout << "MSEZ Number of Blocks: " << blocksnthreads[0] << endl;
//        std::cout << "Threads per block: " << blocksnthreads[1] << endl;



        //TODO  since the number of threads needed is constant through out the minimization, consider storing the pointer.
        //CUDAcommon::handleerror(hipMalloc((void **) &gU_i, blocksnthreadsez[0] * blocksnthreadsez[1] * sizeof (double)));
//        CUDAcommon::handleerror(hipMalloc((void **) &gU_ii, blocksnthreads[0] * blocksnthreads[1] * sizeof(double)));
//        CUDAcommon::handleerror(hipMalloc((void **) &gc1, 3 * blocksnthreads[1] * sizeof(double)));
//        CUDAcommon::handleerror(hipMalloc((void **) &gc2, 3 * blocksnthreads[1] * sizeof(double)));
//        CUDAcommon::handleerror(hipMalloc((void **) &gcheckU, 33 * blocksnthreads[0] * blocksnthreads[1] * sizeof
//        (double)));
        //
//    size_t freeMem, totalMem;
//
//    hipMemGetInfo(&freeMem, &totalMem);
//
//    std::cout<<"Memory "<<freeMem<<" "<<totalMem<<endl;
//        struct hipDeviceProp_t properties;
//        hipGetDeviceProperties(&properties, 0);
//        cout << "using " << properties.multiProcessorCount << " multiprocessors" << endl;
//        cout << "max threads per processor: " << properties.maxThreadsPerMultiProcessor << endl;
//        std::cout << 24 * blocksnthreads[1] * sizeof(double) << endl;

//        if(blocksnthreadsez[1]==THREADSPERBLOCK) {
//            blocksnthreadsez[0] = 2 * blocksnthreadsez[0];
//            blocksnthreadsez[1] = 1/2 * blocksnthreadsez[1];
//        }
        nvtxRangePushA("cmsez");
        MotorGhostStretchingHarmonicenergyz << < blocksnthreadsez[0], blocksnthreadsez[1], (24 * blocksnthreadsez[1]) *
                                                                                                sizeof
                (double), stream>> > (coord, f, beadSet, kstr, eql, pos1, pos2, params, gU_i, z );
        auto cvars = CUDAcommon::getCUDAvars();
        cvars.streamvec.push_back(&stream);
        CUDAcommon::cudavars = cvars;
        nvtxRangePop();
        nvtxRangePushA("cmsezError");
        CUDAcommon::handleerror(hipGetLastError(), "MotorGhostStretchingHarmonicenergyz",
                                "MotorGhostStretchingHarmonic.cu");
        nvtxRangePop();
//    CUDAcommon::handleerror( hipPeekAtLastError() );
//        CUDAcommon::handleerror(hipDeviceSynchronize());
        nvtxRangePushA("cmsezadd");
        double* gpu_Utot = CUDAcommon::getCUDAvars().gpu_energy;

        addvector<<<1,1,0,stream>>>(gU_i,params, gU_sum, gpu_Utot, CUDAcommon::getCUDAvars().gculpritID,
                CUDAcommon::getCUDAvars().gculpritFF,
                CUDAcommon::getCUDAvars().gculpritinteraction, gFF, ginteraction);
//        hipEventRecord(event, stream);
        nvtxRangePop();
        nvtxRangePushA("cmsezError");
        CUDAcommon::handleerror(hipGetLastError(), "MotorGhostStretchingHarmonicenergyz",
                                "MotorGhostStretchingHarmonic.cu");
        nvtxRangePop();

//        nvtxRangePushA("cmsezcopy");
//        double U_i[1];
//        CUDAcommon::handleerror(hipMemcpy(U_i, gU_sum, sizeof(double),
//                                           hipMemcpyDeviceToHost));
//        nvtxRangePop();
//
//        CUDAcommon::handleerror(hipMemcpy(U_i, gU_i, blocksnthreadsez[0] * blocksnthreadsez[1] * sizeof(double),
//                                                                                                       hipMemcpyDeviceToHost));
//        CUDAcommon::handleerror(hipMemcpy(U_ii, gU_ii, blocksnthreads[0] * blocksnthreads[1] * sizeof(double),
//                                           hipMemcpyDeviceToHost));
//        CUDAcommon::handleerror(hipMemcpy(c1, gc1, 3 * blocksnthreads[1] * sizeof(double), hipMemcpyDeviceToHost));
//        CUDAcommon::handleerror(hipMemcpy(c2, gc2, 3 * blocksnthreads[1] * sizeof(double), hipMemcpyDeviceToHost));
//        CUDAcommon::handleerror(hipMemcpy(checkU, gcheckU, 33 * blocksnthreads[0] * blocksnthreads[1] * sizeof(double),
//                                           hipMemcpyDeviceToHost));
//
//        for(auto i=0;i<blocksnthreads[0] * blocksnthreads[1]; i++) {
//            for (auto iter = 0; iter < 33; iter++)
//                std::cout << checkU[33 * i + iter] << " ";
//            std::cout<<U_i[i]<<endl;
//        }
//        if (U_i[0] != -1.0) {
//            for (auto i = 1; i < blocksnthreadsez[0] * blocksnthreadsez[1]; i++) {
//                U_i[0] = U_i[0] + U_i[i];
//                if (U_i[i] == -1.0) {
//                    U_i[0] = -1.0;
//                    break;
//                }
//            }
//        }
//        std::cout << "MSZ Total energy CUDA   " << U_i[0] << endl;

//        CUDAcommon::handleerror(hipFree(gU_i));

//        CUDAcommon::handleerror(hipFree(gU_ii));
//        CUDAcommon::handleerror(hipFree(gc1));
//        CUDAcommon::handleerror(hipFree(gc2));
//        CUDAcommon::handleerror(hipFree(gcheckU));
//    gU_i = NULL;
//    gU_ii = NULL;
//    gc1 = NULL;
//    gc2 = NULL;
//    gcheckU = NULL;
//    free(U_i);


//        CUDAcommon::handleerror(hipStreamSynchronize(stream));
        return gU_sum;
    }else
        return NULL;
}

double MotorGhostStretchingHarmonic::energy(double *coord, double *f, int *beadSet,
                                            double *kstr, double *eql, double *pos1, double *pos2) {

    int n = MotorGhostStretching<MotorGhostStretchingHarmonic>::n;
    int nint = MotorGhost::getMotorGhosts().size();

    double *coord1, *coord2, *coord3, *coord4, dist, U_i;
    double *v1 = new double[3];
    double *v2 = new double[3];

    double U = 0;


    for(int i = 0; i < nint; i += 1) {

        coord1 = &coord[3 * beadSet[n * i]];
        coord2 = &coord[3 * beadSet[n * i + 1]];
        coord3 = &coord[3 * beadSet[n * i + 2]];
        coord4 = &coord[3 * beadSet[n * i + 3]];

        midPointCoordinate(v1, coord1, coord2, pos1[i]);
        midPointCoordinate(v2, coord3, coord4, pos2[i]);

        dist = twoPointDistance(v1, v2) - eql[i];
        U_i = 0.5 * kstr[i] * dist * dist;

        if(fabs(U_i) == numeric_limits<double>::infinity()
           || U_i != U_i || U_i < -1.0) {

            //set culprit and return
            MotorGhostInteractions::_motorCulprit = MotorGhost::getMotorGhosts()[i];

            return -1;
        }

        U += U_i;
//        std::cout<<U_i<<endl;
    }
//    std::cout<<"MS Total energy serial "<< U <<endl;
    delete v1;
    delete v2;

    return U;
}

double MotorGhostStretchingHarmonic::energy(double *coord, double * f, int *beadSet,
                                            double *kstr, double *eql, double *pos1, double *pos2, double d){

    int n = MotorGhostStretching<MotorGhostStretchingHarmonic>::n;
    int nint = MotorGhost::getMotorGhosts().size();

    double *coord1, *coord2, *coord3, *coord4, *f1, *f2, *f3, *f4, dist, U_i;
    double *v1 = new double[3];
    double *v2 = new double[3];

    double U = 0;

    for(int i = 0; i < nint; i += 1) {

        coord1 = &coord[3 * beadSet[n * i]];
        coord2 = &coord[3 * beadSet[n * i + 1]];
        coord3 = &coord[3 * beadSet[n * i + 2]];
        coord4 = &coord[3 * beadSet[n * i + 3]];

        f1 = &f[3 * beadSet[n * i]];
        f2 = &f[3 * beadSet[n * i + 1]];
        f3 = &f[3 * beadSet[n * i + 2]];
        f4 = &f[3 * beadSet[n * i + 3]];

        midPointCoordinateStretched(v1, coord1, f1, coord2, f2, pos1[i], d);
        midPointCoordinateStretched(v2, coord3, f3, coord4, f4, pos2[i], d);

        dist = twoPointDistance(v1,  v2) - eql[i];
        U_i = 0.5 * kstr[i] * dist * dist;

//        std::cout<<v1[0]<<" "<<v1[1]<<" "<<v1[2]<<" "<<v2[0]<<" "<<v2[1]<<" "<<v2[2]<<" "<<coord1[0]<<" "
//                ""<<coord1[1]<<" "<<coord1[2]<<" "<<coord3[0]<<" "<<coord3[1]<<" "<<coord3[2]<<" "<<f1[0]<<" "
//                ""<<f1[1]<<" "<<f1[2]<<" "<<f3[0]<<" "<<f3[1]<<" "<<f3[2]<<" "<<coord2[0]<<" "<<coord2[1]<<" "
//                ""<<coord2[2]<<" "<<coord4[0]<<" "<<coord4[1]<<" "<<coord4[2]<<" "<<f2[0]<<" "<<f2[1]<<" "<<f2[2]<<" "
//                ""<<f4[0]<<" "<<f4[1]<<" "<<f4[2]<<" "<<pos1[i]<<" "<<pos2[i]<<" "<<d<<" "<<U_i<<endl;
        if(fabs(U_i) == numeric_limits<double>::infinity()
           || U_i != U_i || U_i < -1.0) {

            //set culprit and return
            MotorGhostInteractions::_motorCulprit = MotorGhost::getMotorGhosts()[i];

            return -1;
        }

        U += U_i;
    }

//    std::cout<<"MS Total energy serial "<< U <<endl;
    delete v1;
    delete v2;

    return U;

}

void MotorGhostStretchingHarmonic::forces(double *coord, double *f, int *beadSet,
                                          double *kstr, double *eql, double *pos1, double *pos2, int *params){
//    hipEvent_t start, stop;
//    CUDAcommon::handleerror(hipEventCreate( &start));
//    CUDAcommon::handleerror(hipEventCreate( &stop));
//    CUDAcommon::handleerror(hipEventRecord( start, 0));
    if(blocksnthreadsf[1]>0) {
//        double *gU_i;
//        double *gc1, *gc2, *gcheckU;

//        double c1[3 * blocksnthreads[0] * blocksnthreads[1]], c2[3 * blocksnthreads[0] * blocksnthreads[1]];
//        double cvar[36 * blocksnthreads[0] * blocksnthreads[1]];

//        std::cout << "MSF Number of Blocks: " << blocksnthreads[0] << endl;
//        std::cout << "Threads per block: " << blocksnthreads[1] << endl;

        //TODO  since the number of threads needed is constant through out the minimization, consider storing the pointer.
//        CUDAcommon::handleerror(hipMalloc((void **) &gU_i, 36 * blocksnthreads[0] * blocksnthreads[1] * sizeof
//                                                                                                                (double)));


//        double F_c[3*Bead::getBeads().size()];
//        double C_c[3*Bead::getBeads().size()];
//        //TODO remove this later need not copy forces back to CPU.
//        CUDAcommon::handleerror(hipMemcpy(F_c, f, 3 * Bead::getBeads().size() *sizeof(double),
//                                           hipMemcpyDeviceToHost));
//        CUDAcommon::handleerror(hipMemcpy(C_c, coord, 3 * Bead::getBeads().size() *sizeof(double),
//                                           hipMemcpyDeviceToHost));
//        for(int iter=0;iter<Bead::getBeads().size();iter++) {
//            std::cout << C_c[3 * iter] << " " << C_c[3 * iter + 1] << " " << C_c[3 * iter + 2]<<" "<<F_c[3 * iter] <<
//            " " << F_c[3 * iter + 1] << " " << F_c[3 * iter + 2] <<endl;
//        }
//
//        std::cout<<"check ends "<<blocksnthreads[0]<<" "<<blocksnthreads[1]<<endl;
//
//    size_t freeMem, totalMem;
//
//    hipMemGetInfo(&freeMem, &totalMem);
//
//    std::cout<<"Memory "<<freeMem<<" "<<totalMem<<endl;
//        struct hipDeviceProp_t properties;
//        hipGetDeviceProperties(&properties, 0);
//        cout << "using " << properties.multiProcessorCount << " multiprocessors" << endl;
//        cout << "max threads per processor: " << properties.maxThreadsPerMultiProcessor << endl;
//        std::cout << 36 *  blocksnthreads[0] *blocksnthreads[1] * sizeof(double) << endl;

        MotorGhostStretchingHarmonicforces << < blocksnthreadsf[0], blocksnthreadsf[1], (12 *
        blocksnthreadsf[1]) * sizeof (double), stream >> > (coord, f, beadSet, kstr, eql, pos1, pos2, params);
        auto cvars = CUDAcommon::getCUDAvars();
        cvars.streamvec.push_back(&stream);
        CUDAcommon::cudavars = cvars;
        //CUDAcommon::handleerror(hipDeviceSynchronize());
        CUDAcommon::handleerror(hipGetLastError(), "MotorGhostStretchingHarmonicforces",
                                "MotorGhostStretchingHarmonic.cu");

//    CUDAcommon::handleerror( hipPeekAtLastError() );
//        CUDAcommon::handleerror(hipDeviceSynchronize());

//        CUDAcommon::handleerror(hipMemcpy(cvar, gU_i, 36 * blocksnthreads[0]*blocksnthreads[1]*sizeof(double),
//                                           hipMemcpyDeviceToHost));
//        for(auto i=0; i<blocksnthreads[0]*blocksnthreads[1]; i++) {
//            for(auto iter=0;iter<36;iter++) {
//                std::cout <<cvar[36 * i + iter]<<" ";
//            }
//            std::cout<<endl;
//        }

//    hipMemcpy(F_i, f, 3 * blocksnthreads[0]*blocksnthreads[1]*sizeof(double), hipMemcpyDeviceToHost);
//        CUDAcommon::handleerror(hipFree(gU_i));
    }
//    CUDAcommon::handleerror(hipEventRecord( stop, 0));
//    CUDAcommon::handleerror(hipEventSynchronize(stop));
//    float elapsedtime;
//    CUDAcommon::handleerror(hipEventElapsedTime(&elapsedtime, start, stop));
//    CUDAvars cvars=CUDAcommon::getCUDAvars();
//    cvars.Ccforce += elapsedtime;
//    std::cout<<"C CFM "<<elapsedtime<<endl;
//    CUDAcommon::cudavars=cvars;
//    CUDAcommon::handleerror(hipEventDestroy(start));
//    CUDAcommon::handleerror(hipEventDestroy(stop));
}

void MotorGhostStretchingHarmonic::forces(double *coord, double *f, int *beadSet,
                                          double *kstr, double *eql, double *pos1, double *pos2){

    int n = MotorGhostStretching<MotorGhostStretchingHarmonic>::n;
    int nint = MotorGhost::getMotorGhosts().size();

    double *coord1, *coord2, *coord3, *coord4, dist, invL;
    double *v1 = new double[3];
    double *v2 = new double[3];

    double f0, *f1, *f2, *f3, *f4;

    for(int i = 0; i < nint; i += 1) {

        coord1 = &coord[3 * beadSet[n * i]];
        coord2 = &coord[3 * beadSet[n * i + 1]];
        coord3 = &coord[3 * beadSet[n * i + 2]];
        coord4 = &coord[3 * beadSet[n * i + 3]];

        midPointCoordinate(v1, coord1, coord2, pos1[i]);
        midPointCoordinate(v2, coord3, coord4, pos2[i]);


        dist = twoPointDistance(v1, v2) ;
        invL = 1 / dist;

        f0 = kstr[i] * ( dist - eql[i] ) * invL;

        f1 = &f[3 * beadSet[n * i]];
        f2 = &f[3 * beadSet[n * i + 1]];
        f3 = &f[3 * beadSet[n * i + 2]];
        f4 = &f[3 * beadSet[n * i + 3]];

//        std::cout<<coord1[0]<<" "<<coord1[1]<<" "<<coord1[2]<<" "<<coord2[0]<<" "<<coord2[1]<<" "<<coord2[2]<<" "
//                ""<<coord3[0]<<" "<<coord3[1]<<" "<<coord3[2]<<" "<<coord4[0]<<" "<<coord4[1]<<" "<<coord4[2]<<" ";
//        std::cout<<-f0 * ( v1[0] - v2[0] ) * (1 - pos1[i])<<" "
//                 <<-f0 * ( v1[1] - v2[1] ) * (1 - pos1[i])<<" "
//                 <<-f0 * ( v1[2] - v2[2] ) * (1 - pos1[i])<<" "
//                 <<-f0 * ( v1[0] - v2[0] ) * (pos1[i])<<" "
//                 <<-f0 * ( v1[1] - v2[1] ) * (pos1[i])<<" "
//                 <<-f0 * ( v1[2] - v2[2] ) * (pos1[i])<<" "
//                 <<f0 * ( v1[0] - v2[0] ) * (1 - pos2[i])<<" "
//                 <<f0 * ( v1[1] - v2[1] ) * (1 - pos2[i])<<" "
//                 <<f0 * ( v1[2] - v2[2] ) * (1 - pos2[i])<<" "
//                 <<f0 * ( v1[0] - v2[0] ) * (pos2[i])<<" "
//                 <<f0 * ( v1[1] - v2[1] ) * (pos2[i])<<" "
//                 <<f0 * ( v1[2] - v2[2] ) * (pos2[i])<<" ";
//        std::cout<<f1[0]<<" "<<f1[1]<<" "<<f1[2]<<" "<<f2[0]<<" "<<f2[1]<<" "<<f2[2]<<" "<<f3[0]<<" "
//                ""<<f3[1]<<" "<<f3[2]<<" "<<f4[0]<<" "<<f4[1]<<" "<<f4[2]<<endl;

        //force on i
        f1[0] +=   -f0 * ( v1[0] - v2[0] ) * (1 - pos1[i]);
        f1[1] +=   -f0 * ( v1[1] - v2[1] ) * (1 - pos1[i]);
        f1[2] +=   -f0 * ( v1[2] - v2[2] ) * (1 - pos1[i]);

        // force i+1
        f2[0] +=   -f0 * ( v1[0] - v2[0] ) * (pos1[i]);
        f2[1] +=   -f0 * ( v1[1] - v2[1] ) * (pos1[i]);
        f2[2] +=   -f0 * ( v1[2] - v2[2] ) * (pos1[i]);

        //force on j
        f3[0] +=   f0 * ( v1[0] - v2[0] ) * (1 - pos2[i]);
        f3[1] +=   f0 * ( v1[1] - v2[1] ) * (1 - pos2[i]);
        f3[2] +=   f0 * ( v1[2] - v2[2] ) * (1 - pos2[i]);

        // force j+1
        f4[0] +=   f0 * ( v1[0] - v2[0] ) * (pos2[i]);
        f4[1] +=   f0 * ( v1[1] - v2[1] ) * (pos2[i]);
        f4[2] +=   f0 * ( v1[2] - v2[2] ) * (pos2[i]);
    }
    delete v1;
    delete v2;

}
