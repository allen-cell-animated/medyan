
//------------------------------------------------------------------
//  **MEDYAN** - Simulation Package for the Mechanochemical
//               Dynamics of Active Networks, v3.1
//
//  Copyright (2015-2016)  Papoian Lab, University of Maryland
//
//                 ALL RIGHTS RESERVED
//
//  See the MEDYAN web page for more information:
//  http://www.medyan.org
//------------------------------------------------------------------
#include "FilamentBending.h"

#include "FilamentBendingHarmonic.h"
#include "FilamentBendingCosine.h"

#include "Filament.h"
#include "Cylinder.h"
#include "Bead.h"
#include "nvToolsExt.h"
#include "cross_check.h"
template <class FBendingInteractionType>
void FilamentBending<FBendingInteractionType>::vectorize() {

    int numInteractions = Bead::getBeads().size() - 2 * Filament::getFilaments().size();

    beadSet = new int[n * numInteractions];
    kbend = new double[numInteractions];
    eqt = new double[numInteractions];

    int i = 0;

    for (auto f: Filament::getFilaments()) {

        if (f->getCylinderVector().size() > 1){

            for (auto it = f->getCylinderVector().begin()+1;
                 it != f->getCylinderVector().end(); it++){

                auto it2 = it - 1;
                beadSet[n * i] = (*it2)->getFirstBead()->_dbIndex;
                beadSet[n * i + 1] = (*it)->getFirstBead()->_dbIndex;;
                beadSet[n * i + 2] = (*it)->getSecondBead()->_dbIndex;;

                kbend[i] = (*it)->getMCylinder()->getBendingConst();
                eqt[i]  = (*it)->getMCylinder()->getEqTheta();

                i++;
            }
        }
    }

    //CUDA
#ifdef CUDAACCL
//    F_i = new double[3 * Bead::getBeads().size()];
//    nvtxRangePushA("CVFF");
    _FFType.optimalblocksnthreads(numInteractions);

    CUDAcommon::handleerror(hipMalloc((void **) &gpu_beadSet, n * numInteractions * sizeof(int)));
    CUDAcommon::handleerror(hipMemcpy(gpu_beadSet, beadSet, n * numInteractions * sizeof(int),
                                       hipMemcpyHostToDevice));

    CUDAcommon::handleerror(hipMalloc((void **) &gpu_kbend, numInteractions * sizeof(double)));
    CUDAcommon::handleerror(hipMemcpy(gpu_kbend, kbend, numInteractions * sizeof(double), hipMemcpyHostToDevice));

    CUDAcommon::handleerror(hipMalloc((void **) &gpu_eqt, numInteractions * sizeof(double)));
    CUDAcommon::handleerror(hipMemcpy(gpu_eqt, eqt, numInteractions * sizeof(double), hipMemcpyHostToDevice));

    vector<int> params;
    params.push_back(int(n));
    params.push_back(numInteractions);
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_params, 2 * sizeof(int)));
    CUDAcommon::handleerror(hipMemcpy(gpu_params, params.data(), 2 * sizeof(int), hipMemcpyHostToDevice));
//    nvtxRangePop();
#endif
}

template<class FBendingInteractionType>
void FilamentBending<FBendingInteractionType>::deallocate() {

    delete beadSet;
    delete kbend;
    delete eqt;
#ifdef CUDAACCL
    _FFType.deallocate();
    CUDAcommon::handleerror(hipFree(gpu_beadSet));
    CUDAcommon::handleerror(hipFree(gpu_kbend));
    CUDAcommon::handleerror(hipFree(gpu_eqt));
    CUDAcommon::handleerror(hipFree(gpu_params));
#endif
}


template <class FBendingInteractionType>
double FilamentBending<FBendingInteractionType>::computeEnergy(double *coord, double *f, double d){

    double U_i[1], U_ii;
    double* gU_i;
    U_ii = NULL;
#ifdef CUDAACCL
    //has to be changed to accomodate aux force
    double * gpu_coord=CUDAcommon::getCUDAvars().gpu_coord;
    double * gpu_force=CUDAcommon::getCUDAvars().gpu_force;
    double * gpu_d = CUDAcommon::getCUDAvars().gpu_lambda;
//    nvtxRangePushA("CCEFB");

//    if(d == 0.0){
//        gU_i=_FFType.energy(gpu_coord, gpu_force, gpu_beadSet, gpu_kbend, gpu_eqt, gpu_params);
//
//    }
//    else{
        gU_i=_FFType.energy(gpu_coord, gpu_force, gpu_beadSet, gpu_kbend, gpu_eqt, gpu_d,
                            gpu_params);
//    }
//    nvtxRangePop();
#endif
#ifdef SERIAL
//    nvtxRangePushA("SCEFB");
    if (d == 0.0)
        U_ii = _FFType.energy(coord, f, beadSet, kbend, eqt);
    else
        U_ii= _FFType.energy(coord, f, beadSet, kbend, eqt, d);
//    nvtxRangePop();
#endif
    return U_ii;

}

template <class FBendingInteractionType>
void FilamentBending<FBendingInteractionType>::computeForces(double *coord, double *f) {
#ifdef CUDAACCL
    //has to be changed to accomodate aux force
    double * gpu_coord=CUDAcommon::getCUDAvars().gpu_coord;

    double * gpu_force;


    if(cross_checkclass::Aux){
//        nvtxRangePushA("CCFFB");

        gpu_force=CUDAcommon::getCUDAvars().gpu_forceAux;
        _FFType.forces(gpu_coord, gpu_force, gpu_beadSet, gpu_kbend, gpu_eqt, gpu_params);
//        nvtxRangePop();
    }
    else {
//        nvtxRangePushA("CCFFB");

        gpu_force = CUDAcommon::getCUDAvars().gpu_force;
        _FFType.forces(gpu_coord, gpu_force, gpu_beadSet, gpu_kbend, gpu_eqt, gpu_params);
//        nvtxRangePop();
    }
#endif
#ifdef SERIAL
//    nvtxRangePushA("SCFFB");

    _FFType.forces(coord, f, beadSet, kbend, eqt);
//    nvtxRangePop();
#endif

}

///Template specializations
template double FilamentBending<FilamentBendingHarmonic>::computeEnergy(double *coord, double *f, double d);
template void FilamentBending<FilamentBendingHarmonic>::computeForces(double *coord, double *f);
template void FilamentBending<FilamentBendingHarmonic>::vectorize();
template void FilamentBending<FilamentBendingHarmonic>::deallocate();


template double FilamentBending<FilamentBendingCosine>::computeEnergy(double *coord, double *f, double d);
template void FilamentBending<FilamentBendingCosine>::computeForces(double *coord, double *f);
template void FilamentBending<FilamentBendingCosine>::vectorize();
template void FilamentBending<FilamentBendingCosine>::deallocate();
