
//------------------------------------------------------------------
//  **MEDYAN** - Simulation Package for the Mechanochemical
//               Dynamics of Active Networks, v3.1
//
//  Copyright (2015-2016)  Papoian Lab, University of Maryland
//
//                 ALL RIGHTS RESERVED
//
//  See the MEDYAN web page for more information:
//  http://www.medyan.org
//------------------------------------------------------------------

#include "LinkerStretching.h"

#include "LinkerStretchingHarmonic.h"

#include "Cylinder.h"
#include "Linker.h"
#include "Bead.h"
#include "cross_check.h"
#include "nvToolsExt.h"
template <class LStretchingInteractionType>
void LinkerStretching<LStretchingInteractionType>::vectorize() {

    beadSet = new int[n * Linker::getLinkers().size()];
    kstr = new double[Linker::getLinkers().size()];
    eql = new double[Linker::getLinkers().size()];
    pos1 = new double[Linker::getLinkers().size()];
    pos2 = new double[Linker::getLinkers().size()];

    int i = 0;

    for (auto l: Linker::getLinkers()) {

        beadSet[n * i] = l->getFirstCylinder()->getFirstBead()->_dbIndex;
        beadSet[n * i + 1] = l->getFirstCylinder()->getSecondBead()->_dbIndex;
        beadSet[n * i + 2] = l->getSecondCylinder()->getFirstBead()->_dbIndex;
        beadSet[n * i + 3] = l->getSecondCylinder()->getSecondBead()->_dbIndex;

        kstr[i] = l->getMLinker()->getStretchingConstant();
        eql[i] = l->getMLinker()->getEqLength();
        pos1[i] = l->getFirstPosition();
        pos2[i] = l->getSecondPosition();

        i++;
    }

    //CUDA
#ifdef CUDAACCL
//    F_i = new double[3 * Bead::getBeads().size()];
//    hipEvent_t start, stop;
//    CUDAcommon::handleerror(hipEventCreate( &start));
//    CUDAcommon::handleerror(hipEventCreate( &stop));
//    CUDAcommon::handleerror(hipEventRecord( start, 0));
    nvtxRangePushA("CVFF");

    int numInteractions =Linker::getLinkers().size();
    _FFType.optimalblocksnthreads(numInteractions);
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_beadSet, n * numInteractions * sizeof(int)),"cuda data transfer",
                                       "LinkerStretching.cu");
    CUDAcommon::handleerror(hipMemcpy(gpu_beadSet, beadSet, n * numInteractions * sizeof(int),
                                       hipMemcpyHostToDevice),"cuda data transfer", "LinkerStretching.cu");

    CUDAcommon::handleerror(hipMalloc((void **) &gpu_kstr, numInteractions * sizeof(double)),"cuda data transfer",
                                       "LinkerStretching.cu");
    CUDAcommon::handleerror(hipMemcpy(gpu_kstr, kstr, numInteractions * sizeof(double), hipMemcpyHostToDevice),
                            "cuda data transfer", "LinkerStretching.cu");

    CUDAcommon::handleerror(hipMalloc((void **) &gpu_eql, numInteractions * sizeof(double)),"cuda data transfer",
                            "LinkerStretching.cu");
    CUDAcommon::handleerror(hipMemcpy(gpu_eql, eql, numInteractions * sizeof(double), hipMemcpyHostToDevice),
                            "cuda data transfer", "LinkerStretching.cu");
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_pos1, numInteractions * sizeof(double)),"cuda data transfer",
                            "LinkerStretching.cu");
    CUDAcommon::handleerror(hipMemcpy(gpu_pos1, pos1, numInteractions * sizeof(double), hipMemcpyHostToDevice),
                            "cuda data transfer", "LinkerStretching.cu");
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_pos2, numInteractions * sizeof(double)),"cuda data transfer",
                            "LinkerStretching.cu");
    CUDAcommon::handleerror(hipMemcpy(gpu_pos2, pos2, numInteractions * sizeof(double), hipMemcpyHostToDevice),
                            "cuda data transfer", "LinkerStretching.cu");
    vector<int> params;
    params.push_back(int(n));
    params.push_back(numInteractions);

    CUDAcommon::handleerror(hipMalloc((void **) &gpu_params, 2 * sizeof(int)),"cuda data transfer",
                            "LinkerStretching.cu");
    CUDAcommon::handleerror(hipMemcpy(gpu_params, params.data(), 2 * sizeof(int), hipMemcpyHostToDevice),
                            "cuda data transfer", "LinkerStretching.cu");
    nvtxRangePop();
#endif
}

template<class LStretchingInteractionType>
void LinkerStretching<LStretchingInteractionType>::deallocate() {

    delete beadSet;
    delete kstr;
    delete eql;
    delete pos1;
    delete pos2;
#ifdef CUDAACCL
    _FFType.deallocate();
    CUDAcommon::handleerror(hipFree(gpu_beadSet),"hipFree", "LinkerStretching.cu");
    CUDAcommon::handleerror(hipFree(gpu_kstr),"hipFree", "LinkerStretching.cu");
    CUDAcommon::handleerror(hipFree(gpu_pos1),"hipFree", "LinkerStretching.cu");
    CUDAcommon::handleerror(hipFree(gpu_pos2),"hipFree", "LinkerStretching.cu");
    CUDAcommon::handleerror(hipFree(gpu_eql),"hipFree", "LinkerStretching.cu");
    CUDAcommon::handleerror(hipFree(gpu_params),"hipFree", "LinkerStretching.cu");
#endif
}


template <class LStretchingInteractionType>
double LinkerStretching<LStretchingInteractionType>::computeEnergy(double* coord, double *f, double d){

    double U_i[1], U_ii;
    double* gU_i;
    U_ii = NULL;
#ifdef CUDAACCL
    //has to be changed to accomodate aux force
    double * gpu_coord=CUDAcommon::getCUDAvars().gpu_coord;
    double * gpu_force=CUDAcommon::getCUDAvars().gpu_force;
    double * gpu_d = CUDAcommon::getCUDAvars().gpu_lambda;
    nvtxRangePushA("CCEL");

//    if(d == 0.0){
//        gU_i=_FFType.energy(gpu_coord, gpu_force, gpu_beadSet, gpu_kstr, gpu_eql, gpu_pos1, gpu_pos2, gpu_params);
//    }
//    else{
        gU_i=_FFType.energy(gpu_coord, gpu_force, gpu_beadSet, gpu_kstr, gpu_eql, gpu_pos1, gpu_pos2, gpu_d,
                            gpu_params);
//    }
    nvtxRangePop();
#else
    nvtxRangePushA("SCEL");

    if (d == 0.0)
        U_ii = _FFType.energy(coord, f, beadSet, kstr, eql, pos1, pos2);
    else
        U_ii = _FFType.energy(coord, f, beadSet, kstr, eql, pos1, pos2, d);
//    std::cout<<"================="<<endl;
    nvtxRangePop();
#endif

    return U_ii;
}

template <class LStretchingInteractionType>
void LinkerStretching<LStretchingInteractionType>::computeForces(double *coord, double *f) {
#ifdef CUDAACCL
    //has to be changed to accomodate aux force
    double * gpu_coord=CUDAcommon::getCUDAvars().gpu_coord;

    double * gpu_force;

//    //TODO remove this later need not copy forces back to CPU.
//    CUDAcommon::handleerror(hipMemcpy(F_c, gpu_force, 3 * Bead::getBeads().size() *sizeof(double),
//                                       hipMemcpyDeviceToHost));
//    cout.precision(dbl::max_digits10);
//    for(int iter=0;iter<Bead::getBeads().size();iter++) {
//        std::cout << "C " << F_c[3 * iter] << " " << F_c[3 * iter + 1] << " " << F_c[3 * iter + 2] <<" ";
//        std::cout << "V "<<f[3 * iter] << " " << f[3 * iter + 1] << " " << f[3 * iter + 2] << endl;
//    }
//    std::cout<<"check ends "<<blocksnthreads[0]<<" "<<blocksnthreads[1]<<endl;

    if(cross_checkclass::Aux){
        nvtxRangePushA("CCFL");

        gpu_force=CUDAcommon::getCUDAvars().gpu_forceAux;
        _FFType.forces(gpu_coord, gpu_force, gpu_beadSet, gpu_kstr, gpu_eql, gpu_pos1, gpu_pos2, gpu_params);
        nvtxRangePop();
    }
    else {
        nvtxRangePushA("CCFL");

        gpu_force = CUDAcommon::getCUDAvars().gpu_force;
        _FFType.forces(gpu_coord, gpu_force, gpu_beadSet, gpu_kstr, gpu_eql, gpu_pos1, gpu_pos2, gpu_params);
        nvtxRangePop();
    }

    //TODO remove this later need not copy forces back to CPU.
//    CUDAcommon::handleerror(hipMemcpy(F_i, gpu_force, 3 * Bead::getBeads().size() *sizeof(double),
//                                       hipMemcpyDeviceToHost),"cuda data transfer", "LinkerStretching.cu");
#else
    nvtxRangePushA("SCFL");
    _FFType.forces(coord, f, beadSet, kstr, eql, pos1, pos2);
    nvtxRangePop();
#endif
}


///Temlate specializations
template double LinkerStretching<LinkerStretchingHarmonic>::computeEnergy(double *coord, double *f, double d);
template void LinkerStretching<LinkerStretchingHarmonic>::computeForces(double *coord, double *f);
template void LinkerStretching<LinkerStretchingHarmonic>::vectorize();
template void LinkerStretching<LinkerStretchingHarmonic>::deallocate();

