#include "hip/hip_runtime.h"

//------------------------------------------------------------------
//  **MEDYAN** - Simulation Package for the Mechanochemical
//               Dynamics of Active Networks, v3.1
//
//  Copyright (2015-2016)  Papoian Lab, University of Maryland
//
//                 ALL RIGHTS RESERVED
//
//  See the MEDYAN web page for more information:
//  http://www.medyan.org
//------------------------------------------------------------------

#include <cmath>
#include <math.h>

#include "BranchingPositionCosine.h"
#include "BranchingPositionCosineCUDA.h"
#include "BranchingPosition.h"

#include "BranchingPoint.h"
#include "MathFunctions.h"


using namespace mathfunc;
#ifdef CUDAACCL
void BranchingPositionCosine::deallocate(){
    if(!(CUDAcommon::getCUDAvars().conservestreams))
        CUDAcommon::handleerror(hipStreamDestroy(stream));
    CUDAcommon::handleerror(hipFree(gU_i));
    CUDAcommon::handleerror(hipFree(gU_sum));
    CUDAcommon::handleerror(hipFree(gFF));
    CUDAcommon::handleerror(hipFree(ginteraction));
}
void BranchingPositionCosine::optimalblocksnthreads( int nint){
    //CUDA stream create
    if(stream == NULL || !(CUDAcommon::getCUDAvars().conservestreams))
        CUDAcommon::handleerror(hipStreamCreate(&stream));
    blocksnthreadse.clear();
    blocksnthreadsez.clear();
    blocksnthreadsf.clear();
    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the
    // maximum occupancy for a full device launch
    if(nint>0) {
        hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize,
                                                       BranchingPositionCosineenergy, blockToSmemFB, 0);
        blocksnthreadse.push_back((nint + blockSize - 1) / blockSize);
        blocksnthreadse.push_back(blockSize);
        blockSize = 0;

        hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize,
                                                       BranchingPositionCosineenergyz, blockToSmemFB2, 0);
        blocksnthreadsez.push_back((nint + blockSize - 1) / blockSize);
        blocksnthreadsez.push_back(blockSize);
        blockSize = 0;

        hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize,
                                                       BranchingPositionCosineforces, blockToSmemFB, 0);
        blocksnthreadsf.push_back((nint + blockSize - 1) / blockSize);
        blocksnthreadsf.push_back(blockSize);
        //get addition vars
        bntaddvec2.clear();
        bntaddvec2 = getaddred2bnt(nint);
        CUDAcommon::handleerror(hipMalloc((void **) &gU_i, bntaddvec2.at(0)*sizeof(floatingpoint)));
        CUDAcommon::handleerror(hipMemset(gU_i, 0, bntaddvec2.at(0) * sizeof(floatingpoint)));
//        CUDAcommon::handleerror(hipMalloc((void **) &gU_i, nint*sizeof(floatingpoint)));
        CUDAcommon::handleerror(hipMalloc((void **) &gU_sum, sizeof(floatingpoint)));

//        CUDAcommon::handleerror(hipMalloc((void **) &gU_i, nint*sizeof(floatingpoint)));
//        CUDAcommon::handleerror(hipMalloc((void **) &gU_sum, sizeof(floatingpoint)));

        char a[] = "BranchingFF";
        char b[] = "Branching Position Cosine";
        CUDAcommon::handleerror(hipMalloc((void **) &gFF, 100 * sizeof(char)));
        CUDAcommon::handleerror(hipMalloc((void **) &ginteraction, 100 * sizeof(char)));
        CUDAcommon::handleerror(hipMemcpy(gFF, a, 100 * sizeof(char), hipMemcpyHostToDevice));
        CUDAcommon::handleerror(hipMemcpy(ginteraction, b, 100 * sizeof(char), hipMemcpyHostToDevice));
    }
    else{
        blocksnthreadse.push_back(0);
        blocksnthreadse.push_back(0);
        blocksnthreadsez.push_back(0);
        blocksnthreadsez.push_back(0);
        blocksnthreadsf.push_back(0);
        blocksnthreadsf.push_back(0);
    }

}
floatingpoint* BranchingPositionCosine::energy(floatingpoint *coord, floatingpoint *f, int *beadSet,
                                        floatingpoint *kpos, floatingpoint *pos, int *params) {
//    if(blocksnthreadse[1]>0) {
//        BranchingPositionCosineenergy<<<blocksnthreadse[0], blocksnthreadse[1], (9 * blocksnthreadse[1]) * sizeof
//                (floatingpoint), stream>>> (coord, f, beadSet, kpos, pos, params, gU_i, CUDAcommon::getCUDAvars().gculpritID,
//                CUDAcommon::getCUDAvars().gculpritFF,
//                CUDAcommon::getCUDAvars().gculpritinteraction, gFF, ginteraction);
//        auto cvars = CUDAcommon::getCUDAvars();
//        cvars.streamvec.push_back(&stream);
//        CUDAcommon::cudavars = cvars;
//        CUDAcommon::handleerror( hipGetLastError() ,"BranchingPositionCosineenergy", "BranchingPositionCosine.cu");
//        floatingpoint* gpu_Utot = CUDAcommon::getCUDAvars().gpu_energy;
//        addvector<<<1,1,0,stream>>>(gU_i,params, gU_sum, gpu_Utot);
//        CUDAcommon::handleerror( hipGetLastError() ,"BranchingPositionCosineenergy", "BranchingPositionCosine.cu");
//        return gU_sum;}
//    else
//        return NULL;
}


floatingpoint* BranchingPositionCosine::energy(floatingpoint *coord, floatingpoint *f, int *beadSet,
                                        floatingpoint *kpos, floatingpoint *pos, floatingpoint *z, int *params) {
    if(blocksnthreadse[1]>0) {
        BranchingPositionCosineenergy<<<blocksnthreadse[0], blocksnthreadse[1], (9 * blocksnthreadse[1]) * sizeof
                (floatingpoint), stream>>> (coord, f, beadSet, kpos, pos, params, gU_i, z, CUDAcommon::getCUDAvars()
                .gculpritID,
                CUDAcommon::getCUDAvars().gculpritFF,
                CUDAcommon::getCUDAvars().gculpritinteraction, gFF, ginteraction);
//        auto cvars = CUDAcommon::getCUDAvars();
//        cvars.streamvec.push_back(&stream);
//        CUDAcommon::cudavars = cvars;
//        CUDAcommon::handleerror( hipGetLastError() ,"BranchingPositionCosineenergy", "BranchingPositionCosine.cu");
//        floatingpoint* gpu_Utot = CUDAcommon::getCUDAvars().gpu_energy;
//        addvector<<<1,1,0,stream>>>(gU_i,params, gU_sum, gpu_Utot);
//        CUDAcommon::handleerror( hipGetLastError() ,"BranchingPositionCosineenergy", "BranchingPositionCosine.cu");
//        return gU_sum;
    }

    if(blocksnthreadsez[1]>0) {
        BranchingPositionCosineenergyz << < blocksnthreadsez[0], blocksnthreadsez[1], (18 * blocksnthreadsez[1]) *
                                          sizeof(floatingpoint), stream>> > (coord, f, beadSet, kpos, pos, params, gU_i, z,
                CUDAcommon::getCUDAvars().gculpritID,
                CUDAcommon::getCUDAvars().gculpritFF,
                CUDAcommon::getCUDAvars().gculpritinteraction, gFF, ginteraction );
//        auto cvars = CUDAcommon::getCUDAvars();
//        cvars.streamvec.push_back(&stream);
//        CUDAcommon::cudavars = cvars;
//        CUDAcommon::handleerror(hipGetLastError(),"BranchingPositionCosineenergyz", "BranchingPositionCosine.cu");
//        floatingpoint* gpu_Utot = CUDAcommon::getCUDAvars().gpu_energy;
//
//        addvector<<<1,1,0,stream>>>(gU_i,params, gU_sum, gpu_Utot);
//        CUDAcommon::handleerror(hipGetLastError(),"BranchingPositionCosineenergyz", "BranchingPositionCosine.cu");
//        return gU_sum;
    }
    if(blocksnthreadse[1]<=0 && blocksnthreadsez[1]<=0)
        return NULL;
    else{
        auto cvars = CUDAcommon::getCUDAvars();
        cvars.streamvec.push_back(&stream);
        CUDAcommon::cudavars = cvars;
        floatingpoint* gpu_Utot = CUDAcommon::getCUDAvars().gpu_energy;

//        addvector<<<1,1,0,stream>>>(gU_i,params, gU_sum, gpu_Utot);
//        hipStreamSynchronize(stream);
//        addvectorred<<<1,200,200*sizeof(floatingpoint),stream>>>(gU_i,params, gU_sum, gpu_Utot);
//        hipStreamSynchronize(stream);
//        std::cout<<"bntaddvec "<<bntaddvec2.at(0)<<" "<<bntaddvec2.at(1)<<" "<<bntaddvec2.at(0)<<" "
//                ""<<bntaddvec2.at(2)<<" "<<bntaddvec2.at(3)<<endl;
        resetfloatingpointvariableCUDA<<<1,1,0,stream>>>(gU_sum);
        addvectorred2<<<bntaddvec2.at(2),bntaddvec2.at(3), bntaddvec2.at(3) * sizeof(floatingpoint),stream>>>(gU_i,
                params, gU_sum, gpu_Utot);
//        CUDAcommon::handleerror(hipDeviceSynchronize(),"FilamentBendingCosineenergyz", "FilamentBendingCosine.cu");
        CUDAcommon::handleerror(hipGetLastError(),"FilamentBendingCosineenergyz", "FilamentBendingCosine.cu");
        return gU_sum;
    }

}

void BranchingPositionCosine::forces(floatingpoint *coord, floatingpoint *f, int *beadSet,
                                     floatingpoint *kpos, floatingpoint *pos, int *params){
    if(blocksnthreadsf[1]>0) {
        BranchingPositionCosineforces << < blocksnthreadsf[0], blocksnthreadsf[1], (9 * blocksnthreadsf[1]) *
                                                                                   sizeof(floatingpoint), stream >> > (coord, f, beadSet, kpos, pos, params);
        auto cvars = CUDAcommon::getCUDAvars();
        cvars.streamvec.push_back(&stream);
        CUDAcommon::cudavars = cvars;
        CUDAcommon::handleerror(hipGetLastError(),"BranchingPositionCosineforces", "BranchingPositionCosine.cu");
    }
}
void BranchingPositionCosine::checkforculprit() {
    CUDAcommon::printculprit("BranchingPosition","BranchingPositionCosine");
    BranchingPoint* br;
    br = (BranchingPoint::getBranchingPoints()[CUDAcommon::getCUDAvars().culpritID[0]]);
    cout<<"Printing culprit branching point information."<<endl;
    br->printSelf();
    exit(EXIT_FAILURE);
}
#endif

floatingpoint BranchingPositionCosine::energy(floatingpoint *coord, totalforcefloatingpoint *f, int *beadSet,
                                       floatingpoint *kpos, floatingpoint *pos){


    int n = BranchingPosition<BranchingPositionCosine>::n;
    int nint = BranchingPoint::getBranchingPoints().size();

    floatingpoint *coord1, *coord2, *coord3, X, D, XD, xd, theta, posheta, dTheta, U_i;
    floatingpoint *mp = new floatingpoint[3];

    totalenergyfloatingpoint U = 0.0;

    for(int i = 0; i < nint; i += 1) {

        coord1 = &coord[3 * beadSet[n * i]];
        coord2 = &coord[3 * beadSet[n * i + 1]];
        coord3 = &coord[3 * beadSet[n * i + 2]];

        midPointCoordinate(mp, coord1, coord2, pos[i]);
        X = sqrt(scalarProduct(mp, coord2, mp, coord2));
        D = sqrt(scalarProduct(mp, coord3, mp, coord3));

        XD = X * D;

        xd = scalarProduct(mp, coord2, mp, coord3);

        theta = safeacos(xd / XD);
        posheta = 0.5*M_PI;
        dTheta = theta-posheta;

        U_i = kpos[i] * ( 1 - cos(dTheta) );


        if(fabs(U_i) == numeric_limits<floatingpoint>::infinity()
           || U_i != U_i || U_i < -1.0) {

            //set culprit and return
            BranchingInteractions::_branchingCulprit = BranchingPoint::getBranchingPoints()[i];

            return -1;
        }

        U += U_i;
    }
    delete mp;
    return U;
}

floatingpoint BranchingPositionCosine::energy(floatingpoint *coord, totalforcefloatingpoint *f, int *beadSet,
                                       floatingpoint *kpos, floatingpoint *pos, floatingpoint d){

    int n = BranchingPosition<BranchingPositionCosine>::n;
    int nint = BranchingPoint::getBranchingPoints().size();

    floatingpoint *coord1, *coord2, *coord3, X, D, XD, xd, theta, posheta, dTheta, U_i;
    totalforcefloatingpoint *f1, *f2, *f3;
    floatingpoint *mp = new floatingpoint[3];
    floatingpoint *vzero = new floatingpoint[3]; vzero[0] = 0.0; vzero[1] = 0.0; vzero[2] = 0.0;

    totalenergyfloatingpoint U = 0.0;

    for(int i = 0; i < nint; i += 1) {

        coord1 = &coord[3 * beadSet[n * i]];
        coord2 = &coord[3 * beadSet[n * i + 1]];
        coord3 = &coord[3 * beadSet[n * i + 2]];
        f1 = &f[3 * beadSet[n * i]];
        f2 = &f[3 * beadSet[n * i + 1]];
        f3 = &f[3 * beadSet[n * i + 2]];

        midPointCoordinateStretched(mp, coord1, f1, coord2, f2, pos[i], d);
        X = sqrt(scalarProductStretched(mp, vzero, coord2, f2, mp, vzero, coord2, f2, d));
        D = sqrt(scalarProductStretched(mp, vzero, coord3, f3, mp, vzero, coord3, f3, d));

        XD = X * D;

        xd = scalarProductStretched(mp, vzero, coord2, f2, mp, vzero, coord3, f3, d);

        theta = safeacos(xd / XD);
        posheta = 0.5*M_PI;
        dTheta = theta-posheta;

        U_i = kpos[i] * ( 1 - cos(dTheta) );
//    std::cout<<i << U_i<<endl;

        if(fabs(U_i) == numeric_limits<floatingpoint>::infinity()
           || U_i != U_i || U_i < -1.0) {

            //set culprit and return
            BranchingInteractions::_branchingCulprit = BranchingPoint::getBranchingPoints()[i];

            return -1;
        }

        U += U_i;
    }
    delete mp;
    delete vzero;
    return U;
}

void BranchingPositionCosine::forces(floatingpoint *coord, totalforcefloatingpoint *f, int *beadSet,
                                     floatingpoint *kpos, floatingpoint *pos){

    int n = BranchingPosition<BranchingPositionCosine>::n;
    int nint = BranchingPoint::getBranchingPoints().size();

    floatingpoint *coord1, *coord2, *coord3, X, D, XD, xd, invX, invD, position, A, B, C, k, theta, posheta, dTheta;
	totalforcefloatingpoint  *f1, *f2, *f3;
    floatingpoint *mp = new floatingpoint[3];


    for(int i = 0; i < nint; i += 1) {

        coord1 = &coord[3 * beadSet[n * i]];
        coord2 = &coord[3 * beadSet[n * i + 1]];
        coord3 = &coord[3 * beadSet[n * i + 2]];
        f1 = &f[3 * beadSet[n * i]];
        f2 = &f[3 * beadSet[n * i + 1]];
        f3 = &f[3 * beadSet[n * i + 2]];

        midPointCoordinate(mp, coord1, coord2, pos[i]);
        X = sqrt(scalarProduct(mp, coord2, mp, coord2));
        D = sqrt(scalarProduct(mp, coord3, mp, coord3));

        XD = X * D;
        xd = scalarProduct(mp, coord2, mp, coord3);
//        std::cout<<xd<<" "<<scalarProduct(mp, coord2, mp, coord3)<<" "<<mp[0]<<" "<<mp[1]<<" "<<mp[2]<<" "
//                ""<<coord2[0]<<" "
//                ""<<coord2[1]<<" "<<coord2[2]<<" "
//                ""<<coord3[0]<<" "
//                ""<<coord3[1]<<" "<<coord3[2]<<" "<<endl;
        invX = 1/X;
        invD = 1/D;
        A = invX*invD;
        B = invX*invX;
        C = invD*invD;

	    if(abs(xd/XD - 1.0)<0.01){
	        cout<<"isequal "<<xd/XD<<endl;
		    xd = 0.99*XD;
	    }

        theta = safeacos(xd / XD);
        posheta = 0.5*M_PI;
        dTheta = theta-posheta;

        position = pos[i];

        k =  kpos[i] * A * sin(dTheta)/sin(theta);


        if(isnan(theta)||isinf(theta)||isnan(position)||isinf(theta)||isnan(k)||isinf(k)){
            cout<<"Culprit Branching Position Cosine "<<endl;
            cout<<"theta "<<theta<<" position "<<position<<" theta "<<theta<<" k "<<k<<endl;
        }

        //bead 1
        f1[0] +=  k * (1-position)* (- (1-position)*(coord2[0] - coord1[0]) - (coord3[0] - (1-position)*coord1[0] - position*coord2[0])
                                     + xd *(B*(1-position)*(coord2[0] - coord1[0]) + C*(coord3[0] - (1-position)*coord1[0] - position*coord2[0])) );

        f1[1] +=  k * (1-position)* (- (1-position)*(coord2[1] - coord1[1]) - (coord3[1] - (1-position)*coord1[1] - position*coord2[1])
                                     + xd *(B*(1-position)*(coord2[1] - coord1[1]) + C*(coord3[1] - (1-position)*coord1[1] - position*coord2[1])) );

        f1[2] +=  k * (1-position)* (- (1-position)*(coord2[2] - coord1[2]) - (coord3[2] - (1-position)*coord1[2] - position*coord2[2])
                                     + xd *(B*(1-position)*(coord2[2] - coord1[2]) + C*(coord3[2] - (1-position)*coord1[2] - position*coord2[2])) );

        //bead 2

        f2[0] +=  k * (- position*(1-position)*(coord2[0] - coord1[0]) + (1-position)*(coord3[0]- (1-position)*coord1[0] - position*coord2[0])
                       + xd *( (1-position)*B*(1-position)*(coord2[0] - coord1[0]) - position*C*(coord3[0] - (1-position)*coord1[0] - position*coord2[0])) );

        f2[1] +=  k * (- position*(1-position)*(coord2[1] - coord1[1]) + (1-position)*(coord3[1]- (1-position)*coord1[1] - position*coord2[1])
                       + xd *( (1-position)*B*(1-position)*(coord2[1] - coord1[1]) - position*C*(coord3[1] - (1-position)*coord1[1] - position*coord2[1])) );

        f2[2] +=  k * (- position*(1-position)*(coord2[2] - coord1[2]) + (1-position)*(coord3[2]- (1-position)*coord1[2] - position*coord2[2])
                       + xd *( (1-position)*B*(1-position)*(coord2[2] - coord1[2]) - position*C*(coord3[2] - (1-position)*coord1[2] - position*coord2[2])) );

        //bead3

        f3[0] +=  k * ( (1-position)*(coord2[0] - coord1[0]) - xd * C*(coord3[0] - (1-position)*coord1[0] - position*coord2[0]) );
        f3[1] +=  k * ( (1-position)*(coord2[1] - coord1[1]) - xd * C*(coord3[1] - (1-position)*coord1[1] - position*coord2[1]) );
        f3[2] +=  k * ( (1-position)*(coord2[2] - coord1[2]) - xd * C*(coord3[2] - (1-position)*coord1[2] - position*coord2[2]) );

        if(isnan(f1[0])||isinf(f1[0])||isnan(f1[1])||isinf(f1[1])||isnan(f1[2])||isinf(f1[2])
           ||isnan(f2[0])||isinf(f2[0])||isnan(f2[1])||isinf(f2[1])||isnan(f2[2])||isinf(f2[2])
           ||isnan(f3[0])||isinf(f3[0])||isnan(f3[1])||isinf(f3[1])||isnan(f3[2])||isinf(f3[2])) {
            cout << "Culprit is BranchingPositionCosine" << endl;
            cout<<"theta "<<theta<<" position "<<position<<" theta "<<theta<<" k "
            <<k<<"xd " <<xd<<" XD "<<XD<<endl;
            cout<<"forces "<<f1[0]<<" "<<f1[1]<<" "<<f1[2]<<" "<<f2[0]<<" "<<f2[1]<<" "
                <<f2[2]<<" "<<f3[0]<<" "<<f3[1]<<" "<<f3[2]<<endl;
            cout<<"coord "<<coord1[0]<<" "<<coord1[1]<<" "<<coord1[2]<<" "
                    <<coord2[0]<<" "<<coord2[1]<<" "<<coord2[2]<<" "
                    <<coord3[0]<<" "<<coord3[1]<<" "<<coord3[2]<<endl;
        }
    }
    delete mp;
}
