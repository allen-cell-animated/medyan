#include "hip/hip_runtime.h"

//------------------------------------------------------------------
//  **MEDYAN** - Simulation Package for the Mechanochemical
//               Dynamics of Active Networks, v3.1
//
//  Copyright (2015-2016)  Papoian Lab, University of Maryland
//
//                 ALL RIGHTS RESERVED
//
//  See the MEDYAN web page for more information:
//  http://www.medyan.org
//------------------------------------------------------------------

#include <src/Mechanics/Minimizer/CGMethod.h>
#include "BranchingStretching.h"

#include "BranchingStretchingHarmonic.h"

#include "BranchingPoint.h"
#include "Cylinder.h"
#include "Bead.h"
#include "cross_check.h"
#include "nvToolsExt.h"

template <class BStretchingInteractionType>
void BranchingStretching<BStretchingInteractionType>::vectorize() {

    beadSet = new int[n * BranchingPoint::getBranchingPoints().size()];
    kstr = new double[BranchingPoint::getBranchingPoints().size()];
    eql = new double[BranchingPoint::getBranchingPoints().size()];
    pos = new double[BranchingPoint::getBranchingPoints().size()];


    int i = 0;

    for (auto b: BranchingPoint::getBranchingPoints()) {

        beadSet[n * i] = b->getFirstCylinder()->getFirstBead()->_dbIndex;
        beadSet[n * i + 1] = b->getFirstCylinder()->getSecondBead()->_dbIndex;
        beadSet[n * i + 2] = b->getSecondCylinder()->getFirstBead()->_dbIndex;

        kstr[i] = b->getMBranchingPoint()->getStretchingConstant();
        eql[i] = b->getMBranchingPoint()->getEqLength();
        pos[i] = b->getPosition();

        i++;
    }
    //CUDA
#ifdef CUDAACCL
    F_i = new double[CGMethod::N];
    nvtxRangePushA("CVFF");

    int numInteractions = BranchingPoint::getBranchingPoints().size();
    _FFType.optimalblocksnthreads(numInteractions);

    CUDAcommon::handleerror(hipMalloc((void **) &gpu_beadSet, n * numInteractions * sizeof(int)));
    CUDAcommon::handleerror(hipMemcpy(gpu_beadSet, beadSet, n * numInteractions * sizeof(int),
                                       hipMemcpyHostToDevice));

    CUDAcommon::handleerror(hipMalloc((void **) &gpu_kstr, numInteractions * sizeof(double)));
    CUDAcommon::handleerror(hipMemcpy(gpu_kstr, kstr, numInteractions * sizeof(double), hipMemcpyHostToDevice));

    CUDAcommon::handleerror(hipMalloc((void **) &gpu_eql, numInteractions * sizeof(double)));
    CUDAcommon::handleerror(hipMemcpy(gpu_eql, eql, numInteractions * sizeof(double), hipMemcpyHostToDevice));

    CUDAcommon::handleerror(hipMalloc((void **) &gpu_pos, numInteractions * sizeof(double)));
    CUDAcommon::handleerror(hipMemcpy(gpu_pos, pos, numInteractions * sizeof(double), hipMemcpyHostToDevice));

    vector<int> params;
    params.push_back(int(n));
    params.push_back(numInteractions);
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_params, 2 * sizeof(int)));
    CUDAcommon::handleerror(hipMemcpy(gpu_params, params.data(), 2 * sizeof(int), hipMemcpyHostToDevice));
    nvtxRangePop();
#endif

    //
}

template<class BStretchingInteractionType>
void BranchingStretching<BStretchingInteractionType>::deallocate() {

    delete beadSet;
    delete kstr;
    delete eql;
    delete pos;
#ifdef CUDAACCL
    _FFType.deallocate();
    CUDAcommon::handleerror(hipFree(gpu_beadSet));
    CUDAcommon::handleerror(hipFree(gpu_kstr));
    CUDAcommon::handleerror(hipFree(gpu_eql));
    CUDAcommon::handleerror(hipFree(gpu_pos));
    CUDAcommon::handleerror(hipFree(gpu_params));
#endif
}


template <class BStretchingInteractionType>
double BranchingStretching<BStretchingInteractionType>::computeEnergy(double *coord, double *f, double d) {


    double U_i[1], U_ii;
    double* gU_i;
    U_ii = NULL;
#ifdef CUDAACCL
    //has to be changed to accomodate aux force
    double * gpu_coord=CUDAcommon::getCUDAvars().gpu_coord;
    double * gpu_force=CUDAcommon::getCUDAvars().gpu_force;
    double * gpu_d = CUDAcommon::getCUDAvars().gpu_lambda;
    nvtxRangePushA("CCEBS");

    if(d == 0.0){
        gU_i=_FFType.energy(gpu_coord, gpu_force, gpu_beadSet, gpu_kstr, gpu_eql, gpu_pos, gpu_params);

    }
    else{
        gU_i=_FFType.energy(gpu_coord, gpu_force, gpu_beadSet, gpu_kstr, gpu_eql, gpu_pos, gpu_d,
                            gpu_params);
    }
    nvtxRangePop();
#endif
    nvtxRangePushA("SCEBS");
    if (d == 0.0)
        U_ii = _FFType.energy(coord, f, beadSet, kstr, eql, pos);
    else
        U_ii = _FFType.energy(coord, f, beadSet, kstr, eql, pos, d);
    nvtxRangePop();
    if(gU_i!=NULL) {

        CUDAcommon::handleerror(hipMemcpy(U_i, gU_i, sizeof(double), hipMemcpyDeviceToHost),"computeEnergy",
                                "BranchingStretching.cu");
    }
    else
        U_i[0] = 0.0;
    if(fabs(U_ii)>1000000.0) {
        if (fabs((U_ii - U_i[0]) / U_ii) > 0.0001){
            std::cout<<endl;
            std::cout << "CUDA BSE " << U_i[0] << endl;
            std::cout << "Vectorized BSE " << U_ii << endl;
            std::cout << "Precision match error" << fabs(U_ii - U_i[0]) << endl;
        }
    }
    else {
        if (fabs(U_ii - U_i[0]) > 1.0 / 100000000.0){
            std::cout<<endl;
            std::cout << "CUDA BSE " << U_i[0] << endl;
            std::cout << "Vectorized BSE " << U_ii << endl;
            std::cout << "Precision match " << fabs(U_ii - U_i[0]) << endl;
//        exit(EXIT_FAILURE);
        }
    }

    return U_ii;
}

template <class BStretchingInteractionType>
void BranchingStretching<BStretchingInteractionType>::computeForces(double *coord, double *f) {
#ifdef CUDAACCL
    //has to be changed to accomodate aux force
    double * gpu_coord=CUDAcommon::getCUDAvars().gpu_coord;

    double * gpu_force;


    if(cross_checkclass::Aux){
        nvtxRangePushA("CCFBS");

        gpu_force=CUDAcommon::getCUDAvars().gpu_forceAux;
        _FFType.forces(gpu_coord, gpu_force, gpu_beadSet, gpu_kstr, gpu_eql, gpu_pos, gpu_params);
        nvtxRangePop();
    }
    else {
        nvtxRangePushA("CCFBS");

        gpu_force = CUDAcommon::getCUDAvars().gpu_force;
        _FFType.forces(gpu_coord, gpu_force, gpu_beadSet, gpu_kstr, gpu_eql, gpu_pos, gpu_params);
        nvtxRangePop();
    }
    //TODO remove this later need not copy forces back to CPU.
    CUDAcommon::handleerror(hipMemcpy(F_i, gpu_force, CGMethod::N*sizeof(double),
                                       hipMemcpyDeviceToHost));
#endif
    nvtxRangePushA("SCFBS");

    _FFType.forces(coord, f, beadSet, kstr, eql, pos);
    nvtxRangePop();
#ifdef CUDAACCL

    bool state = false;
    for(auto iter=0;iter<Bead::getBeads().size();iter++) {
        if (fabs(F_i[3 * iter] - f[3 * iter]) <=1.0/100000000.0 && fabs(F_i[3 * iter + 1] - f[3 * iter + 1])
                                                                   <=1.0/100000000.0 && fabs(F_i[3 * iter + 2] - f[3 * iter + 2]) <=1.0/100000000.0)
        {state = true;}
        else {
            state = false;
            std::cout<<endl;
            std::cout<<"BS Forces"<<endl;
            std::cout<<"Precision match "<<fabs(F_i[3 * iter] - f[3 * iter])<<" "<<fabs(F_i[3 * iter + 1] - f[3 *
                                                                                                              iter + 1])<<" "<<fabs(F_i[3 * iter + 2] - f[3 * iter + 2])<<endl;
            std::cout << "CUDA       " << F_i[3 * iter] << " " << F_i[3 * iter + 1] << " " << F_i[3 * iter + 2] << endl;
            std::cout << "Vectorized " << f[3 * iter] << " " << f[3 * iter + 1] << " " << f[3 * iter + 2] << endl;
//        exit(EXIT_FAILURE);
        }
    }
#endif
}



///Template specializations
template double
BranchingStretching<BranchingStretchingHarmonic>::computeEnergy(double *coord, double *f, double d);
template void BranchingStretching<BranchingStretchingHarmonic>::computeForces(double *coord, double *f);
template void BranchingStretching<BranchingStretchingHarmonic>::vectorize();
template void BranchingStretching<BranchingStretchingHarmonic>::deallocate();
