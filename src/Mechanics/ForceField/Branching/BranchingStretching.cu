
//------------------------------------------------------------------
//  **MEDYAN** - Simulation Package for the Mechanochemical
//               Dynamics of Active Networks, v3.1
//
//  Copyright (2015-2016)  Papoian Lab, University of Maryland
//
//                 ALL RIGHTS RESERVED
//
//  See the MEDYAN web page for more information:
//  http://www.medyan.org
//------------------------------------------------------------------

#include <src/Mechanics/Minimizer/CGMethod.h>
#include "BranchingStretching.h"

#include "BranchingStretchingHarmonic.h"

#include "BranchingPoint.h"
#include "Cylinder.h"
#include "Bead.h"
#include "cross_check.h"
#include "nvToolsExt.h"

template <class BStretchingInteractionType>
void BranchingStretching<BStretchingInteractionType>::vectorize() {

    beadSet = new int[n * BranchingPoint::getBranchingPoints().size()];
    kstr = new double[BranchingPoint::getBranchingPoints().size()];
    eql = new double[BranchingPoint::getBranchingPoints().size()];
    pos = new double[BranchingPoint::getBranchingPoints().size()];


    int i = 0;

    for (auto b: BranchingPoint::getBranchingPoints()) {

        beadSet[n * i] = b->getFirstCylinder()->getFirstBead()->_dbIndex;
        beadSet[n * i + 1] = b->getFirstCylinder()->getSecondBead()->_dbIndex;
        beadSet[n * i + 2] = b->getSecondCylinder()->getFirstBead()->_dbIndex;

        kstr[i] = b->getMBranchingPoint()->getStretchingConstant();
        eql[i] = b->getMBranchingPoint()->getEqLength();
        pos[i] = b->getPosition();

        i++;
    }
    //CUDA
#ifdef CUDAACCL
//    F_i = new double[CGMethod::N];
    nvtxRangePushA("CVFF");

    int numInteractions = BranchingPoint::getBranchingPoints().size();
    _FFType.optimalblocksnthreads(numInteractions);

    CUDAcommon::handleerror(hipMalloc((void **) &gpu_beadSet, n * numInteractions * sizeof(int)));
    CUDAcommon::handleerror(hipMemcpy(gpu_beadSet, beadSet, n * numInteractions * sizeof(int),
                                       hipMemcpyHostToDevice));

    CUDAcommon::handleerror(hipMalloc((void **) &gpu_kstr, numInteractions * sizeof(double)));
    CUDAcommon::handleerror(hipMemcpy(gpu_kstr, kstr, numInteractions * sizeof(double), hipMemcpyHostToDevice));

    CUDAcommon::handleerror(hipMalloc((void **) &gpu_eql, numInteractions * sizeof(double)));
    CUDAcommon::handleerror(hipMemcpy(gpu_eql, eql, numInteractions * sizeof(double), hipMemcpyHostToDevice));

    CUDAcommon::handleerror(hipMalloc((void **) &gpu_pos, numInteractions * sizeof(double)));
    CUDAcommon::handleerror(hipMemcpy(gpu_pos, pos, numInteractions * sizeof(double), hipMemcpyHostToDevice));

    vector<int> params;
    params.push_back(int(n));
    params.push_back(numInteractions);
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_params, 2 * sizeof(int)));
    CUDAcommon::handleerror(hipMemcpy(gpu_params, params.data(), 2 * sizeof(int), hipMemcpyHostToDevice));
    nvtxRangePop();
#endif

    //
}

template<class BStretchingInteractionType>
void BranchingStretching<BStretchingInteractionType>::deallocate() {

    delete beadSet;
    delete kstr;
    delete eql;
    delete pos;
#ifdef CUDAACCL
    _FFType.deallocate();
    CUDAcommon::handleerror(hipFree(gpu_beadSet));
    CUDAcommon::handleerror(hipFree(gpu_kstr));
    CUDAcommon::handleerror(hipFree(gpu_eql));
    CUDAcommon::handleerror(hipFree(gpu_pos));
    CUDAcommon::handleerror(hipFree(gpu_params));
#endif
}


template <class BStretchingInteractionType>
double BranchingStretching<BStretchingInteractionType>::computeEnergy(double *coord, double *f, double d) {


    double U_i[1], U_ii;
    double* gU_i;
    U_ii = NULL;
#ifdef CUDAACCL
    //has to be changed to accomodate aux force
    double * gpu_coord=CUDAcommon::getCUDAvars().gpu_coord;
    double * gpu_force=CUDAcommon::getCUDAvars().gpu_force;
    double * gpu_d = CUDAcommon::getCUDAvars().gpu_lambda;
    nvtxRangePushA("CCEBS");

//    if(d == 0.0){
//        gU_i=_FFType.energy(gpu_coord, gpu_force, gpu_beadSet, gpu_kstr, gpu_eql, gpu_pos, gpu_params);
//
//    }
//    else{
        gU_i=_FFType.energy(gpu_coord, gpu_force, gpu_beadSet, gpu_kstr, gpu_eql, gpu_pos, gpu_d,
                            gpu_params);
//    }
    nvtxRangePop();
#else
    nvtxRangePushA("SCEBS");
    if (d == 0.0)
        U_ii = _FFType.energy(coord, f, beadSet, kstr, eql, pos);
    else
        U_ii = _FFType.energy(coord, f, beadSet, kstr, eql, pos, d);
    nvtxRangePop();
#endif
    return U_ii;
}

template <class BStretchingInteractionType>
void BranchingStretching<BStretchingInteractionType>::computeForces(double *coord, double *f) {
#ifdef CUDAACCL
    //has to be changed to accomodate aux force
    double * gpu_coord=CUDAcommon::getCUDAvars().gpu_coord;

    double * gpu_force;
    if(cross_checkclass::Aux){
        nvtxRangePushA("CCFBS");

        gpu_force=CUDAcommon::getCUDAvars().gpu_forceAux;
        _FFType.forces(gpu_coord, gpu_force, gpu_beadSet, gpu_kstr, gpu_eql, gpu_pos, gpu_params);
        nvtxRangePop();
    }
    else {
        nvtxRangePushA("CCFBS");

        gpu_force = CUDAcommon::getCUDAvars().gpu_force;
        _FFType.forces(gpu_coord, gpu_force, gpu_beadSet, gpu_kstr, gpu_eql, gpu_pos, gpu_params);
        nvtxRangePop();
    }
#else
    nvtxRangePushA("SCFBS");

    _FFType.forces(coord, f, beadSet, kstr, eql, pos);
    nvtxRangePop();
#endif
}



///Template specializations
template double
BranchingStretching<BranchingStretchingHarmonic>::computeEnergy(double *coord, double *f, double d);
template void BranchingStretching<BranchingStretchingHarmonic>::computeForces(double *coord, double *f);
template void BranchingStretching<BranchingStretchingHarmonic>::vectorize();
template void BranchingStretching<BranchingStretchingHarmonic>::deallocate();
