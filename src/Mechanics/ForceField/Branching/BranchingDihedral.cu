#include "hip/hip_runtime.h"

//------------------------------------------------------------------
//  **MEDYAN** - Simulation Package for the Mechanochemical
//               Dynamics of Active Networks, v3.1
//
//  Copyright (2015-2016)  Papoian Lab, University of Maryland
//
//                 ALL RIGHTS RESERVED
//
//  See the MEDYAN web page for more information:
//  http://www.medyan.org
//------------------------------------------------------------------

#include "BranchingDihedral.h"

#include "BranchingDihedralCosine.h"

#include "BranchingPoint.h"
#include "Cylinder.h"
#include "Bead.h"
#include "cross_check.h"
#include "nvToolsExt.h"

template <class BDihedralInteractionType>
void BranchingDihedral<BDihedralInteractionType>::vectorize() {

    beadSet = new int[n * BranchingPoint::getBranchingPoints().size()];
    kdih = new double[BranchingPoint::getBranchingPoints().size()];
    pos = new double[BranchingPoint::getBranchingPoints().size()];

    int i = 0;

    for (auto b: BranchingPoint::getBranchingPoints()) {

        beadSet[n * i] = b->getFirstCylinder()->getFirstBead()->_dbIndex;
        beadSet[n * i + 1] = b->getFirstCylinder()->getSecondBead()->_dbIndex;
        beadSet[n * i + 2] = b->getSecondCylinder()->getFirstBead()->_dbIndex;
        beadSet[n * i + 3] = b->getSecondCylinder()->getSecondBead()->_dbIndex;

        kdih[i] = b->getMBranchingPoint()->getDihedralConstant();
        pos[i] = b->getPosition();

        i++;
    }
    //CUDA
#ifdef CUDAACCL
//    hipEvent_t start, stop;
//    CUDAcommon::handleerror(hipEventCreate( &start));
//    CUDAcommon::handleerror(hipEventCreate( &stop));
//    CUDAcommon::handleerror(hipEventRecord( start, 0));
    nvtxRangePushA("CVFF");

    int numInteractions =BranchingPoint::getBranchingPoints().size();
    _FFType.optimalblocksnthreads(numInteractions);
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_beadSet, n * numInteractions * sizeof(int)));
    CUDAcommon::handleerror(hipMemcpy(gpu_beadSet, beadSet, n * numInteractions * sizeof(int),
                                       hipMemcpyHostToDevice));

    CUDAcommon::handleerror(hipMalloc((void **) &gpu_kdih, numInteractions * sizeof(double)));
    CUDAcommon::handleerror(hipMemcpy(gpu_kdih, kdih, numInteractions * sizeof(double), hipMemcpyHostToDevice));

    CUDAcommon::handleerror(hipMalloc((void **) &gpu_pos, numInteractions * sizeof(double)));
    CUDAcommon::handleerror(hipMemcpy(gpu_pos, pos, numInteractions * sizeof(double), hipMemcpyHostToDevice));

    vector<int> params;
    params.push_back(int(n));
    params.push_back(numInteractions);

    CUDAcommon::handleerror(hipMalloc((void **) &gpu_params, 2 * sizeof(int)));
    CUDAcommon::handleerror(hipMemcpy(gpu_params, params.data(), 2 * sizeof(int), hipMemcpyHostToDevice));

    nvtxRangePop();

#endif
}

template<class BDihedralInteractionType>
void BranchingDihedral<BDihedralInteractionType>::deallocate() {

    delete beadSet;
    delete kdih;
    delete pos;
#ifdef CUDAACCL
    _FFType.deallocate();
    CUDAcommon::handleerror(hipFree(gpu_beadSet));
    CUDAcommon::handleerror(hipFree(gpu_kdih));
    CUDAcommon::handleerror(hipFree(gpu_pos));
    CUDAcommon::handleerror(hipFree(gpu_params));
#endif
}


template <class BDihedralInteractionType>
double BranchingDihedral<BDihedralInteractionType>::computeEnergy(double *coord, double *f, double d) {

    double U_i[1], U_ii;
    double* gU_i;
    U_ii = NULL;
#ifdef CUDAACCL
    //has to be changed to accomodate aux force
    double * gpu_coord=CUDAcommon::getCUDAvars().gpu_coord;
    double * gpu_force=CUDAcommon::getCUDAvars().gpu_force;
    double * gpu_d = CUDAcommon::getCUDAvars().gpu_lambda;
    nvtxRangePushA("CCEBD");

    if(d == 0.0){
        gU_i=_FFType.energy(gpu_coord, gpu_force, gpu_beadSet, gpu_kdih, gpu_pos, gpu_params);

    }
    else{
        gU_i=_FFType.energy(gpu_coord, gpu_force, gpu_beadSet, gpu_kdih, gpu_pos, gpu_d,
                            gpu_params);
    }
    nvtxRangePop();
#endif
    nvtxRangePushA("SCEBD");

    if (d == 0.0)
        U_ii = _FFType.energy(coord, f, beadSet, kdih, pos);
    else
        U_ii = _FFType.energy(coord, f, beadSet, kdih, pos, d);
    nvtxRangePop();
    if(gU_i!=NULL) {

        CUDAcommon::handleerror(hipMemcpy(U_i, gU_i, sizeof(double),
                                           hipMemcpyDeviceToHost));
    }
    else
        U_i[0] = 0.0;
    if(fabs(U_ii)>1000000.0) {
        if (fabs((U_ii - U_i[0]) / U_ii) > 0.0001){
            std::cout<<endl;
            std::cout << "CUDA BDE " << U_i[0] << endl;
            std::cout << "Vectorized BDE " << U_ii << endl;
            std::cout << "Precision match error" << fabs(U_ii - U_i[0]) << endl;
        }
    }
    else {
        if (fabs(U_ii - U_i[0]) > 1.0 / 100000000.0){
            std::cout<<endl;
            std::cout << "CUDA BDE " << U_i << endl;
            std::cout << "Vectorized BDE " << U_ii << endl;
            std::cout << "Precision match " << fabs(U_ii - U_i[0]) << endl;
        }
    }

    return U_ii;

}

template <class BDihedralInteractionType>
void BranchingDihedral<BDihedralInteractionType>::computeForces(double *coord, double *f) {
#ifdef CUDAACCL
    //has to be changed to accomodate aux force
    double * gpu_coord=CUDAcommon::getCUDAvars().gpu_coord;

    double * gpu_force;

    if(cross_checkclass::Aux){
        nvtxRangePushA("CCFBD");

        gpu_force=CUDAcommon::getCUDAvars().gpu_forceAux;
        _FFType.forces(gpu_coord, gpu_force, gpu_beadSet, gpu_kdih, gpu_pos, gpu_params);
        nvtxRangePop();
    }
    else {
        nvtxRangePushA("CCFBD");

        gpu_force = CUDAcommon::getCUDAvars().gpu_force;
        _FFType.forces(gpu_coord, gpu_force, gpu_beadSet, gpu_kdih, gpu_pos, gpu_params);
        nvtxRangePop();
    }

    //TODO remove this later need not copy forces back to CPU.
    CUDAcommon::handleerror(hipMemcpy(F_i, gpu_force, 3 * Bead::getBeads().size() *sizeof(double),
                                       hipMemcpyDeviceToHost));
#endif
    nvtxRangePushA("SCFBD");

    _FFType.forces(coord, f, beadSet, kdih, pos);
    nvtxRangePop();
#ifdef CUDAACCL
    bool state = false;
    for(auto iter=0;iter<Bead::getBeads().size();iter++) {
        if (fabs(F_i[3 * iter] - f[3 * iter]) <=1.0/100000000.0 && fabs(F_i[3 * iter + 1] - f[3 * iter + 1])
                                                                   <=1.0/100000000.0 && fabs(F_i[3 * iter + 2] - f[3 * iter + 2]) <=1.0/100000000.0)
        {state = true;}
        else {
            state = false;
            std::cout<<endl;
            std::cout<<"BD Forces"<<endl;
            std::cout << "CUDA       " << F_i[3 * iter] << " " << F_i[3 * iter + 1] << " " << F_i[3 * iter + 2] << endl;
            std::cout << "Vectorized " << f[3 * iter] << " " << f[3 * iter + 1] << " " << f[3 * iter + 2] << endl;
            std::cout<<"Precision match "<<fabs(F_i[3 * iter] - f[3 * iter])<<" "<<fabs(F_i[3 * iter + 1] - f[3 *
                                                                                                              iter + 1])<<" "<<fabs(F_i[3 * iter + 2] - f[3 * iter + 2])<<endl;
//        exit(EXIT_FAILURE);
        }
    }
#endif
}

///Template specializations
template double BranchingDihedral<BranchingDihedralCosine>::computeEnergy(double *coord, double *f, double d);
template void BranchingDihedral<BranchingDihedralCosine>::computeForces(double *coord, double *f);
template void BranchingDihedral<BranchingDihedralCosine>::vectorize();
template void BranchingDihedral<BranchingDihedralCosine>::deallocate();
