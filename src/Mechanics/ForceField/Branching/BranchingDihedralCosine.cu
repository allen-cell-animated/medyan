#include "hip/hip_runtime.h"

//------------------------------------------------------------------
//  **MEDYAN** - Simulation Package for the Mechanochemical
//               Dynamics of Active Networks, v3.1
//
//  Copyright (2015-2016)  Papoian Lab, University of Maryland
//
//                 ALL RIGHTS RESERVED
//
//  See the MEDYAN web page for more information:
//  http://www.medyan.org
//------------------------------------------------------------------

#include "BranchingDihedralCosine.h"
#include "BranchingDihedralCosineCUDA.h"
#include "BranchingDihedral.h"

#include "BranchingPoint.h"
#include "Bead.h"

#include "MathFunctions.h"
#ifdef CUDAACCL
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "nvToolsExt.h"
#endif

using namespace mathfunc;
#ifdef CUDAACCL
void BranchingDihedralCosine::deallocate(){
    if(!(CUDAcommon::getCUDAvars().conservestreams))
        CUDAcommon::handleerror(hipStreamDestroy(stream));
    CUDAcommon::handleerror(hipFree(gU_i));
    CUDAcommon::handleerror(hipFree(gU_sum));
    CUDAcommon::handleerror(hipFree(gFF));
    CUDAcommon::handleerror(hipFree(ginteraction));
}
void BranchingDihedralCosine::optimalblocksnthreads( int nint){
    //CUDA stream create
    if(stream == NULL || !(CUDAcommon::getCUDAvars().conservestreams))
        CUDAcommon::handleerror(hipStreamCreate(&stream));
    blocksnthreadse.clear();
    blocksnthreadsez.clear();
    blocksnthreadsf.clear();
    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the
    // maximum occupancy for a full device launch
//    int gridSize;    // The actual grid size needed, based on input size
//    unaryfn::argument_type blksize;
//    unaryfn::result_type result;
//    unaryfn ufn;
    if(nint>0) {
        hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize,
                                                       BranchingDihedralCosineenergy, blockToSmem, 0);
//    std::cout<<(nint +blockSize -1) / blockSize<<" "<<blockSize<<endl;
//
//    hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize,
//                                        CUDAExclVolRepulsionenergy, 0, 0);
        blocksnthreadse.push_back((nint + blockSize - 1) / blockSize);
        blocksnthreadse.push_back(blockSize);
//    std::cout<<(nint +blockSize -1) / blockSize<<" "<<blockSize<<endl;
        blockSize = 0;

        hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize,
                                                       BranchingDihedralCosineenergyz, blockToSmemez, 0);
        blocksnthreadsez.push_back((nint + blockSize - 1) / blockSize);
        blocksnthreadsez.push_back(blockSize);
        blockSize = 0;

        hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize,
                                                       BranchingDihedralCosineforces, blockToSmem, 0);
        blocksnthreadsf.push_back((nint + blockSize - 1) / blockSize);
        blocksnthreadsf.push_back(blockSize);
//get addition vars
        bntaddvec2.clear();
        bntaddvec2 = getaddred2bnt(nint);
        CUDAcommon::handleerror(hipMalloc((void **) &gU_i, bntaddvec2.at(0)*sizeof(floatingpoint)));
        CUDAcommon::handleerror(hipMemset(gU_i, 0, bntaddvec2.at(0) * sizeof(floatingpoint)));
//        CUDAcommon::handleerror(hipMalloc((void **) &gU_i, nint*sizeof(floatingpoint)));
        CUDAcommon::handleerror(hipMalloc((void **) &gU_sum, sizeof(floatingpoint)));

//        CUDAcommon::handleerror(hipMalloc((void **) &gU_i, nint*sizeof(floatingpoint)));
//        CUDAcommon::handleerror(hipMalloc((void **) &gU_sum, sizeof(floatingpoint)));

        char a[] = "BranchingFF";
        char b[] = "Branching Dihedral Cosine";
        CUDAcommon::handleerror(hipMalloc((void **) &gFF, 100 * sizeof(char)));
        CUDAcommon::handleerror(hipMalloc((void **) &ginteraction, 100 * sizeof(char)));
        CUDAcommon::handleerror(hipMemcpy(gFF, a, 100 * sizeof(char), hipMemcpyHostToDevice));
        CUDAcommon::handleerror(hipMemcpy(ginteraction, b, 100 * sizeof(char), hipMemcpyHostToDevice));
    }
    else{
        blocksnthreadse.push_back(0);
        blocksnthreadse.push_back(0);
        blocksnthreadsez.push_back(0);
        blocksnthreadsez.push_back(0);
        blocksnthreadsf.push_back(0);
        blocksnthreadsf.push_back(0);
    }

}
floatingpoint* BranchingDihedralCosine::energy(floatingpoint *coord, floatingpoint *f, int *beadSet,
                                         floatingpoint *kdih, floatingpoint *pos, int *params) {
//    if(blocksnthreadse[1]>0) {
//
//        BranchingDihedralCosineenergy<<<blocksnthreadse[0], blocksnthreadse[1], (12 * blocksnthreadse[1]) * sizeof
//                (floatingpoint), stream>>>
//                          (coord, f, beadSet, kdih, pos, params, gU_i, CUDAcommon::getCUDAvars().gculpritID,
//                                  CUDAcommon::getCUDAvars().gculpritFF,
//                                  CUDAcommon::getCUDAvars().gculpritinteraction, gFF, ginteraction);
//        auto cvars = CUDAcommon::getCUDAvars();
//        cvars.streamvec.push_back(&stream);
//        CUDAcommon::cudavars = cvars;
//        CUDAcommon::handleerror( hipGetLastError(),"BranchingDihedralCosineenergy", "BranchingDihedralCosine.cu");
//        floatingpoint* gpu_Utot = CUDAcommon::getCUDAvars().gpu_energy;
//        addvector<<<1,1,0,stream>>>(gU_i, params, gU_sum, gpu_Utot);
//        CUDAcommon::handleerror( hipGetLastError() ,"BranchingDihedralCosineenergy", "BranchingDihedralCosine.cu");
//        return gU_sum;}
//    else
//        return NULL;
}


floatingpoint* BranchingDihedralCosine::energy(floatingpoint *coord, floatingpoint *f, int *beadSet,
                                         floatingpoint *kdih, floatingpoint *pos, floatingpoint *z,
                                         int *params) {
        if(blocksnthreadse[1]>0) {

        BranchingDihedralCosineenergy<<<blocksnthreadse[0], blocksnthreadse[1], (12 * blocksnthreadse[1]) * sizeof
                (floatingpoint), stream>>>
                          (coord, f, beadSet, kdih, pos, params, gU_i, z, CUDAcommon::getCUDAvars().gculpritID,
                                  CUDAcommon::getCUDAvars().gculpritFF,
                                  CUDAcommon::getCUDAvars().gculpritinteraction, gFF, ginteraction);
//        auto cvars = CUDAcommon::getCUDAvars();
//        cvars.streamvec.push_back(&stream);
//        CUDAcommon::cudavars = cvars;
//        CUDAcommon::handleerror( hipGetLastError(),"BranchingDihedralCosineenergy", "BranchingDihedralCosine.cu");
//        floatingpoint* gpu_Utot = CUDAcommon::getCUDAvars().gpu_energy;
//        addvector<<<1,1,0,stream>>>(gU_i, params, gU_sum, gpu_Utot);
        CUDAcommon::handleerror( hipGetLastError() ,"BranchingDihedralCosineenergy", "BranchingDihedralCosine.cu");
//        return gU_sum;
        }

    if(blocksnthreadsez[1]>0) {
        BranchingDihedralCosineenergyz << < blocksnthreadsez[0], blocksnthreadsez[1], (24 * blocksnthreadsez[1]) *
                                            sizeof(floatingpoint), stream>> > (coord, f, beadSet, kdih, pos,
                                            params, gU_i, z, CUDAcommon::getCUDAvars().gculpritID,
                CUDAcommon::getCUDAvars().gculpritFF,
                CUDAcommon::getCUDAvars().gculpritinteraction, gFF, ginteraction );
//        auto cvars = CUDAcommon::getCUDAvars();
//        cvars.streamvec.push_back(&stream);
//        CUDAcommon::cudavars = cvars;
//        CUDAcommon::handleerror(hipGetLastError(),"BranchingDihedralCosineenergyz", "BranchingDihedralCosine.cu");
//        floatingpoint* gpu_Utot = CUDAcommon::getCUDAvars().gpu_energy;
//        addvector<<<1,1,0,stream>>>(gU_i, params, gU_sum, gpu_Utot);
        CUDAcommon::handleerror(hipGetLastError(),"BranchingDihedralCosineenergyz", "BranchingDihedralCosine.cu");

        return gU_sum;
    }
    if(blocksnthreadse[1]<=0 && blocksnthreadsez[1]<=0)
        return NULL;
    else{
        auto cvars = CUDAcommon::getCUDAvars();
        cvars.streamvec.push_back(&stream);
        CUDAcommon::cudavars = cvars;
        floatingpoint* gpu_Utot = CUDAcommon::getCUDAvars().gpu_energy;

//        addvector<<<1,1,0,stream>>>(gU_i,params, gU_sum, gpu_Utot);
//        hipStreamSynchronize(stream);
//        addvectorred<<<1,200,200*sizeof(floatingpoint),stream>>>(gU_i,params, gU_sum, gpu_Utot);
//        hipStreamSynchronize(stream);
//        std::cout<<"bntaddvec "<<bntaddvec2.at(0)<<" "<<bntaddvec2.at(1)<<" "<<bntaddvec2.at(0)<<" "
//                ""<<bntaddvec2.at(2)<<" "<<bntaddvec2.at(3)<<endl;
        resetfloatingpointvariableCUDA<<<1,1,0,stream>>>(gU_sum);
        addvectorred2<<<bntaddvec2.at(2),bntaddvec2.at(3), bntaddvec2.at(3) * sizeof(floatingpoint),stream>>>(gU_i,
                params, gU_sum, gpu_Utot);
//        CUDAcommon::handleerror(hipDeviceSynchronize(),"FilamentBendingCosineenergyz", "FilamentBendingCosine.cu");
        CUDAcommon::handleerror(hipGetLastError(),"FilamentBendingCosineenergyz", "FilamentBendingCosine.cu");
        return gU_sum;
    }
}
void BranchingDihedralCosine::forces(floatingpoint *coord, floatingpoint *f, int *beadSet,
                                      floatingpoint *kdih,  floatingpoint *pos, int *params) {
    if (blocksnthreadsf[1] > 0) {
        BranchingDihedralCosineforces << < blocksnthreadsf[0], blocksnthreadsf[1], (12 * blocksnthreadsf[1]) *
                                            sizeof(floatingpoint), stream >> > (coord, f, beadSet, kdih, pos, params);
        auto cvars = CUDAcommon::getCUDAvars();
        cvars.streamvec.push_back(&stream);
        CUDAcommon::cudavars = cvars;
        CUDAcommon::handleerror(hipGetLastError(),"BranchingDihedralCosineforces", "BranchingDihedralCosine.cu");
    }
}

void BranchingDihedralCosine::checkforculprit() {
    CUDAcommon::printculprit("BranchingDihedral","BranchingDihedralCosine");
    BranchingPoint *br;
    br = (BranchingPoint::getBranchingPoints()[CUDAcommon::getCUDAvars().culpritID[0]]);
    cout<<"Printing culprit branching point information."<<endl;
    br->printSelf();
    exit(EXIT_FAILURE);
}
#endif
floatingpoint BranchingDihedralCosine::energy(floatingpoint *coord, totalforcefloatingpoint *f, int *beadSet,
                                       floatingpoint *kdih, floatingpoint *pos){


    int n = BranchingDihedral<BranchingDihedralCosine>::n;
    int nint = BranchingPoint::getBranchingPoints().size();


    floatingpoint *coord1, *coord2, *coord3, *coord4, n1n2, U_i;
    floatingpoint *mp = new floatingpoint[3];
    floatingpoint *n1 = new floatingpoint[3];
    floatingpoint *n2 = new floatingpoint[3];

    totalenergyfloatingpoint U = 0.0;

    for(int i = 0; i < nint; i += 1) {

        coord1 = &coord[3 * beadSet[n * i]];
        coord2 = &coord[3 * beadSet[n * i + 1]];
        coord3 = &coord[3 * beadSet[n * i + 2]];
        coord4 = &coord[3 * beadSet[n * i + 3]];

        midPointCoordinate(mp, coord1, coord2, pos[i]);

        vectorProduct(n1, mp, coord2, mp, coord3);
        vectorProduct(n2, coord3, coord4, mp, coord3);

        normalizeVector(n1);
        normalizeVector(n2);
        n1n2 = dotProduct(n1, n2);

        U_i = kdih[i] * ( 1 - n1n2 );

        if(fabs(U_i) == numeric_limits<floatingpoint>::infinity()
           || U_i != U_i || U_i < -1.0) {

            //set culprit and return
            BranchingInteractions::_branchingCulprit = BranchingPoint::getBranchingPoints()[i];

            return -1;
        }

        U += U_i;
    }
    delete [] mp;
    delete [] n1;
    delete [] n2;

    return U;
}


floatingpoint BranchingDihedralCosine::energy(floatingpoint *coord, totalforcefloatingpoint *f, int *beadSet,
                                       floatingpoint *kdih, floatingpoint *pos, floatingpoint d){

    int n = BranchingDihedral<BranchingDihedralCosine>::n;
    int nint = BranchingPoint::getBranchingPoints().size();


    floatingpoint *coord1, *coord2, *coord3, *coord4, n1n2, U_i;
    totalforcefloatingpoint *f1, *f2, *f3, *f4;
    floatingpoint *mp = new floatingpoint[3];
    floatingpoint *n1 = new floatingpoint[3];
    floatingpoint *n2 = new floatingpoint[3];
    floatingpoint *zero = new floatingpoint[3]; zero[0] = 0; zero[1] = 0; zero[2] = 0;

    totalenergyfloatingpoint U = 0.0;

    for(int i = 0; i < nint; i += 1) {

        coord1 = &coord[3 * beadSet[n * i]];
        coord2 = &coord[3 * beadSet[n * i + 1]];
        coord3 = &coord[3 * beadSet[n * i + 2]];
        coord4 = &coord[3 * beadSet[n * i + 3]];

        f1 = &f[3 * beadSet[n * i]];
        f2 = &f[3 * beadSet[n * i + 1]];
        f3 = &f[3 * beadSet[n * i + 2]];
        f4 = &f[3 * beadSet[n * i + 3]];

        midPointCoordinateStretched(mp, coord1, f1, coord2, f2, pos[i], d);

        vectorProductStretched(n1, mp, zero, coord2, f2, mp, zero, coord3, f3, d);
        vectorProductStretched(n2, coord3, f3, coord4, f4, mp, zero, coord3, f3, d);

        normalizeVector(n1);
        normalizeVector(n2);
        n1n2 = dotProduct(n1, n2);

        U_i = kdih[i] * ( 1 - n1n2 );

        if(fabs(U_i) == numeric_limits<floatingpoint>::infinity()
           || U_i != U_i || U_i < -1.0) {

            //set culprit and return
            BranchingInteractions::_branchingCulprit = BranchingPoint::getBranchingPoints()[i];

            return -1;
        }

        U += U_i;
    }
    delete [] mp;
    delete [] n1;
    delete [] n2;
    delete [] zero;
    return U;
}

void BranchingDihedralCosine::forces(floatingpoint *coord, totalforcefloatingpoint *f, int *beadSet,
                                     floatingpoint *kdih, floatingpoint *pos){

    int n = BranchingDihedral<BranchingDihedralCosine>::n;
    int nint = BranchingPoint::getBranchingPoints().size();


    floatingpoint *coord1, *coord2, *coord3, *coord4, N1, N2, n1n2, f0, NN1, NN2, X, D, Y, position;
    totalforcefloatingpoint *f1, *f2, *f3, *f4;
	floatingpoint n2x, n1y, xd, yd, xx, xy, yy, XD, X1, X2, Y1, Y2, D1, D2, YD;
    floatingpoint *mp = new floatingpoint[3];
    floatingpoint *n1 = new floatingpoint[3];
    floatingpoint *n2 = new floatingpoint[3];
    floatingpoint *zero = new floatingpoint[3]; zero[0] = 0; zero[1] = 0; zero[2] = 0;

    floatingpoint U = 0;

    for(int i = 0; i < nint; i += 1) {

        coord1 = &coord[3 * beadSet[n * i]];
        coord2 = &coord[3 * beadSet[n * i + 1]];
        coord3 = &coord[3 * beadSet[n * i + 2]];
        coord4 = &coord[3 * beadSet[n * i + 3]];

        f1 = &f[3 * beadSet[n * i]];
        f2 = &f[3 * beadSet[n * i + 1]];
        f3 = &f[3 * beadSet[n * i + 2]];
        f4 = &f[3 * beadSet[n * i + 3]];

        midPointCoordinate(mp, coord1, coord2, pos[i]);

        vectorProduct(n1, mp, coord2, mp, coord3);
        vectorProduct(n2, coord3, coord4, mp, coord3);

        N1 = sqrt(dotProduct(n1, n1));
        N2 = sqrt(dotProduct(n2, n2));
        n1n2 = dotProduct(n1, n2);

        f0 = kdih[i]/N1/N2;

        NN1 = n1n2/N1/N1;
        NN2 = n1n2/N2/N2;

        X = sqrt(scalarProduct(mp, coord2, mp, coord2));
        D = sqrt(scalarProduct(mp, coord3, mp, coord3));
        Y = sqrt(scalarProduct(coord3, coord4, coord3, coord4));

        n2x = scalarProduct(zero, n2, mp, coord2);
        n1y = scalarProduct(zero, n1, coord3, coord4);
        xd = scalarProduct(mp, coord2, mp, coord3);
        yd = scalarProduct(coord3, coord4, mp, coord3);

        xx = scalarProduct(mp, coord2, mp, coord2);
        xy = scalarProduct(mp, coord2, coord3, coord4);
        yy = scalarProduct(coord3, coord4, coord3, coord4);

        XD = n2x/D/X/X/X;
        X1 = -NN2*xd/D/X + yd/D/Y + yd/D/D/X/Y;
        X2 = xd*yd/D/D/X/X/X/Y;
        Y1 = -xd/D/X - xd/D/D/X/Y + NN1*yd/D/Y;
        Y2 = xd*yd/D/D/X/Y/Y/Y;
        D1 = NN2*xx/D/X - xy/D/X-xy/D/Y - 2*xy/D/D/X/Y + NN1*yy/D/Y;
        D2 = xd*xy/D/D/X/X/X/Y;
        YD = n1y/D/Y/Y/Y;

        position = pos[i];

        //force on b1:
        f1[0] += f0*(- (1 - position)*XD*(1-position)*( (coord2[1] - coord1[1])*(coord3[2] - (1-position)*coord1[2] - position*coord2[2]) - (coord2[2] - coord1[2])*(coord3[1] - (1-position)*coord1[1] - position*coord2[1]) ) + (1 - position)*(X1 - X2)*(1-position)*(coord2[0] - coord1[0]) - (1 - position)*Y1*(coord4[0] - coord3[0]) + (1 - position)*(D1 + D2)*(coord3[0] - (1-position)*coord1[0] - position*coord2[0]));


        f1[1] += f0*(- (1 - position)*XD*(1-position)*( (coord2[2] - coord1[2])*(coord3[0] - (1-position)*coord1[0] - position*coord2[0]) - (coord2[0] - coord1[0])*(coord3[2] - (1-position)*coord1[2] - position*coord2[2]) ) + (1 - position)*(X1 - X2)*(1-position)*(coord2[1] - coord1[1]) - (1 - position)*Y1*(coord4[1] - coord3[1]) + (1 - position)*(D1 + D2)*(coord3[1] - (1-position)*coord1[1] - position*coord2[1]));

        f1[2] += f0*(- (1 - position)*XD*(1-position)*( (coord2[0] - coord1[0])*(coord3[1] - (1-position)*coord1[1] - position*coord2[1]) - (coord2[1] - coord1[1])*(coord3[0] - (1-position)*coord1[0] - position*coord2[0]) ) + (1 - position)*(X1 - X2)*(1-position)*(coord2[2] - coord1[2]) - (1 - position)*Y1*(coord4[2] - coord3[2]) + (1 - position)*(D1 + D2)*(coord3[2] - (1-position)*coord1[2] - position*coord2[2]));


        //force on b2:
        f2[0] += f0*( (1 - position)*XD*(1-position)*( (coord2[1] - coord1[1])*(coord3[2] - (1-position)*coord1[2] - position*coord2[2]) - (coord2[2] - coord1[2])*(coord3[1] - (1-position)*coord1[1] - position*coord2[1]) ) + (X2 + position*(X1 - X2))*(1-position)*(coord2[0] - coord1[0]) - position*Y1*(coord4[0] - coord3[0]) + (position*(D1 + D2) - D2)*(coord3[0] - (1-position)*coord1[0] - position*coord2[0]) );

        f2[1] += f0*( (1 - position)*XD*(1-position)*( (coord2[2] - coord1[2])*(coord3[0] - (1-position)*coord1[0] - position*coord2[0]) - (coord2[0] - coord1[0])*(coord3[2] - (1-position)*coord1[2] - position*coord2[2]) ) + (X2 + position*(X1 - X2))*(1-position)*(coord2[1] - coord1[1]) - position*Y1*(coord4[1] - coord3[1]) + (position*(D1 + D2) - D2)*(coord3[1] - (1-position)*coord1[1] - position*coord2[1]) );

        f2[2] += f0*( (1 - position)*XD*(1-position)*( (coord2[0] - coord1[0])*(coord3[1] - (1-position)*coord1[1] - position*coord2[1]) - (coord2[1] - coord1[1])*(coord3[0] - (1-position)*coord1[0] - position*coord2[0]) ) + (X2 + position*(X1 - X2))*(1-position)*(coord2[2] - coord1[2]) - position*Y1*(coord4[2] - coord3[2]) + (position*(D1 + D2) - D2)*(coord3[2] - (1-position)*coord1[2] - position*coord2[2]) );

        //force on b3:
        f3[0] += f0*(-YD*( (coord4[1] - coord3[1])*(coord3[2] - (1-position)*coord1[2] - position*coord2[2]) - (coord4[2] - coord3[2])*(coord3[1] - (1-position)*coord1[1] - position*coord2[1]) ) - X1*(1-position)*(coord2[0] - coord1[0]) + (Y1 - Y2)*(coord4[0] - coord3[0]) + (D2 - D1)*(coord3[0] - (1-position)*coord1[0] - position*coord2[0]));

        f3[1] += f0*(-YD*( (coord4[2] - coord3[2])*(coord3[0] - (1-position)*coord1[0] - position*coord2[0]) - (coord4[0] - coord3[0])*(coord3[2] - (1-position)*coord1[2] - position*coord2[2]) ) - X1*(1-position)*(coord2[1] - coord1[1]) + (Y1 - Y2)*(coord4[1] - coord3[1]) + (D2 - D1)*(coord3[1] - (1-position)*coord1[1] - position*coord2[1]));

        f3[2] += f0*(-YD*( (coord4[0] - coord3[0])*(coord3[1] - (1-position)*coord1[1] - position*coord2[1]) - (coord4[1] - coord3[1])*(coord3[0] - (1-position)*coord1[0] - position*coord2[0]) ) - X1*(1-position)*(coord2[2] - coord1[2]) + (Y1 - Y2)*(coord4[2] - coord3[2]) + (D2 - D1)*(coord3[2] - (1-position)*coord1[2] - position*coord2[2]));


        //force on b4:
        f4[0] +=f0*( YD*( (coord4[1] - coord3[1])*(coord3[2] - (1-position)*coord1[2] - position*coord2[2]) - (coord4[2] - coord3[2])*(coord3[1] - (1-position)*coord1[1] - position*coord2[1]) ) + Y2*(coord4[0] - coord3[0]) - D2*(coord3[0] - (1-position)*coord1[0] - position*coord2[0]) );

        f4[1] +=f0*( YD*( (coord4[2] - coord3[2])*(coord3[0] - (1-position)*coord1[0] - position*coord2[0]) - (coord4[0] - coord3[0])*(coord3[2] - (1-position)*coord1[2] - position*coord2[2]) ) + Y2*(coord4[1] - coord3[1]) - D2*(coord3[1] - (1-position)*coord1[1] - position*coord2[1]) );

        f4[2] +=f0*( YD*( (coord4[0] - coord3[0])*(coord3[1] - (1-position)*coord1[1] - position*coord2[1]) - (coord4[1] - coord3[1])*(coord3[0] - (1-position)*coord1[0] - position*coord2[0]) ) + Y2*(coord4[2] - coord3[2]) - D2*(coord3[2] - (1-position)*coord1[2] - position*coord2[2]) );
    }
    delete [] mp;
    delete [] n1;
    delete [] n2;
    delete [] zero;
}
