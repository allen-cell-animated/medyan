#include "hip/hip_runtime.h"

//------------------------------------------------------------------
//  **MEDYAN** - Simulation Package for the Mechanochemical
//               Dynamics of Active Networks, v3.1
//
//  Copyright (2015-2016)  Papoian Lab, University of Maryland
//
//                 ALL RIGHTS RESERVED
//
//  See the MEDYAN web page for more information:
//  http://www.medyan.org
//------------------------------------------------------------------

#include "BranchingDihedralCosine.h"
#include "BranchingDihedralCosineCUDA.h"
#include "BranchingDihedral.h"

#include "BranchingPoint.h"
#include "Bead.h"

#include "MathFunctions.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "nvToolsExt.h"

using namespace mathfunc;
#ifdef CUDAACCL
void BranchingDihedralCosine::deallocate(){
    CUDAcommon::handleerror(hipStreamDestroy(stream));
    CUDAcommon::handleerror(hipFree(gU_i));
    CUDAcommon::handleerror(hipFree(gU_sum));
}
void BranchingDihedralCosine::optimalblocksnthreads( int nint){
    //CUDA stream create
    CUDAcommon::handleerror(hipStreamCreate(&stream));
    blocksnthreadse.clear();
    blocksnthreadsez.clear();
    blocksnthreadsf.clear();
    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the
    // maximum occupancy for a full device launch
//    int gridSize;    // The actual grid size needed, based on input size
//    unaryfn::argument_type blksize;
//    unaryfn::result_type result;
//    unaryfn ufn;
    if(nint>0) {
        hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize,
                                                       BranchingDihedralCosineenergy, blockToSmem, 0);
//    std::cout<<(nint +blockSize -1) / blockSize<<" "<<blockSize<<endl;
//
//    hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize,
//                                        CUDAExclVolRepulsionenergy, 0, 0);
        blocksnthreadse.push_back((nint + blockSize - 1) / blockSize);
        blocksnthreadse.push_back(blockSize);
//    std::cout<<(nint +blockSize -1) / blockSize<<" "<<blockSize<<endl;
        blockSize = 0;

        hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize,
                                                       BranchingDihedralCosineenergyz, blockToSmemez, 0);
        blocksnthreadsez.push_back((nint + blockSize - 1) / blockSize);
        blocksnthreadsez.push_back(blockSize);
        blockSize = 0;

        hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize,
                                                       BranchingDihedralCosineforces, blockToSmem, 0);
        blocksnthreadsf.push_back((nint + blockSize - 1) / blockSize);
        blocksnthreadsf.push_back(blockSize);

        CUDAcommon::handleerror(hipMalloc((void **) &gU_i, nint*sizeof(double)));
        CUDAcommon::handleerror(hipMalloc((void **) &gU_sum, sizeof(double)));
    }
    else{
        blocksnthreadse.push_back(0);
        blocksnthreadse.push_back(0);
        blocksnthreadsez.push_back(0);
        blocksnthreadsez.push_back(0);
        blocksnthreadsf.push_back(0);
        blocksnthreadsf.push_back(0);
    }

}
double* BranchingDihedralCosine::energy(double *coord, double *f, int *beadSet,
                                         double *kdih, double *pos, int *params) {
    if(blocksnthreadse[1]>0) {

        BranchingDihedralCosineenergy<<<blocksnthreadse[0], blocksnthreadse[1], (12 * blocksnthreadse[1]) * sizeof
                (double), stream>>>
                          (coord, f, beadSet, kdih, pos, params, gU_i);
        auto cvars = CUDAcommon::getCUDAvars();
        cvars.streamvec.push_back(&stream);
        CUDAcommon::cudavars = cvars;
        CUDAcommon::handleerror( hipGetLastError(),"BranchingDihedralCosineenergy", "BranchingDihedralCosine.cu");
        double* gpu_Utot = CUDAcommon::getCUDAvars().gpu_energy;
        addvector<<<1,1,0,stream>>>(gU_i, params, gU_sum, gpu_Utot);
        CUDAcommon::handleerror( hipGetLastError() ,"BranchingDihedralCosineenergy", "BranchingDihedralCosine.cu");
        return gU_sum;}
    else
        return NULL;
}


double* BranchingDihedralCosine::energy(double *coord, double *f, int *beadSet,
                                         double *kdih, double *pos, double *z,
                                         int *params) {

    if(blocksnthreadsez[1]>0) {
        BranchingDihedralCosineenergyz << < blocksnthreadsez[0], blocksnthreadsez[1], (24 * blocksnthreadsez[1]) *
                                            sizeof(double), stream>> > (coord, f, beadSet, kdih, pos,
                                            params, gU_i, z );
        auto cvars = CUDAcommon::getCUDAvars();
        cvars.streamvec.push_back(&stream);
        CUDAcommon::cudavars = cvars;
        CUDAcommon::handleerror(hipGetLastError(),"BranchingDihedralCosineenergyz", "BranchingDihedralCosine.cu");
        double* gpu_Utot = CUDAcommon::getCUDAvars().gpu_energy;
        addvector<<<1,1,0,stream>>>(gU_i, params, gU_sum, gpu_Utot);
        CUDAcommon::handleerror(hipGetLastError(),"BranchingDihedralCosineenergyz", "BranchingDihedralCosine.cu");

        return gU_sum;
    }else
        return NULL;
}
void BranchingDihedralCosine::forces(double *coord, double *f, int *beadSet,
                                      double *kdih,  double *pos, int *params) {
    if (blocksnthreadsf[1] > 0) {
        BranchingDihedralCosineforces << < blocksnthreadsf[0], blocksnthreadsf[1], (12 * blocksnthreadsf[1]) *
                                            sizeof(double), stream >> > (coord, f, beadSet, kdih, pos, params);
        auto cvars = CUDAcommon::getCUDAvars();
        cvars.streamvec.push_back(&stream);
        CUDAcommon::cudavars = cvars;
        CUDAcommon::handleerror(hipGetLastError(),"BranchingDihedralCosineforces", "BranchingDihedralCosine.cu");
    }
}
#endif
double BranchingDihedralCosine::energy(double *coord, double *f, int *beadSet,
                                       double *kdih, double *pos){


    int n = BranchingDihedral<BranchingDihedralCosine>::n;
    int nint = BranchingPoint::getBranchingPoints().size();


    double *coord1, *coord2, *coord3, *coord4, n1n2, U_i;
    double *mp = new double[3];
    double *n1 = new double[3];
    double *n2 = new double[3];

    double U = 0;

    for(int i = 0; i < nint; i += 1) {

        coord1 = &coord[3 * beadSet[n * i]];
        coord2 = &coord[3 * beadSet[n * i + 1]];
        coord3 = &coord[3 * beadSet[n * i + 2]];
        coord4 = &coord[3 * beadSet[n * i + 3]];

        midPointCoordinate(mp, coord1, coord2, pos[i]);

        vectorProduct(n1, mp, coord2, mp, coord3);
        vectorProduct(n2, coord3, coord4, mp, coord3);

        normalizeVector(n1);
        normalizeVector(n2);
        n1n2 = dotProduct(n1, n2);

        U_i = kdih[i] * ( 1 - n1n2 );

        if(fabs(U_i) == numeric_limits<double>::infinity()
           || U_i != U_i || U_i < -1.0) {

            //set culprit and return
            BranchingInteractions::_branchingCulprit = BranchingPoint::getBranchingPoints()[i];

            return -1;
        }

        U += U_i;
    }
    delete mp;
    delete n1;
    delete n2;

    return U;
}


double BranchingDihedralCosine::energy(double *coord, double *f, int *beadSet,
                                       double *kdih, double *pos, double d){

    int n = BranchingDihedral<BranchingDihedralCosine>::n;
    int nint = BranchingPoint::getBranchingPoints().size();


    double *coord1, *coord2, *coord3, *coord4, *f1, *f2, *f3, *f4, n1n2, U_i;
    double *mp = new double[3];
    double *n1 = new double[3];
    double *n2 = new double[3];
    double *zero = new double[3]; zero[0] = 0; zero[1] = 0; zero[2] = 0;

    double U = 0;

    for(int i = 0; i < nint; i += 1) {

        coord1 = &coord[3 * beadSet[n * i]];
        coord2 = &coord[3 * beadSet[n * i + 1]];
        coord3 = &coord[3 * beadSet[n * i + 2]];
        coord4 = &coord[3 * beadSet[n * i + 3]];

        f1 = &f[3 * beadSet[n * i]];
        f2 = &f[3 * beadSet[n * i + 1]];
        f3 = &f[3 * beadSet[n * i + 2]];
        f4 = &f[3 * beadSet[n * i + 3]];

        midPointCoordinateStretched(mp, coord1, f1, coord2, f2, pos[i], d);

        vectorProductStretched(n1, mp, zero, coord2, f2, mp, zero, coord3, f3, d);
        vectorProductStretched(n2, coord3, f3, coord4, f4, mp, zero, coord3, f3, d);

        normalizeVector(n1);
        normalizeVector(n2);
        n1n2 = dotProduct(n1, n2);

        U_i = kdih[i] * ( 1 - n1n2 );

        if(fabs(U_i) == numeric_limits<double>::infinity()
           || U_i != U_i || U_i < -1.0) {

            //set culprit and return
            BranchingInteractions::_branchingCulprit = BranchingPoint::getBranchingPoints()[i];

            return -1;
        }

        U += U_i;
    }
    delete mp;
    delete n1;
    delete n2;
    delete zero;
    return U;
}

void BranchingDihedralCosine::forces(double *coord, double *f, int *beadSet,
                                     double *kdih, double *pos){

    int n = BranchingDihedral<BranchingDihedralCosine>::n;
    int nint = BranchingPoint::getBranchingPoints().size();


    double *coord1, *coord2, *coord3, *coord4, *f1, *f2, *f3, *f4, N1, N2, n1n2, f0, NN1, NN2, X, D, Y, position;
    double n2x, n1y, xd, yd, xx, xy, yy, XD, X1, X2, Y1, Y2, D1, D2, YD;
    double *mp = new double[3];
    double *n1 = new double[3];
    double *n2 = new double[3];
    double *zero = new double[3]; zero[0] = 0; zero[1] = 0; zero[2] = 0;

    double U = 0;

    for(int i = 0; i < nint; i += 1) {

        coord1 = &coord[3 * beadSet[n * i]];
        coord2 = &coord[3 * beadSet[n * i + 1]];
        coord3 = &coord[3 * beadSet[n * i + 2]];
        coord4 = &coord[3 * beadSet[n * i + 3]];

        f1 = &f[3 * beadSet[n * i]];
        f2 = &f[3 * beadSet[n * i + 1]];
        f3 = &f[3 * beadSet[n * i + 2]];
        f4 = &f[3 * beadSet[n * i + 3]];

        midPointCoordinate(mp, coord1, coord2, pos[i]);

        vectorProduct(n1, mp, coord2, mp, coord3);
        vectorProduct(n2, coord3, coord4, mp, coord3);

        N1 = sqrt(dotProduct(n1, n1));
        N2 = sqrt(dotProduct(n2, n2));
        n1n2 = dotProduct(n1, n2);

        f0 = kdih[i]/N1/N2;

        NN1 = n1n2/N1/N1;
        NN2 = n1n2/N2/N2;

        X = sqrt(scalarProduct(mp, coord2, mp, coord2));
        D = sqrt(scalarProduct(mp, coord3, mp, coord3));
        Y = sqrt(scalarProduct(coord3, coord4, coord3, coord4));

        n2x = scalarProduct(zero, n2, mp, coord2);
        n1y = scalarProduct(zero, n1, coord3, coord4);
        xd = scalarProduct(mp, coord2, mp, coord3);
        yd = scalarProduct(coord3, coord4, mp, coord3);

        xx = scalarProduct(mp, coord2, mp, coord2);
        xy = scalarProduct(mp, coord2, coord3, coord4);
        yy = scalarProduct(coord3, coord4, coord3, coord4);

        XD = n2x/D/X/X/X;
        X1 = -NN2*xd/D/X + yd/D/Y + yd/D/D/X/Y;
        X2 = xd*yd/D/D/X/X/X/Y;
        Y1 = -xd/D/X - xd/D/D/X/Y + NN1*yd/D/Y;
        Y2 = xd*yd/D/D/X/Y/Y/Y;
        D1 = NN2*xx/D/X - xy/D/X-xy/D/Y - 2*xy/D/D/X/Y + NN1*yy/D/Y;
        D2 = xd*xy/D/D/X/X/X/Y;
        YD = n1y/D/Y/Y/Y;

        position = pos[i];

        //force on b1:
        f1[0] += f0*(- (1 - position)*XD*(1-position)*( (coord2[1] - coord1[1])*(coord3[2] - (1-position)*coord1[2] - position*coord2[2]) - (coord2[2] - coord1[2])*(coord3[1] - (1-position)*coord1[1] - position*coord2[1]) ) + (1 - position)*(X1 - X2)*(1-position)*(coord2[0] - coord1[0]) - (1 - position)*Y1*(coord4[0] - coord3[0]) + (1 - position)*(D1 + D2)*(coord3[0] - (1-position)*coord1[0] - position*coord2[0]));


        f1[1] += f0*(- (1 - position)*XD*(1-position)*( (coord2[2] - coord1[2])*(coord3[0] - (1-position)*coord1[0] - position*coord2[0]) - (coord2[0] - coord1[0])*(coord3[2] - (1-position)*coord1[2] - position*coord2[2]) ) + (1 - position)*(X1 - X2)*(1-position)*(coord2[1] - coord1[1]) - (1 - position)*Y1*(coord4[1] - coord3[1]) + (1 - position)*(D1 + D2)*(coord3[1] - (1-position)*coord1[1] - position*coord2[1]));

        f1[2] += f0*(- (1 - position)*XD*(1-position)*( (coord2[0] - coord1[0])*(coord3[1] - (1-position)*coord1[1] - position*coord2[1]) - (coord2[1] - coord1[1])*(coord3[0] - (1-position)*coord1[0] - position*coord2[0]) ) + (1 - position)*(X1 - X2)*(1-position)*(coord2[2] - coord1[2]) - (1 - position)*Y1*(coord4[2] - coord3[2]) + (1 - position)*(D1 + D2)*(coord3[2] - (1-position)*coord1[2] - position*coord2[2]));


        //force on b2:
        f2[0] += f0*( (1 - position)*XD*(1-position)*( (coord2[1] - coord1[1])*(coord3[2] - (1-position)*coord1[2] - position*coord2[2]) - (coord2[2] - coord1[2])*(coord3[1] - (1-position)*coord1[1] - position*coord2[1]) ) + (X2 + position*(X1 - X2))*(1-position)*(coord2[0] - coord1[0]) - position*Y1*(coord4[0] - coord3[0]) + (position*(D1 + D2) - D2)*(coord3[0] - (1-position)*coord1[0] - position*coord2[0]) );

        f2[1] += f0*( (1 - position)*XD*(1-position)*( (coord2[2] - coord1[2])*(coord3[0] - (1-position)*coord1[0] - position*coord2[0]) - (coord2[0] - coord1[0])*(coord3[2] - (1-position)*coord1[2] - position*coord2[2]) ) + (X2 + position*(X1 - X2))*(1-position)*(coord2[1] - coord1[1]) - position*Y1*(coord4[1] - coord3[1]) + (position*(D1 + D2) - D2)*(coord3[1] - (1-position)*coord1[1] - position*coord2[1]) );

        f2[2] += f0*( (1 - position)*XD*(1-position)*( (coord2[0] - coord1[0])*(coord3[1] - (1-position)*coord1[1] - position*coord2[1]) - (coord2[1] - coord1[1])*(coord3[0] - (1-position)*coord1[0] - position*coord2[0]) ) + (X2 + position*(X1 - X2))*(1-position)*(coord2[2] - coord1[2]) - position*Y1*(coord4[2] - coord3[2]) + (position*(D1 + D2) - D2)*(coord3[2] - (1-position)*coord1[2] - position*coord2[2]) );

        //force on b3:
        f3[0] += f0*(-YD*( (coord4[1] - coord3[1])*(coord3[2] - (1-position)*coord1[2] - position*coord2[2]) - (coord4[2] - coord3[2])*(coord3[1] - (1-position)*coord1[1] - position*coord2[1]) ) - X1*(1-position)*(coord2[0] - coord1[0]) + (Y1 - Y2)*(coord4[0] - coord3[0]) + (D2 - D1)*(coord3[0] - (1-position)*coord1[0] - position*coord2[0]));

        f3[1] += f0*(-YD*( (coord4[2] - coord3[2])*(coord3[0] - (1-position)*coord1[0] - position*coord2[0]) - (coord4[0] - coord3[0])*(coord3[2] - (1-position)*coord1[2] - position*coord2[2]) ) - X1*(1-position)*(coord2[1] - coord1[1]) + (Y1 - Y2)*(coord4[1] - coord3[1]) + (D2 - D1)*(coord3[1] - (1-position)*coord1[1] - position*coord2[1]));

        f3[2] += f0*(-YD*( (coord4[0] - coord3[0])*(coord3[1] - (1-position)*coord1[1] - position*coord2[1]) - (coord4[1] - coord3[1])*(coord3[0] - (1-position)*coord1[0] - position*coord2[0]) ) - X1*(1-position)*(coord2[2] - coord1[2]) + (Y1 - Y2)*(coord4[2] - coord3[2]) + (D2 - D1)*(coord3[2] - (1-position)*coord1[2] - position*coord2[2]));


        //force on b4:
        f4[0] +=f0*( YD*( (coord4[1] - coord3[1])*(coord3[2] - (1-position)*coord1[2] - position*coord2[2]) - (coord4[2] - coord3[2])*(coord3[1] - (1-position)*coord1[1] - position*coord2[1]) ) + Y2*(coord4[0] - coord3[0]) - D2*(coord3[0] - (1-position)*coord1[0] - position*coord2[0]) );

        f4[1] +=f0*( YD*( (coord4[2] - coord3[2])*(coord3[0] - (1-position)*coord1[0] - position*coord2[0]) - (coord4[0] - coord3[0])*(coord3[2] - (1-position)*coord1[2] - position*coord2[2]) ) + Y2*(coord4[1] - coord3[1]) - D2*(coord3[1] - (1-position)*coord1[1] - position*coord2[1]) );

        f4[2] +=f0*( YD*( (coord4[0] - coord3[0])*(coord3[1] - (1-position)*coord1[1] - position*coord2[1]) - (coord4[1] - coord3[1])*(coord3[0] - (1-position)*coord1[0] - position*coord2[0]) ) + Y2*(coord4[2] - coord3[2]) - D2*(coord3[2] - (1-position)*coord1[2] - position*coord2[2]) );
    }
    delete mp;
    delete n1;
    delete n2;
    delete zero;
}
