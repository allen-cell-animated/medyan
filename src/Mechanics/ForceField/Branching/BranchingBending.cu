
//------------------------------------------------------------------
//  **MEDYAN** - Simulation Package for the Mechanochemical
//               Dynamics of Active Networks, v3.1
//
//  Copyright (2015-2016)  Papoian Lab, University of Maryland
//
//                 ALL RIGHTS RESERVED
//
//  See the MEDYAN web page for more information:
//  http://www.medyan.org
//------------------------------------------------------------------

#include "BranchingBending.h"

#include "BranchingBendingCosine.h"

#include "BranchingPoint.h"
#include "Cylinder.h"
#include "Bead.h"
#include "nvToolsExt.h"
#include "cross_check.h"

template <class BBendingInteractionType>
void BranchingBending<BBendingInteractionType>::vectorize() {

    beadSet = new int[n * BranchingPoint::getBranchingPoints().size()];
    kbend = new double[BranchingPoint::getBranchingPoints().size()];
    eqt = new double[BranchingPoint::getBranchingPoints().size()];

    int i = 0;

    for (auto b: BranchingPoint::getBranchingPoints()) {

        beadSet[n * i] = b->getFirstCylinder()->getFirstBead()->_dbIndex;
        beadSet[n * i + 1] = b->getFirstCylinder()->getSecondBead()->_dbIndex;
        beadSet[n * i + 2] = b->getSecondCylinder()->getFirstBead()->_dbIndex;
        beadSet[n * i + 3] = b->getSecondCylinder()->getSecondBead()->_dbIndex;

        kbend[i] = b->getMBranchingPoint()->getStretchingConstant();
        eqt[i] = b->getMBranchingPoint()->getEqTheta();

        i++;
    }
    //CUDA
#ifdef CUDAACCL
    F_i = new double [3 * Bead::getBeads().size()];
    nvtxRangePushA("CVFF");
    int numInteractions = BranchingPoint::getBranchingPoints().size();
    _FFType.optimalblocksnthreads(numInteractions);

    CUDAcommon::handleerror(hipMalloc((void **) &gpu_beadSet, n * numInteractions * sizeof(int)));
    CUDAcommon::handleerror(hipMemcpy(gpu_beadSet, beadSet, n * numInteractions * sizeof(int),
                                       hipMemcpyHostToDevice));

    CUDAcommon::handleerror(hipMalloc((void **) &gpu_kbend, numInteractions * sizeof(double)));
    CUDAcommon::handleerror(hipMemcpy(gpu_kbend, kbend, numInteractions * sizeof(double), hipMemcpyHostToDevice));

    CUDAcommon::handleerror(hipMalloc((void **) &gpu_eqt, numInteractions * sizeof(double)));
    CUDAcommon::handleerror(hipMemcpy(gpu_eqt, eqt, numInteractions * sizeof(double), hipMemcpyHostToDevice));

    vector<int> params;
    params.push_back(int(n));
    params.push_back(numInteractions);
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_params, 2 * sizeof(int)));
    CUDAcommon::handleerror(hipMemcpy(gpu_params, params.data(), 2 * sizeof(int), hipMemcpyHostToDevice));
    nvtxRangePop();
#endif
}

template<class BBendingInteractionType>
void BranchingBending<BBendingInteractionType>::deallocate() {

    delete beadSet;
    delete kbend;
    delete eqt;
#ifdef CUDAACCL
    _FFType.deallocate();
    CUDAcommon::handleerror(hipFree(gpu_beadSet));
    CUDAcommon::handleerror(hipFree(gpu_kbend));
    CUDAcommon::handleerror(hipFree(gpu_eqt));
    CUDAcommon::handleerror(hipFree(gpu_params));
#endif
}



template <class BBendingInteractionType>
double BranchingBending<BBendingInteractionType>::computeEnergy(double *coord, double *f, double d) {

    double U_i[1], U_ii;
    double* gU_i;
    U_ii = NULL;
#ifdef CUDAACCL
    //has to be changed to accomodate aux force
    double * gpu_coord=CUDAcommon::getCUDAvars().gpu_coord;
    double * gpu_force=CUDAcommon::getCUDAvars().gpu_force;
    double * gpu_d = CUDAcommon::getCUDAvars().gpu_lambda;
    nvtxRangePushA("CCEBB");

    if(d == 0.0){
        nvtxRangePushA("CCEBB1");
        gU_i=_FFType.energy(gpu_coord, gpu_force, gpu_beadSet, gpu_kbend, gpu_eqt, gpu_params);
        nvtxRangePop();
    }
    else{
        nvtxRangePushA("CCEBB2");
        gU_i=_FFType.energy(gpu_coord, gpu_force, gpu_beadSet, gpu_kbend, gpu_eqt, gpu_d,
                            gpu_params);
        nvtxRangePop();
    }
    nvtxRangePop();
#else
    nvtxRangePushA("SCEBB");

    if (d == 0.0)
        U_ii = _FFType.energy(coord, f, beadSet, kbend, eqt);
    else
        U_ii = _FFType.energy(coord, f, beadSet, kbend, eqt, d);
    nvtxRangePop();
#endif
    return U_ii;
}

template <class BBendingInteractionType>
void BranchingBending<BBendingInteractionType>::computeForces(double *coord, double *f) {
#ifdef CUDAACCL
    //has to be changed to accomodate aux force
    double * gpu_coord=CUDAcommon::getCUDAvars().gpu_coord;

    double * gpu_force;

    if(cross_checkclass::Aux){
        nvtxRangePushA("CCFBB");

        gpu_force=CUDAcommon::getCUDAvars().gpu_forceAux;
        _FFType.forces(gpu_coord, gpu_force, gpu_beadSet, gpu_kbend, gpu_eqt, gpu_params);
        nvtxRangePop();
    }
    else {
        nvtxRangePushA("CCFBB");

        gpu_force = CUDAcommon::getCUDAvars().gpu_force;
        _FFType.forces(gpu_coord, gpu_force, gpu_beadSet, gpu_kbend, gpu_eqt, gpu_params);
        nvtxRangePop();
    }
#else
    nvtxRangePushA("SCFBB");

    _FFType.forces(coord, f, beadSet, kbend, eqt);
    nvtxRangePop();
#endif
}

///Template specializations
template double BranchingBending<BranchingBendingCosine>::computeEnergy(double *coord, double *f, double d);
template void BranchingBending<BranchingBendingCosine>::computeForces(double *coord, double *f);
template void BranchingBending<BranchingBendingCosine>::vectorize();
template void BranchingBending<BranchingBendingCosine>::deallocate();
