#include "hip/hip_runtime.h"

//------------------------------------------------------------------
//  **MEDYAN** - Simulation Package for the Mechanochemical
//               Dynamics of Active Networks, v3.1
//
//  Copyright (2015-2016)  Papoian Lab, University of Maryland
//
//                 ALL RIGHTS RESERVED
//
//  See the MEDYAN web page for more information:
//  http://www.medyan.org
//------------------------------------------------------------------

#include <cmath>

#include "BranchingBendingCosine.h"
#include "BranchingBending.h"
#include "BranchingBendingCosineCUDA.h"

#include "BranchingPoint.h"
#include "Bead.h"

#include "MathFunctions.h"
#ifdef CUDAACCL
#include "nvToolsExt.h"
#endif

using namespace mathfunc;
#ifdef CUDAACCL
void BranchingBendingCosine::deallocate(){
    if(!(CUDAcommon::getCUDAvars().conservestreams))
        CUDAcommon::handleerror(hipStreamDestroy(stream));
    CUDAcommon::handleerror(hipFree(gU_i));
    CUDAcommon::handleerror(hipFree(gU_sum));
    CUDAcommon::handleerror(hipFree(gFF));
    CUDAcommon::handleerror(hipFree(ginteraction));
}
void BranchingBendingCosine::optimalblocksnthreads( int nint){
    //CUDA stream create
    if(stream == NULL || !(CUDAcommon::getCUDAvars().conservestreams))
        CUDAcommon::handleerror(hipStreamCreate(&stream));
    blocksnthreadse.clear();
    blocksnthreadsez.clear();
    blocksnthreadsf.clear();
    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the
    // maximum occupancy for a full device launch
    if(nint>0) {
        hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize,
                                                       BranchingBendingCosineenergy, blockToSmem, 0);
        blocksnthreadse.push_back((nint + blockSize - 1) / blockSize);
        blocksnthreadse.push_back(blockSize);
        blockSize = 0;

        hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize,
                                                       BranchingBendingCosineenergyz, blockToSmemez, 0);
        blocksnthreadsez.push_back((nint + blockSize - 1) / blockSize);
        blocksnthreadsez.push_back(blockSize);
        blockSize = 0;

        hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize,
                                                       BranchingBendingCosineforces, blockToSmem, 0);
        blocksnthreadsf.push_back((nint + blockSize - 1) / blockSize);
        blocksnthreadsf.push_back(blockSize);

//        CUDAcommon::handleerror(hipMalloc((void **) &gU_i, nint*sizeof(double)));
//        CUDAcommon::handleerror(hipMalloc((void **) &gU_sum, sizeof(double)));
        //get addition vars
        bntaddvec2.clear();
        bntaddvec2 = getaddred2bnt(nint);
        CUDAcommon::handleerror(hipMalloc((void **) &gU_i, bntaddvec2.at(0)*sizeof(double)));
        CUDAcommon::handleerror(hipMemset(gU_i, 0, bntaddvec2.at(0) * sizeof(double)));
//        CUDAcommon::handleerror(hipMalloc((void **) &gU_i, nint*sizeof(double)));
        CUDAcommon::handleerror(hipMalloc((void **) &gU_sum, sizeof(double)));

        char a[] = "BranchingFF";
        char b[] = "Branching Bending Cosine";
        CUDAcommon::handleerror(hipMalloc((void **) &gFF, 100 * sizeof(char)));
        CUDAcommon::handleerror(hipMalloc((void **) &ginteraction, 100 * sizeof(char)));
        CUDAcommon::handleerror(hipMemcpy(gFF, a, 100 * sizeof(char), hipMemcpyHostToDevice));
        CUDAcommon::handleerror(hipMemcpy(ginteraction, b, 100 * sizeof(char), hipMemcpyHostToDevice));
    }
    else{
        blocksnthreadse.push_back(0);
        blocksnthreadse.push_back(0);
        blocksnthreadsez.push_back(0);
        blocksnthreadsez.push_back(0);
        blocksnthreadsf.push_back(0);
        blocksnthreadsf.push_back(0);
    }

}
double* BranchingBendingCosine::energy(double *coord, double *f, int *beadSet,
                                       double *kbend, double *eqt, int *params) {
//    if(blocksnthreadse[1]>0) {
//        BranchingBendingCosineenergy<<<blocksnthreadse[0], blocksnthreadse[1], (12 * blocksnthreadse[1]) * sizeof
//                (double), stream>>> (coord, f, beadSet, kbend, eqt, params, gU_i, CUDAcommon::getCUDAvars().gculpritID,
//                CUDAcommon::getCUDAvars().gculpritFF,
//                CUDAcommon::getCUDAvars().gculpritinteraction, gFF, ginteraction);
//        auto cvars = CUDAcommon::getCUDAvars();
//        cvars.streamvec.push_back(&stream);
//        CUDAcommon::cudavars = cvars;
//        CUDAcommon::handleerror( hipGetLastError() ,"BranchingBendingCosineenergy", "BranchingBendingCosine.cu");
//        double* gpu_Utot = CUDAcommon::getCUDAvars().gpu_energy;
//        addvector<<<1,1,0,stream>>>(gU_i,params, gU_sum, gpu_Utot);
//        CUDAcommon::handleerror( hipGetLastError(),"BranchingBendingCosineenergy", "BranchingBendingCosine.cu");
//        return gU_sum;}
//    else
//        return NULL;
}


double* BranchingBendingCosine::energy(double *coord, double *f, int *beadSet,
                                       double *kbend, double *eqt, double *z, int *params) {
        if(blocksnthreadse[1]>0) {
        BranchingBendingCosineenergy<<<blocksnthreadse[0], blocksnthreadse[1], (12 * blocksnthreadse[1]) * sizeof
                (double), stream>>> (coord, f, beadSet, kbend, eqt, params, gU_i, z, CUDAcommon::getCUDAvars()
                .gculpritID,
                CUDAcommon::getCUDAvars().gculpritFF,
                CUDAcommon::getCUDAvars().gculpritinteraction, gFF, ginteraction);
            CUDAcommon::handleerror( hipGetLastError() ,"BranchingBendingCosineenergy", "BranchingBendingCosine.cu");
//        auto cvars = CUDAcommon::getCUDAvars();
//        cvars.streamvec.push_back(&stream);
//        CUDAcommon::cudavars = cvars;
//        double* gpu_Utot = CUDAcommon::getCUDAvars().gpu_energy;
//        addvector<<<1,1,0,stream>>>(gU_i,params, gU_sum, gpu_Utot);
//        CUDAcommon::handleerror( hipGetLastError(),"BranchingBendingCosineenergy", "BranchingBendingCosine.cu");
//        return gU_sum;
        }
    if(blocksnthreadsez[1]>0) {

        BranchingBendingCosineenergyz << < blocksnthreadsez[0], blocksnthreadsez[1], (24 * blocksnthreadsez[1]) *
                                      sizeof(double), stream>> > (coord, f, beadSet, kbend, eqt, params, gU_i, z,
                CUDAcommon::getCUDAvars().gculpritID,
                CUDAcommon::getCUDAvars().gculpritFF,
                CUDAcommon::getCUDAvars().gculpritinteraction, gFF, ginteraction );
        CUDAcommon::handleerror(hipGetLastError(),"BranchingBendingCosineenergyz", "BranchingBendingCosine.cu");
//        auto cvars = CUDAcommon::getCUDAvars();
//        cvars.streamvec.push_back(&stream);
//        CUDAcommon::cudavars = cvars;
//        double* gpu_Utot = CUDAcommon::getCUDAvars().gpu_energy;
//        addvector<<<1,1,0,stream>>>(gU_i,params, gU_sum, gpu_Utot);
//        CUDAcommon::handleerror(hipGetLastError(),"BranchingBendingCosineenergyz", "BranchingBendingCosine.cu");
//        nvtxRangePushA("CCEBBzv2");
//        BranchingBendingCosineenergyz2 << < blocksnthreadsf[0], blocksnthreadsf[1], (12 * blocksnthreadsf[1]) *
//                                                                                     sizeof(double), stream>> > (coord, f, beadSet, kbend, eqt, params, gU_i, z );
//        CUDAcommon::handleerror(hipGetLastError(),"BranchingBendingCosineenergyz2", "BranchingBendingCosine.cu");
//        nvtxRangePop();
//        return gU_sum;
    }
    if(blocksnthreadse[1]<=0 && blocksnthreadsez[1]<=0)
        return NULL;
    else {
        auto cvars = CUDAcommon::getCUDAvars();
        cvars.streamvec.push_back(&stream);
        CUDAcommon::cudavars = cvars;
        double *gpu_Utot = CUDAcommon::getCUDAvars().gpu_energy;

//        addvector<<<1,1,0,stream>>>(gU_i,params, gU_sum, gpu_Utot);
//        hipStreamSynchronize(stream);
//        addvectorred<<<1,200,200*sizeof(double),stream>>>(gU_i,params, gU_sum, gpu_Utot);
//        hipStreamSynchronize(stream);
//        std::cout<<"bntaddvec "<<bntaddvec2.at(0)<<" "<<bntaddvec2.at(1)<<" "<<bntaddvec2.at(0)<<" "
//                ""<<bntaddvec2.at(2)<<" "<<bntaddvec2.at(3)<<endl;
        resetdoublevariableCUDA << < 1, 1, 0, stream >> > (gU_sum);
        addvectorred2 << < bntaddvec2.at(2), bntaddvec2.at(3), bntaddvec2.at(3) * sizeof(double), stream >> > (gU_i,
                params, gU_sum, gpu_Utot);
//        CUDAcommon::handleerror(hipDeviceSynchronize(),"FilamentBendingCosineenergyz", "FilamentBendingCosine.cu");
        CUDAcommon::handleerror(hipGetLastError(), "FilamentBendingCosineenergyz", "FilamentBendingCosine.cu");
        return gU_sum;
    }
}

void BranchingBendingCosine::forces(double *coord, double *f, int *beadSet,
                                    double *kbend, double *eqt, int *params){
    if(blocksnthreadsf[1]>0) {
        BranchingBendingCosineforces << < blocksnthreadsf[0], blocksnthreadsf[1], (12 * blocksnthreadsf[1]) *
                                                                                  sizeof(double), stream >> > (coord, f, beadSet, kbend, eqt, params);
        auto cvars = CUDAcommon::getCUDAvars();
        cvars.streamvec.push_back(&stream);
        CUDAcommon::cudavars = cvars;
        CUDAcommon::handleerror(hipGetLastError(),"BranchingBendingCosineforces", "BranchingBendingCosine.cu");
    }
}
void BranchingBendingCosine::checkforculprit() {
    CUDAcommon::printculprit("BranchingBending","BranchingBendingCosine");
    BranchingPoint* br;
    br = (BranchingPoint::getBranchingPoints()[CUDAcommon::getCUDAvars().culpritID[0]]);
    cout<<"Printing culprit branching point information."<<endl;
    br->printSelf();
    exit(EXIT_FAILURE);
}
#endif

double BranchingBendingCosine::energy(double *coord, double *f, int *beadSet,
                                      double *kbend, double *eqt){

    int n = BranchingBending<BranchingBendingCosine>::n;
    int nint = BranchingPoint::getBranchingPoints().size();

    double *coord1, *coord2, *coord3, *coord4, U_i, L1, L2, L1L2, l1l2, phi, dPhi;

    double U = 0.0;

    for(int i = 0; i < nint; i += 1) {

        coord1 = &coord[3 * beadSet[n * i]];
        coord2 = &coord[3 * beadSet[n * i + 1]];
        coord3 = &coord[3 * beadSet[n * i + 2]];
        coord4 = &coord[3 * beadSet[n * i + 3]];

        L1 = sqrt(scalarProduct(coord1, coord2,
                                coord1, coord2));
        L2 = sqrt(scalarProduct(coord3, coord4,
                                coord3, coord4));

        L1L2 = L1*L2;
        l1l2 = scalarProduct(coord1, coord2,
                             coord3, coord4);

        phi = safeacos(l1l2 / L1L2);
        dPhi = phi-eqt[i];

        U_i = kbend[i] * ( 1 - cos(dPhi) );

        if(fabs(U_i) == numeric_limits<double>::infinity()
           || U_i != U_i || U_i < -1.0) {

            //set culprit and return
            BranchingInteractions::_branchingCulprit = BranchingPoint::getBranchingPoints()[i];

            return -1;
        }

        U += U_i;
    }

    return U;
}

double BranchingBendingCosine::energy(double *coord, double *f, int *beadSet,
                                      double *kbend, double *eqt, double d){

    int n = BranchingBending<BranchingBendingCosine>::n;
    int nint = BranchingPoint::getBranchingPoints().size();

    double *coord1, *coord2, *coord3, *coord4, *force1, *force2, *force3, *force4, U_i, L1, L2, L1L2, l1l2, phi, dPhi;

    double U = 0.0;

    for(int i = 0; i < nint; i += 1) {

        coord1 = &coord[3 * beadSet[n * i]];
        coord2 = &coord[3 * beadSet[n * i + 1]];
        coord3 = &coord[3 * beadSet[n * i + 2]];
        coord4 = &coord[3 * beadSet[n * i + 3]];

        force1 = &f[3 * beadSet[n * i]];
        force2 = &f[3 * beadSet[n * i + 1]];
        force3 = &f[3 * beadSet[n * i + 2]];
        force4 = &f[3 * beadSet[n * i + 3]];


        L1 = sqrt(scalarProductStretched(coord1, force1, coord2, force2,
                                         coord1, force1, coord2, force2, d));
        L2 = sqrt(scalarProductStretched(coord3, force3, coord4, force4,
                                         coord3, force3, coord4, force4, d));

        L1L2 = L1*L2;
        l1l2 = scalarProductStretched(coord1, force1, coord2, force2,
                                      coord3, force3, coord4, force4, d);

        phi = safeacos(l1l2 / L1L2);
        dPhi = phi-eqt[i];

        U_i = kbend[i] * ( 1 - cos(dPhi) );

        if(fabs(U_i) == numeric_limits<double>::infinity()
           || U_i != U_i || U_i < -1.0) {

            //set culprit and return
            BranchingInteractions::_branchingCulprit = BranchingPoint::getBranchingPoints()[i];

            return -1;
        }

        U += U_i;
    }

    return U;
}

void BranchingBendingCosine::forces(double *coord, double *f, int *beadSet,
                                    double *kbend, double *eqt){


    int n = BranchingBending<BranchingBendingCosine>::n;
    int nint = BranchingPoint::getBranchingPoints().size();

    double *coord1, *coord2, *coord3, *coord4, *force1, *force2, *force3, *force4;
    double L1, L2, L1L2, l1l2, phi, dPhi, A, B, C, invL1, invL2, k;

//    double U = 0;

    for(int i = 0; i < nint; i += 1) {

        coord1 = &coord[3 * beadSet[n * i]];
        coord2 = &coord[3 * beadSet[n * i + 1]];
        coord3 = &coord[3 * beadSet[n * i + 2]];
        coord4 = &coord[3 * beadSet[n * i + 3]];

        force1 = &f[3 * beadSet[n * i]];
        force2 = &f[3 * beadSet[n * i + 1]];
        force3 = &f[3 * beadSet[n * i + 2]];
        force4 = &f[3 * beadSet[n * i + 3]];

        L1 = sqrt(scalarProduct(coord1, coord2,
                                coord1, coord2));
        L2 = sqrt(scalarProduct(coord3, coord4,
                                coord3, coord4));

        L1L2 = L1*L2;
        l1l2 = scalarProduct(coord1, coord2,
                             coord3, coord4);

        invL1 = 1/L1;
        invL2 = 1/L2;
        A = invL1*invL2;
        B = l1l2*invL1*A*A*L2;
        C = l1l2*invL2*A*A*L1;

//        phi = safeacos(l1l2 / L1L2);
        phi = safeacos(l1l2 * A);
        dPhi = phi-eqt[i];

        k =  kbend[i] * sin(dPhi)/sin(phi);

        //force on i, f = k*(-A*l2 + 2*B*l1):
        force1[0] += k * ((coord3[0] - coord4[0])*A +
                          (coord2[0] - coord1[0])*B );
        force1[1] += k * ((coord3[1] - coord4[1])*A +
                          (coord2[1] - coord1[1])*B );
        force1[2] += k * ((coord3[2] - coord4[2])*A +
                          (coord2[2] - coord1[2])*B );


        //force on i+1, f = k*(A*l2 - 2*B*l1):
        force2[0] += k * ((-coord3[0] + coord4[0])*A -
                          (coord2[0] - coord1[0])*B );
        force2[1] += k * ((-coord3[1] + coord4[1])*A -
                          (coord2[1] - coord1[1])*B );
        force2[2] += k * ((-coord3[2] + coord4[2])*A -
                          (coord2[2] - coord1[2])*B );

        //force on j, k*(-A*l1 + 2*C*l2):
        force3[0] += k *((coord1[0] - coord2[0])*A +
                         (coord4[0] - coord3[0])*C );
        force3[1] += k *((coord1[1] - coord2[1])*A +
                         (coord4[1] - coord3[1])*C );
        force3[2] += k *((coord1[2] - coord2[2])*A +
                         (coord4[2] - coord3[2])*C );

        //force on j+1, k*(A*l1 - 2*C*l2):
        force4[0] += k *((-coord1[0] + coord2[0])*A -
                         (coord4[0] - coord3[0])*C );
        force4[1] += k *((-coord1[1] + coord2[1])*A -
                         (coord4[1] - coord3[1])*C );
        force4[2] += k *((-coord1[2] + coord2[2])*A -
                         (coord4[2] - coord3[2])*C );
    }
}
