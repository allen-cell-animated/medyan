#include "hip/hip_runtime.h"

//------------------------------------------------------------------
//  **MEDYAN** - Simulation Package for the Mechanochemical
//               Dynamics of Active Networks, v3.1
//
//  Copyright (2015-2016)  Papoian Lab, University of Maryland
//
//                 ALL RIGHTS RESERVED
//
//  See the MEDYAN web page for more information:
//  http://www.medyan.org
//------------------------------------------------------------------
#include "SubSystem.h"
#include "BoundaryElement.h"
#include "CompartmentGrid.h"
#include "BindingManager.h"
#include "BindingManagerCUDA.h"
#include "MathFunctions.h"
#include "BoundaryElement.h"
#include "BoundaryElementImpl.h"
#include <vector>
#include "dist_driver.h"
#include "dist_coords.h"
#include "dist_common.h"
#include "Cylinder.h"
using namespace mathfunc;
void SubSystem::resetNeighborLists() {
#ifdef CUDAACCL_NL
    coord = new double[CGMethod::N];
                coord_com = new double[3 * Cylinder::getCylinders().size()];
                beadSet = new int[2 * Cylinder::getCylinders().size()];
                cylID = new int[Cylinder::getCylinders().size()];
                filID = new int[Cylinder::getCylinders().size()];
                filType = new int[Cylinder::getCylinders().size()];
                cmpID = new unsigned int[Cylinder::getCylinders().size()];
                fvecpos = new int[Cylinder::getCylinders().size()];
        //        cylstate= new bool[Cylinder::getCylinders().size()];
        //        cylvecpospercmp = new int[2 * _compartmentGrid->getCompartments().size()];

                if(SysParams::Chemistry().numFilaments > 1) {
                    cout << "CUDA NL cannot handle more than one type of filaments." << endl;
                    exit(EXIT_FAILURE);
                }
                int numBindingSites = SysParams::Chemistry().bindingSites[0].size();
                if(SysParams::Chemistry().numBrancherSpecies[0] > 0)
                    cmon_state_brancher = new int[ numBindingSites * Cylinder::getCylinders().size()];
                if(SysParams::Chemistry().numLinkerSpecies[0] > 0)
                    cmon_state_linker = new int[numBindingSites * Cylinder::getCylinders().size()];
                if(SysParams::Chemistry().numMotorSpecies[0] > 0)
                    cmon_state_motor = new int[numBindingSites * Cylinder::getCylinders().size()];

                int i = 0; //int cID = 0;
                for(auto b: Bead::getBeads()) {
                    //flatten indices
                    int index = 3 * i;
                    coord[index] = b->coordinate[0];
                    coord[index + 1] = b->coordinate[1];
                    coord[index + 2] = b->coordinate[2];
                    i++;
                }
                i = 0; //int countcyl = 0;
        //        for(auto C : _compartmentGrid->getCompartments()) {
        //            int iter = 0;
        //            for(auto nC : C->getNeighbours()) {
        //                cmp_neighbors[nneighbors * cID + iter] = GController::getCompartmentID(nC->coordinates());
        //                        iter++;
        //            }
        //            for(auto k = iter; k < nneighbors; k++ )
        //                cmp_neighbors[nneighbors * cID + k] = -1;

        //            cylvecpospercmp[2 * cID] = countcyl;
        //            countcyl += C->getCylinders().size();
        //            cylvecpospercmp[2 * cID + 1] = countcyl;

        //            if(C->getCylinders().size()>maxnumCyl)
        //                maxnumCyl = C->getCylinders().size();
        //            cID++;
        //            for(auto c:C->getCylinders()){
                 for(auto c:Cylinder::getCylinders()){
                            //flatten indices
                            int index = 3 * i;
                            coord_com[index] = c->coordinate[0];
                            coord_com[index + 1] = c->coordinate[1];
                            coord_com[index + 2] = c->coordinate[2];

                        beadSet[2 * i] = c->getFirstBead()->_dbIndex;
                        beadSet[2 * i + 1] = c->getSecondBead()->_dbIndex;
                        cylID[i] = c->getID();
                        c->_dcIndex = i;
                        fvecpos[i] = c->getPosition();
                        auto fil = dynamic_cast<Filament*>(c->getParent());
                        filID[i] =  fil->getID();
                        cmpID[i] = GController::getCompartmentID(c->getCompartment()->coordinates());
                        filType[i] = fil->getType();
        //                cylstate[i] = c->isFullLength();
                        int j = 0;
                        for(auto it2 = SysParams::Chemistry().bindingSites[fil->getType()].begin();
                            it2 != SysParams::Chemistry().bindingSites[fil->getType()].end(); it2++) {
                            if(SysParams::Chemistry().numBrancherSpecies[0] > 0)
                                cmon_state_brancher[numBindingSites * i + j ] = c->getCCylinder()->getCMonomer(*it2)
                                        ->speciesBound(SysParams::Chemistry().brancherBoundIndex[fil->getType()])->getN();
                            if(SysParams::Chemistry().numLinkerSpecies[0] > 0)
                                cmon_state_linker[numBindingSites * i + j ] = c->getCCylinder()->getCMonomer(*it2)
                                        ->speciesBound(SysParams::Chemistry().linkerBoundIndex[fil->getType()])->getN();
                            if(SysParams::Chemistry().numMotorSpecies[0] > 0)
                                cmon_state_motor[numBindingSites * i + j ] = c->getCCylinder()->getCMonomer(*it2)
                                        ->speciesBound(SysParams::Chemistry().motorBoundIndex[fil->getType()])->getN();
                            j++;
        //                    for(auto k = 0; k< SysParams::Chemistry().numBoundSpecies[0]; k ++) {
        //                        cmon_state[SysParams::Chemistry().bindingSites[fil->getType()].size() * SysParams::Chemistry
        //                                ().numBoundSpecies[0] * i + j] =
        //                                c->getCCylinder()->getCMonomer(*it2)->speciesBound(k)->getN();
        //                        j++;
        //                    }
                        }
                        i++;
                    }
        //        }//Compartment
                //CUDAMALLOC
                //@{
        //        size_t free, total;
        //        CUDAcommon::handleerror(hipMemGetInfo(&free, &total));
        //        fprintf(stdout,"\t### Available VRAM : %g Mo/ %g Mo(total)\n\n",
        //                free/1e6, total/1e6);
        //
        //        hipFree(0);
        //
        //        CUDAcommon::handleerror(hipMemGetInfo(&free, &total));
        //        fprintf(stdout,"\t### Available VRAM : %g Mo/ %g Mo(total)\n\n",
        //                free/1e6, total/1e6);
                CUDAcommon::handleerror(hipMalloc((void **) &gpu_coord, CGMethod::N * sizeof(double)),"cuda data "
                                        "transfer", " SubSystem.h");
                CUDAcommon::handleerror(hipMalloc((void **) &gpu_coord_com, 3 * Cylinder::getCylinders().size() * sizeof
                                        (double)),"cuda data transfer", " SubSystem.h");
                CUDAcommon::handleerror(hipMalloc((void **) &gpu_beadSet, 2 * Cylinder::getCylinders().size() * sizeof
                                        (int)), "cuda data transfer", " SubSystem.h");
                CUDAcommon::handleerror(hipMalloc((void **) &gpu_cylID, Cylinder::getCylinders().size() * sizeof(int)),
                                        "cuda data transfer", " SubSystem.h");
                CUDAcommon::handleerror(hipMalloc((void **) &gpu_fvecpos, Cylinder::getCylinders().size() * sizeof(int)),
                                        "cuda data transfer", " SubSystem.h");
                CUDAcommon::handleerror(hipMalloc((void **) &gpu_filID, Cylinder::getCylinders().size() * sizeof(int)),
                                        "cuda data transfer", " SubSystem.h");
                CUDAcommon::handleerror(hipMalloc((void **) &gpu_filType, Cylinder::getCylinders().size() * sizeof(int)),
                                        "cuda data transfer", " SubSystem.h");
                CUDAcommon::handleerror(hipMalloc((void **) &gpu_cmpID, Cylinder::getCylinders().size() * sizeof(unsigned
                                        int)), "cuda data transfer", " SubSystem.h");
        //        CUDAcommon::handleerror(hipMalloc((void **) &gpu_cylstate, Cylinder::getCylinders().size() * sizeof(int)),
        //                                "cuda data transfer", " SubSystem.h");

                if(SysParams::Chemistry().numBrancherSpecies[0] > 0)
                CUDAcommon::handleerror(hipMalloc((void **) &gpu_cmon_state_brancher, numBindingSites *
                        Cylinder::getCylinders().size() * sizeof(int)), "cuda data transfer", " SubSystem.h");
                if(SysParams::Chemistry().numLinkerSpecies[0] > 0)
                    CUDAcommon::handleerror(hipMalloc((void **) &gpu_cmon_state_linker, numBindingSites *
                                            Cylinder::getCylinders().size() * sizeof(int)), "cuda data transfer", " SubSystem.h");
                if(SysParams::Chemistry().numMotorSpecies[0] > 0)
                    CUDAcommon::handleerror(hipMalloc((void **) &gpu_cmon_state_motor, numBindingSites *
                                            Cylinder::getCylinders().size() * sizeof(int)), "cuda data transfer", " SubSystem.h");

        //        CUDAcommon::handleerror(hipMalloc((void **) &gpu_cylvecpospercmp,
        //                                2 * _compartmentGrid->getCompartments().size()), "cuda data transfer", " SubSystem.h");
        //        CUDAcommon::handleerror(hipMalloc((void **) &gpu_cmp_neighbors, nneighbors *
        //                                 _compartmentGrid->getCompartments().size() * sizeof(int)), "cuda data transfer",
        //                                " SubSystem.h");
                //@}
                //CUDAMEMCPY
                //@{
                CUDAcommon::handleerror(hipMemcpy(gpu_coord, coord, CGMethod::N *sizeof(double), hipMemcpyHostToDevice));
                CUDAcommon::handleerror(hipMemcpy(gpu_coord_com, coord_com, 3 * Cylinder::getCylinders().size() *sizeof
                                                   (double), hipMemcpyHostToDevice));
                CUDAcommon::handleerror(hipMemcpy(gpu_beadSet, beadSet, 2 * Cylinder::getCylinders().size() *sizeof(int),
                                                   hipMemcpyHostToDevice));
                CUDAcommon::handleerror(hipMemcpy(gpu_cylID, cylID, Cylinder::getCylinders().size() *sizeof(int),
                                                   hipMemcpyHostToDevice));
                CUDAcommon::handleerror(hipMemcpy(gpu_fvecpos, fvecpos, Cylinder::getCylinders().size() *sizeof(int),
                                                   hipMemcpyHostToDevice));
                CUDAcommon::handleerror(hipMemcpy(gpu_filID, filID, Cylinder::getCylinders().size() *sizeof(int),
                                                   hipMemcpyHostToDevice));
                CUDAcommon::handleerror(hipMemcpy(gpu_filType, filType, Cylinder::getCylinders().size() *sizeof(int),
                                                   hipMemcpyHostToDevice));
                CUDAcommon::handleerror(hipMemcpy(gpu_cmpID, cmpID, Cylinder::getCylinders().size() *sizeof(unsigned int),
                                                   hipMemcpyHostToDevice));
        //        CUDAcommon::handleerror(hipMemcpy(gpu_cylstate, cylstate, Cylinder::getCylinders().size() *sizeof(int),
        //                                           hipMemcpyHostToDevice));
                if(SysParams::Chemistry().numBrancherSpecies[0] > 0)
                CUDAcommon::handleerror(hipMemcpy(gpu_cmon_state_brancher, cmon_state_brancher, numBindingSites *
                                        Cylinder::getCylinders().size() * sizeof(int), hipMemcpyHostToDevice));
                if(SysParams::Chemistry().numLinkerSpecies[0] > 0)
                    CUDAcommon::handleerror(hipMemcpy(gpu_cmon_state_linker, cmon_state_linker, numBindingSites *
                                        Cylinder::getCylinders().size() *sizeof(int), hipMemcpyHostToDevice));
                if(SysParams::Chemistry().numMotorSpecies[0] > 0)
                    CUDAcommon::handleerror(hipMemcpy(gpu_cmon_state_motor, cmon_state_motor, numBindingSites *
                                        Cylinder::getCylinders().size() *sizeof(int), hipMemcpyHostToDevice));

        //        CUDAcommon::handleerror(hipMemcpy(gpu_cylvecpospercmp, cylvecpospercmp,
        //                                           2 * _compartmentGrid->getCompartments().size(), hipMemcpyHostToDevice));
                CylCylNLvars cylcylnlvars;
                cylcylnlvars.gpu_coord = gpu_coord;
                cylcylnlvars.gpu_coord_com = gpu_coord_com;
                cylcylnlvars.gpu_beadSet = gpu_beadSet;
                cylcylnlvars.gpu_cylID = gpu_cylID;
                cylcylnlvars.gpu_fvecpos = gpu_fvecpos;
                cylcylnlvars.gpu_filID = gpu_filID;
                cylcylnlvars.gpu_filType = gpu_filType;
                cylcylnlvars.gpu_cmpID = gpu_cmpID;
        //        cylcylnlvars.gpu_cylstate = gpu_cylstate;
                cylcylnlvars.gpu_cmon_state_brancher = gpu_cmon_state_brancher;
                cylcylnlvars.gpu_cmon_state_linker = gpu_cmon_state_linker;
                cylcylnlvars.gpu_cmon_state_motor = gpu_cmon_state_motor;
        //        cylcylnlvars.gpu_cylvecpospercmp = gpu_cylvecpospercmp;

                CUDAcommon::cylcylnlvars = cylcylnlvars;
        //        CUDAcommon::handleerror(hipMemcpy(gpu_cmp_neighbors, cmp_neighbors, nneighbors * _compartmentGrid
        //                                           ->getCompartments().size() *sizeof(int),
        //                                           hipMemcpyHostToDevice));
                //@}
#endif
    //@{ check begins
    /*cylinder* cylindervec  = CUDAcommon::serlvars.cylindervec;
    Cylinder** Cylinderpointervec = CUDAcommon::serlvars.cylinderpointervec;
    CCylinder** ccylindervec = CUDAcommon::serlvars.ccylindervec;
    double* coord = CUDAcommon::serlvars.coord;
    for(auto cyl:Cylinder::getCylinders()){
        int i = cyl->_dcIndex;
        int id1 = cylindervec[i].ID;
        int id2 = Cylinderpointervec[i]->getID();
        int id3 = ccylindervec[i]->getCylinder()->getID();
        if(id1 != id2 || id2 != id3 || id3 != id1)
            std::cout<<id1<<" "<<id2<<" "<<id3<<endl;
        auto b1 = cyl->getFirstBead();
        auto b2 = cyl->getSecondBead();
        long idx1 = b1->_dbIndex;
        long idx2 = b2->_dbIndex;
        cylinder c = cylindervec[i];
        std::cout << "4 bindices for cyl with ID "<<cyl->getID()<<" cindex " << i <<
                  " are "<< idx1 << " " << idx2 << " " << c.bindices[0] << " " << c.bindices[1] << endl;
        if(c.bindices[0] != idx1 || c.bindices[1] != idx2) {

            std::cout << "Bead " << b1->coordinate[0] << " " << b1->coordinate[1] << " "
                    "" << b1->coordinate[2] << " " << " " << b2->coordinate[0] << " "
                              "" << b2->coordinate[1] << " " << b2->coordinate[2] << " idx "
                      << b1->_dbIndex << " "
                              "" << b2->_dbIndex << endl;

            std::cout << coord[3 * idx1] << " " << coord[3 * idx1 + 1] << " "
                      << coord[3 * idx1 + 2] << " "
                              "" << coord[3 * idx2] << " " << coord[3 * idx2 + 1] << " "
                      << coord[3 * idx2 + 2] << endl;
        }

    }*/
    //check ends
    chrono::high_resolution_clock::time_point mins, mine;
    mins = chrono::high_resolution_clock::now();

#ifdef HYBRID_NLSTENCILLIST
    _HneighborList->reset();
    mine= chrono::high_resolution_clock::now();
    chrono::duration<double> elapsed_H(mine - mins);
    std::cout<<"H NLSTEN reset time "<<elapsed_H.count()<<endl;
    mins = chrono::high_resolution_clock::now();
    for (auto nlist : __bneighborLists.getElements())
        nlist->reset();
    mine= chrono::high_resolution_clock::now();
    chrono::duration<double> elapsed_B(mine - mins);
    std::cout<<"H NLSTEN B reset time "<<elapsed_B.count()<<endl;

#elif defined(NLORIGINAL) || defined(NLSTENCILLIST)
#ifndef HYBRID_NLSTENCILLIST
    for (auto nl: _neighborLists.getElements())
            nl->reset();
#endif
#endif
    /*mins = chrono::high_resolution_clock::now();
    for (auto nl: _neighborLists.getElements())
        nl->reset();
    mine= chrono::high_resolution_clock::now();
    chrono::duration<double> elapsed_NL(mine - mins);
    std::cout<<"NLSTEN reset time "<<elapsed_NL.count()<<endl;*/

}
void SubSystem::updateBindingManagers() {
#ifdef CUDAACCL_NL
    if(SysParams::Chemistry().numFilaments > 1) {
        cout << "CUDA Binding Manager cannot handle more than one type of filaments." << endl;
        exit(EXIT_FAILURE);
    }
    initializebindingsitesearchCUDA();

    if(CUDAcommon::getCUDAvars().conservestreams)
        numbindmgrs = 0;
    //Calculate binding sites in CUDA
    Compartment* C0 = _compartmentGrid->getCompartments()[0];
    for(auto &manager : C0->getFilamentBindingManagers()) {

        LinkerBindingManager *lManager;
        MotorBindingManager *mManager;
        BranchingManager *bManager;
        auto cylcylnlvars = CUDAcommon::getCylCylNLvars();
        auto coord = cylcylnlvars.gpu_coord;
        auto beadSet = cylcylnlvars.gpu_beadSet;
        auto cylID = cylcylnlvars.gpu_cylID;
        auto filType = cylcylnlvars.gpu_filType;
        auto filID = cylcylnlvars.gpu_filID;
        int *cmpID = cylcylnlvars.gpu_cmpID;
        //Linker
        if ((lManager = dynamic_cast<LinkerBindingManager *>(manager.get()))) {
            //calculate all binding Sites.
            getallpossiblelinkerbindingsitesCUDA(lManager, cylcylnlvars.gpu_cmon_state_linker);
        }
        //Motor
        else if ((mManager = dynamic_cast<MotorBindingManager *>(manager.get()))) {
            //calculate all binding Sites.
            getallpossiblemotorbindingsitesCUDA(mManager, cylcylnlvars
                    .gpu_cmon_state_motor);
            }
        //Brancher
        else if ((bManager = dynamic_cast<BranchingManager *>(manager.get()))) {
            //calculate all binding Sites.
            getallpossiblebrancherbindingsitesCUDA(bManager, cylcylnlvars
                    .gpu_cmon_state_brancher);

            }
    }

    //Free vars
    terminatebindingsitesearchCUDA();
    //Assign to respective possible bindings.
    assigntorespectivebindingmanagersCUDA();

//    for(auto gpb:gpu_possibleBindings_vec)
//        CUDAcommon::handleerror(hipFree(gpb),"hipFree","SubSystem.cu");
//    for(auto pb:possibleBindings_vec)
//        CUDAcommon::handleerror(hipHostFree(pb), "hipFree", "SubSystem.cu");
//    for(auto np:numpairs_vec)
//        CUDAcommon::handleerror(hipHostFree(np),"hipFree","SubSystem.cu");

    //hipFree
    endresetCUDA();
#endif
#if defined(NLSTENCILLIST) || defined(HYBRID_NLSTENCILLIST)
    //vectorize
    SysParams::MParams.speciesboundvec.clear();
    int cidx = 0;
    vector<int> ncylvec(SysParams::CParams.numFilaments);// Number of cylinders
    // corresponding to each filament type.
//    vector<int> bspeciesoffsetvec(SysParams::CParams.numFilaments);
    auto cylvec = Cylinder::getCylinders();
    int ncyl = cylvec.size();
    delete [] cylsqmagnitudevector;
    cylsqmagnitudevector = new double[Cylinder::vectormaxsize];
    unsigned long maxbindingsitespercyl = 0;
    for(auto ftype = 0; ftype < SysParams::CParams.numFilaments; ftype++) {
        maxbindingsitespercyl = max(maxbindingsitespercyl,SysParams::Chemistry()
                .bindingSites[ftype].size());
    }
    long vectorsize = maxbindingsitespercyl * Cylinder::vectormaxsize;
    vector<bool> branchspeciesbound(vectorsize);
    vector<bool> linkerspeciesbound(vectorsize);
    vector<bool> motorspeciesbound(vectorsize);//stores species bound corresponding to each
    // cylinder.

    //set the size of each species bound vector
    fill(branchspeciesbound.begin(),branchspeciesbound.begin()+vectorsize, 0);
    fill(linkerspeciesbound.begin(),linkerspeciesbound.begin()+vectorsize, 0);
    fill(motorspeciesbound.begin(),motorspeciesbound.begin()+vectorsize, 0);

    //fill with appropriate values.
    for (auto cyl: cylvec) {
//        cout<<cyl->_dcIndex<<" "<<cyl->getID()<<endl;
/*        if(cyl->_dcIndex > Cylinder::vectormaxsize)
            std::cout<<"Cindex "<<cyl->_dcIndex<<" greater than vectorsize "
                    ""<<Cylinder::vectormaxsize<<endl;*/
        //cyl->_dcIndex = cidx;
        auto _filamentType = cyl->getType();
        auto x1 = cyl->getFirstBead()->coordinate;
        auto x2 = cyl->getSecondBead()->coordinate;
        vector<double> X1X2 = {x2[0] - x1[0], x2[1] - x1[1], x2[2] - x1[2]};
        cylsqmagnitudevector[cyl->_dcIndex] = sqmagnitude(X1X2);
        auto cc = cyl->getCCylinder();
        int idx = 0;
        for (auto it1 = SysParams::Chemistry().bindingSites[_filamentType].begin();
             it1 != SysParams::Chemistry().bindingSites[_filamentType].end(); it1++) {

            branchspeciesbound[maxbindingsitespercyl * cyl->_dcIndex + idx] =
                    (cc->getCMonomer(*it1)->speciesBound(
                            SysParams::Chemistry().brancherBoundIndex[_filamentType])->getN());
            linkerspeciesbound[maxbindingsitespercyl * cyl->_dcIndex + idx] =
                    (cc->getCMonomer(*it1)->speciesBound(
                            SysParams::Chemistry().linkerBoundIndex[_filamentType])->getN());
            motorspeciesbound[maxbindingsitespercyl * cyl->_dcIndex + idx] =
                    (cc->getCMonomer(*it1)->speciesBound(
                            SysParams::Chemistry().motorBoundIndex[_filamentType])->getN());
            idx++;
        }
    }
    //@}

    /*for(auto ftype = 0; ftype < SysParams::CParams.numFilaments; ftype++) {
        ncylvec.at(ftype) = cidx;//number of cylinders in each filament.
        bspeciesoffsetvec.at(ftype) = branchspeciesbound.size();
        cidx = 0;
        for (auto cyl: cylvec) {
            //cyl->_dcIndex = cidx;
            auto _filamentType = cyl->getParent()->getType();
            if (_filamentType == ftype) {

                auto x1 = cyl->getFirstBead()->coordinate;
                auto x2 = cyl->getSecondBead()->coordinate;
                vector<double> X1X2 = {x2[0] - x1[0], x2[1] - x1[1], x2[2] - x1[2]};
                cylsqmagnitudevector[cyl->_dcIndex] = sqmagnitude(X1X2);
                auto cc = cyl->getCCylinder();
                int idx = 0;
                for (auto it1 = SysParams::Chemistry().bindingSites[_filamentType].begin();
                     it1 != SysParams::Chemistry().bindingSites[_filamentType].end(); it1++) {
                    branchspeciesbound.push_back (cc->getCMonomer(*it1)->speciesBound(
                            SysParams::Chemistry().brancherBoundIndex[_filamentType])->getN());
                    linkerspeciesbound.push_back (cc->getCMonomer(*it1)->speciesBound(
                            SysParams::Chemistry().linkerBoundIndex[_filamentType])->getN());
                    motorspeciesbound.push_back (cc->getCMonomer(*it1)->speciesBound(
                            SysParams::Chemistry().motorBoundIndex[_filamentType])->getN());
                    idx++;
                }
                cidx++;
            }
        }
    }
    std::cout<<"max cindex "<<Cylinder::maxcindex<<" removed cylinders "
            ""<<Cylinder::removedcindex.size()<<endl;
    std::cout<<"speciesbound size "<<branchspeciesbound.size()<<endl;*/


    SysParams::MParams.speciesboundvec.push_back(branchspeciesbound);
    SysParams::MParams.speciesboundvec.push_back(linkerspeciesbound);
    SysParams::MParams.speciesboundvec.push_back(motorspeciesbound);
    SysParams::CParams.maxbindingsitespercylinder = maxbindingsitespercyl;
    SysParams::MParams.cylsqmagnitudevector = cylsqmagnitudevector;
//    SysParams::MParams.bsoffsetvec = bspeciesoffsetvec;
    SysParams::MParams.ncylvec = ncylvec;
//    std::cout<<SysParams::Mechanics().speciesboundvec.size()<<endl;
//    std::cout<<motorspeciesbound.size()<<endl;
#endif

    chrono::high_resolution_clock::time_point mins, mine;
    mins = chrono::high_resolution_clock::now();
    //SIMD cylinder update
#ifdef SIMDBINDINGSEARCH2
    minsSIMD = chrono::high_resolution_clock::now();
    for(auto C : _compartmentGrid->getCompartments()) {
        C->SIMDcoordinates();
        C->SIMDcoordinates4linkersearch(1);
        C->SIMDcoordinates4motorsearch(1);
        C->getHybridBindingSearchManager()->resetpossibleBindings();
    }
#endif

    if(!initialize) {
        HybridBindingSearchManager::setdOut();
        initialize = true;
    }

#ifdef SIMDBINDINGSEARCH3
    mineSIMD = chrono::high_resolution_clock::now();
    chrono::duration<double> elapsed_runSIMD2(mineSIMD - minsSIMD);
    SIMDtime += elapsed_runSIMD2.count();
    cout<<"SIMD create time "<<elapsed_runSIMD2.count()<<endl;

    minsSIMD = chrono::high_resolution_clock::now();
    for(auto C : _compartmentGrid->getCompartments()) {
        C->SIMDcoordinates_section();
        C->SIMDcoordinates4linkersearch_section(1);
        C->SIMDcoordinates4motorsearch_section(1);
    }
    mineSIMD = chrono::high_resolution_clock::now();
    chrono::duration<double> elapsed_SIMDpart(mineSIMD - minsSIMD);
    cout<<"SIMD create time "<<elapsed_SIMDpart.count()<<endl;
#endif

    //PROTOCOL 1 This call calculates Binding pairs according to SIMD protocol V1
    if(false) {
        minsSIMD = chrono::high_resolution_clock::now();
        for (auto C : _compartmentGrid->getCompartments()) {
#ifdef SIMBDINDINGSEARCH

            C->getHybridBindingSearchManager()->updateAllPossibleBindingsstencil();
//        C->getHybridBindingSearchManager()->checkoccupancy(_idvec);
            for (auto &manager : C->getFilamentBindingManagers()) {
#ifdef NLSTENCILLIST
                BranchingManager *bManager;
                if (bManager = dynamic_cast<BranchingManager *>(manager.get()))
                    manager->updateAllPossibleBindingsstencil();
#endif
            }
#else
            for(auto &manager : C->getFilamentBindingManagers()) {
#ifdef NLORIGINAL
                manager->updateAllPossibleBindings();
#endif
#ifdef NLSTENCILLIST
                manager->updateAllPossibleBindingsstencil();
#endif
#if defined(NLORIGINAL) && defined(NLSTENCILLIST)
                manager->crosscheck();
#endif
            }
#endif
        }
        mineSIMD = chrono::high_resolution_clock::now();
        chrono::duration<double> elapsed_runSIMD(mineSIMD - minsSIMD);
        SIMDtime += elapsed_runSIMD.count();
        cout << "SIMD time " << elapsed_runSIMD.count() << endl;
        cout << "find time " << HybridBindingSearchManager::findtime << endl;
    }

    //PRINT
/*    for(auto C : _compartmentGrid->getCompartments()) {
       C->getHybridBindingSearchManager()->printbindingsizes();
    }*/



    //PROTOCOL #2 SIMD V2
/*    for(auto C : _compartmentGrid->getCompartments()) {
        C->getHybridBindingSearchManager()->resetpossibleBindings();
    }*/

    //This call calculates Binding pairs according to SIMD protocol V2
    if(true) {
/*        int totalupn = 0;
        for (auto C : _compartmentGrid->getCompartments()) {
            totalupn += C->getuniquepermuteNeighbours().size();
            cout<<C->getuniquepermuteNeighbours().size()<<" ";
        }
        cout<<endl;
        cout<<"Unique permutation neighbors "<<totalupn<<endl;*/
		#ifdef SIMDBINDINGSEARCH2
        minsSIMD = chrono::high_resolution_clock::now();
        for (auto C : _compartmentGrid->getCompartments()) {

            C->getHybridBindingSearchManager()->updateAllPossibleBindingsstencilSIMDV2();

            /*for(auto &manager : C->getFilamentBindingManagers()) {ad
    #ifdef NLSTENCILLIST
                BranchingManager* bManager;
                if(bManager = dynamic_cast<BranchingManager *>(manager.get()))
                    manager->updateAllPossibleBindingsstencil();
    #endif
            }*/

        }
        //PRINT
/*            for(auto C : _compartmentGrid->getCompartments()) {
                C->getHybridBindingSearchManager()->printbindingsizes();
            }*/
        mineSIMD = chrono::high_resolution_clock::now();
        chrono::duration<double> elapsed_runSIMDV2(mineSIMD - minsSIMD);
        SIMDtimeV2 += elapsed_runSIMDV2.count();
        cout << "SIMDV2 time " << elapsed_runSIMDV2.count() << endl;
        cout << "findV2 time " << HybridBindingSearchManager::findtimeV2 << endl;
        cout << "Append time " << HybridBindingSearchManager::appendtime << endl;
/*        cout << "Time taken to parse SIMD " << HybridBindingSearchManager::SIMDparse1SIMDparse1 << endl;
        cout << "Time taken to merge SIMD " << HybridBindingSearchManager::SIMDparse2 << endl;
        cout << "Time taken to copy to main google map "
                "" << HybridBindingSearchManager::SIMDparse3 << endl;
        cout << "Time taken to update bs "
                "" << HybridBindingSearchManager::SIMDcountbs << endl;*/
        #endif

    }

#ifdef SIMDBINDINGSEARCH3
    for (auto C : _compartmentGrid->getCompartments())
        C->getHybridBindingSearchManager()->resetpossibleBindings();
    minsSIMD = chrono::high_resolution_clock::now();
    HybridBindingSearchManager::findtimeV3 = 0.0;
    HybridBindingSearchManager::SIMDV3appendtime = 0.0;
    for (auto C : _compartmentGrid->getCompartments()) {
        C->getHybridBindingSearchManager()->updateAllPossibleBindingsstencilSIMDV3(0);
    }
    mineSIMD = chrono::high_resolution_clock::now();
    chrono::duration<double> elapsed_runSIMDV3(mineSIMD - minsSIMD);
    cout << "SIMDV3 time " << elapsed_runSIMDV3.count() << endl;
    cout << "findV3 time " << HybridBindingSearchManager::findtimeV3 << endl;
    cout << "Append time " << HybridBindingSearchManager::SIMDV3appendtime << endl;
    cout<<"-------"<<endl;
#endif
    //PROTOCOL #3 This call calculates Binding pairs according to HYBRID protocol
    // (non-SIMD).
#ifdef HYBRID_NLSTENCILLIST
if(false) {
/*    for (auto C : _compartmentGrid->getCompartments()) {
        C->getHybridBindingSearchManager()->resetpossibleBindings();
    }*/

    minsHYBD = chrono::high_resolution_clock::now();
    for (auto C : _compartmentGrid->getCompartments()) {
#ifdef HYBRID_NLSTENCILLIST
        C->getHybridBindingSearchManager()->updateAllPossibleBindingsstencilHYBD();
/*        for (auto &manager : C->getFilamentBindingManagers()) {
#ifdef NLSTENCILLIST
            BranchingManager *bManager;
            if (bManager = dynamic_cast<BranchingManager *>(manager.get()))
                manager->updateAllPossibleBindingsstencil();
#endif
        }*/
    }
#endif
    mineHYBD = chrono::high_resolution_clock::now();
    chrono::duration<double> elapsed_runHYBD(mineHYBD - minsHYBD);
    HYBDtime += elapsed_runHYBD.count();
    cout << "HYBD time " << elapsed_runHYBD.count() << endl;
    cout<<"HYBD map time "<<HybridBindingSearchManager::HYBDappendtime<<endl;
}
#endif

    mine= chrono::high_resolution_clock::now();
    chrono::duration<double> elapsed_orig(mine - mins);
    std::cout<<"BMgr update time "<<elapsed_orig.count()<<endl;
    //PRINT
/*    for(auto C : _compartmentGrid->getCompartments()) {
        C->getHybridBindingSearchManager()->printbindingsizes();
    }*/
//    exit(EXIT_FAILURE);
}

//OBSOLETE
void SubSystem::vectorizeCylinder() {
    delete [] cylindervec;
    delete [] ccylindervec;
    delete [] cylinderpointervec;
    int Ncyl = Cylinder::getCylinders().size();
    cylindervec = new cylinder[Ncyl];
    ccylindervec = new CCylinder*[Ncyl];
    cylinderpointervec = new Cylinder*[Ncyl];
    //Create cylinder structure
    int i = 0;
    for(auto cyl:Cylinder::getCylinders()){
        //set _dcIndex
        cyl->_dcIndex = i;
        //copy attributes to a structure
        cylindervec[i].filamentID = dynamic_cast<Filament*>(cyl->getParent())->getID();
        cylindervec[i].filamentposition = cyl->getPosition();
        cylindervec[i].bindices[0] = cyl->getFirstBead()->_dbIndex;
        cylindervec[i].bindices[1] = cyl->getSecondBead()->_dbIndex;
        cylindervec[i].cmpID = cyl->getCompartment()->getID();
        cylindervec[i].cindex = i;
        auto coord = cyl->coordinate;
        cylindervec[i].coord[0] = coord[0];
        cylindervec[i].coord[1] = coord[1];
        cylindervec[i].coord[2] = coord[2];
        cylindervec[i].type = cyl->getType();
        cylindervec[i].ID = cyl->getID();
        ccylindervec[i] = cyl->getCCylinder();
        cylinderpointervec[i] = cyl;
        i++;
//        for(int bsc = 0; bsc < nbs; bsc++){
//            double c[3], bead1[3],bead2[3];
//
//            memcpy(bead1, &coord[3*cylindervec[i].bindices[0]], 3 * sizeof(double));
//            memcpy(bead2, &coord[3*cylindervec[i].bindices[1]], 3 * sizeof(double));
//            midPointCoordinate(c,bead1,bead2,bindingsitevec[bsc]);
//            bscoord[12*i+bsc*3] = c[0];
//            bscoord[12*i+bsc*3+1] = c[1];
//            bscoord[12*i+bsc*3+2] = c[2];te<<endl;
//        }
    }
/*    std::cout<<"print for consistency "<<endl;
    for(int idx = 0; idx < Ncyl; idx++) {
        if (cylindervec[idx].cindex != ccylindervec[cylindervec[idx].cindex]->getCylinder()
                ->_dcIndex)
            std::cout << "Fatal mismatch " << cylindervec[idx].cindex << " "
                    ""<<ccylindervec[cylindervec[idx].cindex]->getCylinder()->_dcIndex << endl;
    }*/
    CUDAcommon::serlvars.ccylindervec = ccylindervec;
    CUDAcommon::serlvars.cylindervec = cylindervec;
    CUDAcommon::serlvars.cylinderpointervec = cylinderpointervec;

}

#ifdef CUDAACCL_NL
void SubSystem::initializebindingsitesearchCUDA() {
    //@{ 1. InitializeBSsearch
    //Reset variables
    numpairs_vec.clear();
    possibleBindings_vec.clear();
    gpu_possibleBindings_vec.clear();
//   auto x = CMonomer::_numBSpecies;
//    auto var = SysParams::Chemistry().bmanagerdistances;
    //Malloc params

    //Copy necessary cylinder data to GPU memory
    //@{
    //        if (gpu_params == NULL) {
    int params[3];
    params[0] = SysParams::Chemistry().numBindingSites[0];//filType dependant
    params[1] = 0;//filType dependant
    params[2] = SysParams::Geometry().cylinderNumMon[0];//filType dependant.
    params[3] = Cylinder::getCylinders().size();
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_params, 4 * sizeof(int)),
                            "cuda data transfer", "SubSystem.cu");
    CUDAcommon::handleerror(hipMemcpy(gpu_params, params, 4 * sizeof(int),
                                       hipMemcpyHostToDevice));
//        }
//        if (gpu_bindingSites == NULL) {
    auto bindingSites = SysParams::Chemistry().bindingSites[0];//filType dependant
    int cpu_bindingSites[bindingSites.size()];
    int iii = 0;
    for (auto bs:bindingSites)
    {cpu_bindingSites[iii] = int(bs); iii++;}
    CUDAcommon::handleerror(hipMalloc((void **) &gpu_bindingSites, bindingSites.size() *
                                                                    sizeof(int)), "cuda data transfer", "SubSystem.cu");
    CUDAcommon::handleerror(hipMemcpy(gpu_bindingSites, cpu_bindingSites,
                                       bindingSites.size() *  sizeof(int), hipMemcpyHostToDevice));
//        }
    //@}
}

void SubSystem::getallpossiblelinkerbindingsitesCUDA(LinkerBindingManager* lManager,
                                                     int* cmon_state_linker){
    lManager->assigncudavars();
    hipStream_t  s;
    if(numbindmgrs + 1 > strvec.size() )
    { hipStreamCreate(&s); strvec.push_back(s);}
    else
        s = strvec.at(numbindmgrs);
    numbindmgrs++;
//    int *cmon_state_linker = cylcylnlvars.gpu_cmon_state_linker;
    //1. Assign optimal blocks and threads
    vector<int> blocksnthreads;
    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the maximum occupancy for a full device launch
    int nint = lManager->getNLsize();
    int *NL = lManager->getNLCUDA();
    int *numNLpairs = lManager->getNLsizeCUDA();
    int *numpairs = lManager->getpossiblebindingssizeCUDA();
    double *params2 = lManager->getdistancesCUDA();
    std::cout<<"Total Linker NL size "<<nint<<endl;
//            int *numpairs, test[1];test[0] = 0;
//            CUDAcommon::handleerror(hipMalloc((void **) &numpairs, sizeof(int)), "cuda data transfer", "SubSystem.cu");
//            CUDAcommon::handleerror(hipMemcpy(numpairs, test, sizeof(int), hipMemcpyHostToDevice));
    //2. Calculate binding sites
    if (nint > 0) {
        int *gpu_possibleBindings;

        CUDAcommon::handleerror(hipMalloc((void **) &gpu_possibleBindings, SysParams::Chemistry()
                                                                                    .numBindingSites[0] * 5 * nint * sizeof(int)));
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
                                           updateAllPossibleBindingsCUDA, 0, 0);
        blocksnthreads.push_back((nint + blockSize - 1) / blockSize);
        blocksnthreads.push_back(blockSize);
        std::cout << "Linker blocks and threads " << blocksnthreads.at(0) << " " << blocksnthreads.at(1)
                  << endl;
        resetintvariableCUDA<<<1,1,0,s>>>(numpairs);
        //call CUDA kernel function
        updateAllPossibleBindingsCUDA << < blocksnthreads[0], blocksnthreads[1],0,s >> >
                                                                                  (coord, beadSet, cylID, filID, filType, cmpID, NL, numNLpairs, numpairs,
                                                                                          gpu_params, params2, gpu_possibleBindings, cmon_state_linker,
                                                                                          gpu_bindingSites);

//                CUDAcommon::handleerror(hipDeviceSynchronize());
        //copy binding sites back to CPU
        int *cpu_numpairs, *possibleBindings;
        CUDAcommon::handleerror(hipHostAlloc(&cpu_numpairs, sizeof(int), hipHostMallocDefault),"Copy",
                                "Subsystem.cu");
        CUDAcommon::handleerror(hipMemcpyAsync(cpu_numpairs, numpairs, sizeof(int), hipMemcpyDeviceToHost,
                                                s),"Copy", "Subsystem.cu");
        CUDAcommon::handleerror(hipStreamSynchronize(s),"Stream Sync","Subsystem.cu");
        CUDAcommon::handleerror(hipFree(NL),"hipFree","NeighborListImpl.cu");
        std::cout << "Number of possibleBindings " << cpu_numpairs[0] << endl;
        numpairs_vec.push_back(cpu_numpairs);
        if(cpu_numpairs[0] > 0) {
            CUDAcommon::handleerror(hipHostAlloc(&possibleBindings, 5 * cpu_numpairs[0] * sizeof(int),
                                                  hipHostMallocDefault), "Copy", "Subsystem.cu");
            CUDAcommon::handleerror(
                    hipMemcpyAsync(possibleBindings, gpu_possibleBindings, 5 * cpu_numpairs[0] *
                                                                            sizeof(int), hipMemcpyDeviceToHost,
                                    s), "Copy", "Subsystem.cu");
            possibleBindings_vec.push_back(possibleBindings);
        }
        gpu_possibleBindings_vec.push_back(gpu_possibleBindings);
//                int cpu_numpairs[1];
//                hipMemcpy(cpu_numpairs, numpairs, sizeof(int), hipMemcpyDeviceToHost);
//                std::cout << "Number of possibleBindings " << cpu_numpairs[0] << endl;
//                int possibleBindings[5 * cpu_numpairs[0]];
//                hipMemcpy(possibleBindings, gpu_possibleBindings, 5 * cpu_numpairs[0] * sizeof(int),
//                           hipMemcpyDeviceToHost);
    }
    lManager->freecudavars();
    //Free NL numpairs
    CUDAcommon::handleerror(hipFree(numNLpairs),"hipFree", "SubSystem.cu");
}

void SubSystem::getallpossiblemotorbindingsitesCUDA(MotorBindingManager* mManager, int*
                                                            cmon_state_motor){
    mManager->assigncudavars();
    hipStream_t  s;
    if(numbindmgrs + 1 > strvec.size() )
    { hipStreamCreate(&s); strvec.push_back(s);}
    else
        s = strvec.at(numbindmgrs);
    numbindmgrs++;
//    int *cmon_state_motor = cylcylnlvars.gpu_cmon_state_motor;
    //2. Assign optimal blocks and threads
    vector<int> blocksnthreads;
    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the maximum occupancy for a full device launch
    int nint = mManager->getNLsize();
    int *NL = mManager->getNLCUDA();
    int *numNLpairs = mManager->getNLsizeCUDA();
    int *numpairs = mManager->getpossiblebindingssizeCUDA();
    double *params2 = mManager->getdistancesCUDA();
    std::cout<<"Total Motor NL size "<<nint<<endl;
    if (nint > 0) {
        int *gpu_possibleBindings;

        CUDAcommon::handleerror(hipMalloc((void **) &gpu_possibleBindings, SysParams::Chemistry()
                                                                                    .numBindingSites[0] * 5 * nint * sizeof(int)));
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
                                           updateAllPossibleBindingsCUDA, 0, 0);
        blocksnthreads.push_back((nint + blockSize - 1) / blockSize);
        blocksnthreads.push_back(blockSize);
        std::cout << "Motor blocks and threads " << blocksnthreads.at(0) << " " << blocksnthreads.at(1)
                  << endl;
        resetintvariableCUDA << < 1, 1, 0, s >> > (numpairs);

        updateAllPossibleBindingsCUDA << < blocksnthreads[0], blocksnthreads[1],0,s >> >
                                                                                  (coord, beadSet, cylID, filID, filType, cmpID, NL, numNLpairs, numpairs,
                                                                                          gpu_params, params2,
                                                                                          gpu_possibleBindings, cmon_state_motor,
                                                                                          gpu_bindingSites);

        //copy back to CPU
        int *cpu_numpairs, *possibleBindings;
        CUDAcommon::handleerror(hipHostAlloc(&cpu_numpairs, sizeof(int), hipHostMallocDefault),"Copy",
                                "Subsystem.cu");
        CUDAcommon::handleerror(hipMemcpyAsync(cpu_numpairs, numpairs, sizeof(int), hipMemcpyDeviceToHost,
                                                s),"Copy", "Subsystem.cu");
        CUDAcommon::handleerror(hipStreamSynchronize(s),"Stream Sync","Subsystem.cu");
        CUDAcommon::handleerror(hipFree(NL),"hipFree","NeighborListImpl.cu");
        std::cout << "Number of possibleBindings " << cpu_numpairs[0] << endl;
        numpairs_vec.push_back(cpu_numpairs);
        if(cpu_numpairs[0] > 0) {
            CUDAcommon::handleerror(hipHostAlloc(&possibleBindings, 5 * cpu_numpairs[0] * sizeof(int),
                                                  hipHostMallocDefault), "Copy", "Subsystem.cu");
            CUDAcommon::handleerror(
                    hipMemcpyAsync(possibleBindings, gpu_possibleBindings, 5 * cpu_numpairs[0] *
                                                                            sizeof(int), hipMemcpyDeviceToHost,
                                    s), "Copy", "Subsystem.cu");
            possibleBindings_vec.push_back(possibleBindings);
        }
        gpu_possibleBindings_vec.push_back(gpu_possibleBindings);

//                CUDAcommon::handleerror(hipDeviceSynchronize());
//                CUDAcommon::handleerror(hipDeviceSynchronize());
//                //copy back to CPU
//                int cpu_numpairs[1];
//                hipMemcpy(cpu_numpairs, numpairs, sizeof(int), hipMemcpyDeviceToHost);
//                std::cout << "Number of possibleBindings " << cpu_numpairs[0] << endl;
//                int possibleBindings[5 * cpu_numpairs[0]];
//                hipMemcpy(possibleBindings, gpu_possibleBindings, 5 * cpu_numpairs[0] * sizeof(int),
//                           hipMemcpyDeviceToHost);

    }
    mManager->freecudavars();
    //Free NL numpairs
    CUDAcommon::handleerror(hipFree(numNLpairs),"hipFree", "SubSystem.cu");
}

void SubSystem::getallpossiblebrancherbindingsitesCUDA(BranchingManager* bManager,
                                                       int* cmon_state_brancher) {
    bManager->assigncudavars();
    hipStream_t  s;
    if(numbindmgrs + 1 > strvec.size() )
    { hipStreamCreate(&s); strvec.push_back(s);}
    else
        s = strvec.at(numbindmgrs);
    numbindmgrs++;
//    int *cmon_state_brancher = cylcylnlvars.gpu_cmon_state_brancher;
    //2. Assign optimal blocks and threads
    vector<int> blocksnthreads;
    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the maximum occupancy for a full device launch
    int nint = Cylinder::getCylinders().size();
    //int *NL = bManager->getNLCUDA();
    //int *numNLpairs = bManager->getNLsizeCUDA();
    int *numpairs = bManager->getpossiblebindingssizeCUDA();
    double *params2 = bManager->getdistancesCUDA();
    int *zone = bManager->getzoneCUDA();
    //std::cout<<"Total Motor NL size "<<nint<<endl;
    if (nint > 0) {
        //Boundary plane
        auto beList = BoundaryElement::getBoundaryElements();
        int nbe = BoundaryElement::getBoundaryElements().size();
        double *beListplane = new double[4 * nbe];
        double *gpu_beListplane;
        for (int i = 0; i < nbe; i++) {

            if(dynamic_cast<PlaneBoundaryElement*>(beList[i])) {
                double *x = new double[4];
                beList[i]->elementeqn(x);
                beListplane[4 * i] = x[0];
                beListplane[4 * i +1] = x[1];
                beListplane[4 * i +2] = x[2];
                beListplane[4 * i +3] = x[3];
            }
            else{
                cout<<"CUDA cannot handle non-plane type boundaries. Exiting..."<<endl;
                exit(EXIT_FAILURE);
            }
        }
        CUDAcommon::handleerror(hipMalloc((void **) &gpu_beListplane, 4 * nbe * sizeof(double)));
        CUDAcommon::handleerror(hipMemcpy(gpu_beListplane, beListplane, 4 * nbe * sizeof(double), hipMemcpyHostToDevice));
        //
        int *gpu_possibleBindings;

        CUDAcommon::handleerror(hipMalloc((void **) &gpu_possibleBindings, SysParams::Chemistry()
                                                                                    .numBindingSites[0] * 3 * nint * sizeof(int)));
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
                                           updateAllPossibleBindingsCUDA, 0, 0);
        blocksnthreads.push_back((nint + blockSize - 1) / blockSize);
        blocksnthreads.push_back(blockSize);
        std::cout << "Brancher blocks and threads " << blocksnthreads.at(0) << " " << blocksnthreads.at(1)
                  << endl;
        resetintvariableCUDA << < 1, 1, 0, s >> > (numpairs);

        updateAllPossibleBindingsBrancherCUDA << < blocksnthreads[0], blocksnthreads[1],0,s >> >
                                                                                          (coord, beadSet, cylID, filID, filType, cmpID, numpairs,
                                                                                                  gpu_params, params2, zone, gpu_possibleBindings,gpu_bindingSites,
                                                                                                  cmon_state_brancher, gpu_beListplane);

//                CUDAcommon::handleerror(hipDeviceSynchronize());
        //copy back to CPU
        int *cpu_numpairs, *possibleBindings;
        CUDAcommon::handleerror(hipHostAlloc(&cpu_numpairs, sizeof(int), hipHostMallocDefault),"Copy",
                                "Subsystem.cu");
        CUDAcommon::handleerror(hipMemcpyAsync(cpu_numpairs, numpairs, sizeof(int), hipMemcpyDeviceToHost,
                                                s),"Copy", "Subsystem.cu");
        CUDAcommon::handleerror(hipStreamSynchronize(s),"Stream Sync","Subsystem.cu");
        //CUDAcommon::handleerror(hipFree(NL),"hipFree","NeighborListImpl.cu");
        std::cout << "Number of possibleBindings " << cpu_numpairs[0] << endl;
        numpairs_vec.push_back(cpu_numpairs);
        if(cpu_numpairs[0] > 0) {
            CUDAcommon::handleerror(hipHostAlloc(&possibleBindings, cpu_numpairs[0] * 3 * sizeof(int),
                                                  hipHostMallocDefault), "Copy", "Subsystem.cu");
            CUDAcommon::handleerror(
                    hipMemcpyAsync(possibleBindings, gpu_possibleBindings, cpu_numpairs[0] * 3 *
                                                                            sizeof(int), hipMemcpyDeviceToHost,
                                    s), "Copy", "Subsystem.cu");
            possibleBindings_vec.push_back(possibleBindings);
        }
        gpu_possibleBindings_vec.push_back(gpu_possibleBindings);
        CUDAcommon::handleerror(hipFree(gpu_beListplane),"hipFree", "Subsystem.cu");
    }
    bManager->freecudavars();
}

void SubSystem::terminatebindingsitesearchCUDA(){
    CUDAcommon::handleerror(hipFree(gpu_params),"hipFree", "Subsystem.cu");
    CUDAcommon::handleerror(hipFree(gpu_bindingSites),"hipFree", "Subsystem.cu");
    //Synchronize streams
    for(auto s:strvec) CUDAcommon::handleerror(hipStreamSynchronize(s),"stream sync","SubsSystem.cu");
    //Delete sterams
    if(CUDAcommon::getCUDAvars().conservestreams == false) {
        for (auto s:strvec)
            CUDAcommon::handleerror(hipStreamDestroy(s), "stream destroy", "SubsSystem.cu");
        strvec.clear();
    }
    //clear all possible bindings.
    for(auto c:_compartmentGrid->getCompartments()){
        for(auto &Mgr : c->getFilamentBindingManagers()) {
            Mgr->clearpossibleBindings();
        }
    }
}

void SubSystem::assigntorespectivebindingmanagersCUDA(){
    Compartment* C0 = _compartmentGrid->getCompartments()[0];
    int count = 0;
    for(auto &manager : C0->getFilamentBindingManagers()) {
        LinkerBindingManager *lManager;
        MotorBindingManager *mManager;
        BranchingManager *bManager;
        //Linkers
        if ((lManager = dynamic_cast<LinkerBindingManager *>(manager.get()))) {
            auto numpairs = numpairs_vec[count];
            auto possibleBindings = possibleBindings_vec[count];
            for(auto i = 0; i < numpairs[0]; i++){
                int cID = possibleBindings[5* i];
                int cIndex = possibleBindings[5 * i +1];
                short cbs = short(possibleBindings[5 * i + 2]);
                int cnIndex = possibleBindings[5 * i +3];
                short cnbs = short(possibleBindings[5 * i + 4]);
                auto cylinder = Cylinder::getCylinders()[cIndex];
                auto ncylinder = Cylinder::getCylinders()[cnIndex];
                //get the compartment.
                Compartment* cmp = GController::getCompartment(cID);
                //get corresponding binding manager
                for(auto &cmanager : cmp->getFilamentBindingManagers()) {
                    if ((lManager = dynamic_cast<LinkerBindingManager *>(cmanager.get()))) {
                        auto t1 = tuple<CCylinder*, short>(cylinder->getCCylinder(), cbs);
                        auto t2 = tuple<CCylinder*, short>(ncylinder->getCCylinder(), cnbs);
                        cmanager->appendpossibleBindings(t1,t2);
                    }
                }
            }
            if(numpairs[0] > 0)
                count++;
        }
            //MOTORS
        else if ((mManager = dynamic_cast<MotorBindingManager *>(manager.get()))) {
            auto numpairs = numpairs_vec[count];
            auto possibleBindings = possibleBindings_vec[count];
            for(auto i = 0; i < numpairs[0]; i++){
                int cID = possibleBindings[5 * i];
                int cIndex = possibleBindings[5 * i +1];
                short cbs = short(possibleBindings[5 * i + 2]);
                int cnIndex = possibleBindings[5 * i +3];
                short cnbs = short(possibleBindings[5 * i + 4]);
                auto cylinder = Cylinder::getCylinders()[cIndex];
                auto ncylinder = Cylinder::getCylinders()[cnIndex];
                //get the compartment
                Compartment* cmp = GController::getCompartment(cID);
                //get corresponding binding manager
                for(auto &cmanager : cmp->getFilamentBindingManagers()) {
                    if ((mManager = dynamic_cast<MotorBindingManager *>(cmanager.get()))) {
                        auto t1 = tuple<CCylinder*, short>(cylinder->getCCylinder(), cbs);
                        auto t2 = tuple<CCylinder*, short>(ncylinder->getCCylinder(), cnbs);
                        cmanager->appendpossibleBindings(t1,t2);
                    }
                }
            }
            if(numpairs[0] > 0)
                count++;
        }
        else if ((bManager = dynamic_cast<BranchingManager *>(manager.get()))) {
            auto numpairs = numpairs_vec[count];
            auto possibleBindings = possibleBindings_vec[count];
            for(auto i = 0; i < numpairs[0]; i++){
                int cID = possibleBindings[3 * i];
                int cIndex = possibleBindings[3 * i +1];
                short cbs = short(possibleBindings[3 * i + 2]);
                auto cylinder = Cylinder::getCylinders()[cIndex];
                Compartment* cmp = GController::getCompartment(cID);
                for(auto &cmanager : cmp->getFilamentBindingManagers()) {
                    if ((bManager = dynamic_cast<BranchingManager *>(cmanager.get()))) {
                        auto t1 = tuple<CCylinder*, short>(cylinder->getCCylinder(), cbs);
                        dynamic_cast<BranchingManager *>(cmanager.get())->appendpossibleBindings(t1);
                    }
                }
            }
//            int n = 0;
//            std::cout<<"-----serial----"<<endl;
//            for(auto C : _compartmentGrid->getCompartments()) {
//                for(auto &bbmanager : C->getFilamentBindingManagers()) {
//                    if ((bManager = dynamic_cast<BranchingManager *>(bbmanager.get()))) {
//                        bbmanager->updateAllPossibleBindings();
//                        n += dynamic_cast<BranchingManager *>(bbmanager.get())->getpossibleBindings().size();
//                    }
//                }
//            }
//            std::cout<<n<<" "<<numpairs_vec[count][0]<<endl;
            if(numpairs[0] > 0)
                count++;
            std::cout<<endl;
        }
    }
}

#endif
CompartmentGrid* SubSystem::_staticgrid;
bool SubSystem::initialize = false;
double SubSystem::SIMDtime  = 0.0;
double SubSystem::SIMDtimeV2  = 0.0;
double SubSystem::HYBDtime  = 0.0;


