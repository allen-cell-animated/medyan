
#include <hip/hip_runtime.h>

//------------------------------------------------------------------
//  **MEDYAN** - Simulation Package for the Mechanochemical
//               Dynamics of Active Networks, v3.1
//
//  Copyright (2015-2016)  Papoian Lab, University of Maryland
//
//                 ALL RIGHTS RESERVED
//
//  See the MEDYAN web page for more information:
//  http://www.medyan.org
//------------------------------------------------------------------
#ifdef HYBRID_NLSTENCILLIST
#include "HybridNeighborListImpl.h"

#include "Bead.h"
#include "Filament.h"
#include "Cylinder.h"
#include "Bubble.h"
#include "BoundaryElement.h"

#include "GController.h"
#include "MathFunctions.h"
#include "CUDAcommon.h"

using namespace mathfunc;

short HybridCylinderCylinderNL::totalhybridNL = 0;

void HybridCylinderCylinderNL::updateallcylinderstobin() {
    for(auto cyl:Cylinder::getCylinders())
        updatebin(cyl);
}

void HybridCylinderCylinderNL::assignallcylinderstobin() {
    for(auto cyl:Cylinder::getCylinders())
        assignbin(cyl);
/*    std::cout<<"H Total number of bins "<< _binGrid->getBins().size()<<endl;
    for(auto bin:_binGrid->getBins()){
        std::cout<<bin->getCylinders().size()<<" ";
    }
    std::cout<<endl;*/
}

void HybridCylinderCylinderNL::assignbin(Cylinder* cyl){
    Bin* _bin;
    try {_bin = getBin(cyl->coordinate);}
    catch (exception& e) {
        cout << e.what() << endl;
        exit(EXIT_FAILURE);
    }
    _bin->addCylinder(cyl);
    cyl->_hbinvec.push_back(_bin);
}

void HybridCylinderCylinderNL::unassignbin(Cylinder* cyl, Bin* bin){
    bin->removeCylinder(cyl);
}

void HybridCylinderCylinderNL::updatebin(Cylinder *cyl){
    Bin* _bin;
//    std::cout<<coordinate[0]<<" "<<coordinate[1]<<" "<<coordinate[2]<<endl;
    try {_bin = getBin(cyl->coordinate);}
    catch (exception& e) {
        cout << e.what();
        cyl->printSelf();
        exit(EXIT_FAILURE);
    }
    if(_bin != cyl->_hbinvec.at(_ID)) {
#ifdef CHEMISTRY
        auto oldBin = cyl->_hbinvec.at(_ID);
        auto newBin = _bin;
#endif
        //remove from old compartment, add to new
        oldBin->removeCylinder(cyl);
        cyl->_hbinvec.at(_ID) = newBin;
        _bin->addCylinder(cyl);
    }
}

void HybridCylinderCylinderNL::generateConnections() {
    for(size_t i=0U; i<_grid[0]; ++i) {

        for(size_t j=0U; j<_grid[1]; ++j) {

            for(size_t k=0U; k<_grid[2]; ++k) {
                vector<size_t> indices{i,j,k};
                Bin *target = getBin(indices);//defined in this file.

                vector<double> coordinates =
                        {indices[0] * _binSize[0] + _binSize[0] / 2,
                         indices[1] * _binSize[1] + _binSize[1] / 2,
                         indices[2] * _binSize[2] + _binSize[2] / 2};
                target->setCoordinates(coordinates);
                int stencilcount = 0;

                //Go through all neighbors to get the neighbors list
                short nneighbors = 1;
                for(int ii = -nneighbors; ii <= nneighbors; ii++){
                    for(int jj = -nneighbors; jj <= nneighbors; jj++){
                        for(int kk = -nneighbors; kk <= nneighbors; kk++){
                            //Consider the target bin itself as a neighbor.

                            stencilcount++;
                            int iprime = i+ii;
                            int jprime = j+jj;
                            int kprime = k+kk;

/*                            cout<<ii<<" "<<jj<<" "<<kk<<" "<<iprime<<" "<<jprime<<" "
                                <<kprime<<" "<<_grid[0]<<" "<<_grid[1]<<" "<<_grid[2]<<endl;*/

                            if(iprime<0 or iprime>=int(_grid[0]) or jprime<0 or
                               jprime>=int(_grid[1]) or kprime<0 or
                               kprime>=int(_grid[2]))
                                continue;
                            vector<size_t> currentIndices{size_t(iprime), size_t
                                    (jprime), size_t(kprime)};
                            Bin *neighbor = getBin(currentIndices);
                            target->addNeighbour(neighbor);
                            target->stencilID.push_back(stencilcount-1);//All 125
                            //Only 63 neighbors will be added
                            if(j>=1 || (j==0 && k>0) || (j==0 && k == 0 && i <=0)){
                                target->adduniquepermutationNeighbour(neighbor);
                            }
                        }
                    }
                }
            }
        }
    }


    /*for(size_t i=0U; i<_grid[0]; ++i) {

        for (size_t j = 0U; j < _grid[1]; ++j) {

            for (size_t k = 0U; k < _grid[2]; ++k) {
                vector<size_t> indices{i, j, k};
                Bin *target = getBin(indices);
                std::cout << "Target " << target->coordinates()[0] << " " <<
                          target->coordinates()[1] << " " <<
                          target->coordinates()[2] << " " << endl;
                std::cout<<"Bin size "<<_binSize[0]<<endl;
                for (int ii: {-1, 0, 1}) {
                    for (int jj: {-1, 0, 1}) {
                        for (int kk: {-1, 0, 1}) {
                            int iprime = i + ii;
                            int jprime = j + jj;
                            int kprime = k + kk;
                            if (iprime < 0 or iprime == int(_grid[0]) or jprime < 0 or
                                jprime == int(_grid[1]) or kprime < 0 or
                                kprime == int(_grid[2]))
                                continue;
                            vector<size_t> currentIndices{size_t(iprime), size_t
                                    (jprime), size_t(kprime)};
                            Bin *neighbor = getBin(currentIndices);
                            std::cout << "Neighbor " << neighbor->coordinates()[0]
                                      << " " <<
                                      neighbor->coordinates()[1] << " " <<
                                      neighbor->coordinates()[2] << " " << endl;
                        }
                    }
                }
            }
        }
    }*/
}

void HybridCylinderCylinderNL::initializeBinGrid() {

//    //Initial parameters of system
    auto _nDim = SysParams::Geometry().nDim;
    double searchdist = 1.125 * (sqrt(_largestrMaxsq));
    std::cout<<"H searchdist "<<searchdist<<" rMax "<<sqrt(_largestrMaxsq)<<endl;
    _binSize = {searchdist, searchdist, searchdist};
    if(_nDim >=1) {
        _size.push_back(int(SysParams::Geometry().NX * SysParams::Geometry()
                .compartmentSizeX));
        if( (_size[0]) % int(_binSize[0]) ==0)
            _grid.push_back(_size[0]/_binSize[0]);
        else
            _grid.push_back(_size[0]/_binSize[0] + 1);
        cout<<_grid[0]<<" "<<_size[0]<<" "<<_binSize[0]<<endl;
    }
    if (_nDim >= 2) {
        _size.push_back(int(SysParams::Geometry().NY * SysParams::Geometry()
                .compartmentSizeY));
        if( (_size[1]) % int(_binSize[1]) ==0)
            _grid.push_back(_size[1]/_binSize[1]);
        else
            _grid.push_back(_size[1]/_binSize[1] + 1);
        cout<<_grid[1]<<" "<<_size[1]<<" "<<_binSize[1]<<endl;
    }
    if (_nDim == 3) {
        _size.push_back(int(SysParams::Geometry().NZ * SysParams::Geometry()
                .compartmentSizeZ));
        if( (_size[2]) % int(_binSize[2]) ==0)
            _grid.push_back(_size[2]/_binSize[2]);
        else
            _grid.push_back(_size[2]/_binSize[2] + 1);
        cout<<_grid[2]<<" "<<_size[2]<<" "<<_binSize[2]<<endl;
    }

    //Check that grid and compartmentSize match nDim
    if((_nDim == 3 &&
        _grid[0] != 0 && _grid[1] != 0 && _grid[2]!=0 &&
        _binSize[0] != 0 &&
        _binSize[1] != 0 &&
        _binSize[2] != 0)){
    }
    else {
        cout << "Bin parameters for CylinderCylinderNeighborLists are invalid. Exiting." <<
             endl;
        exit(EXIT_FAILURE);
    }
    int size = 1;
    for(auto x: _grid) {
        if(x != 0) size*=x;
    }
    cout<<_grid[0]<<" "<<_grid[1]<<" "<<_grid[2]<<endl;
    cout<<size<<endl;
    //Set the instance of this grid with given parameters
    _binGrid = new BinGrid(size, _ID, _binSize);
    //Create connections based on dimensionality
    generateConnections();
}

//You need a vector of all grids so you can loop through and update respective coordinates.
Bin* HybridCylinderCylinderNL::getBin(const vector<double> &coords) {
    //Check if out of bounds
    size_t index = 0;
    size_t i = 0;
    for(auto x: coords)
    {
        //Flatten the coordinates to 1D, get integer index
        if(i == 0) {
            if(x < 0 || x >= (_binSize[0] * _grid[0])) {
                cout<<"get Bin coords x"<<endl;
                throw OutOfBoundsException();
            }

            index += int(x / _binSize[0]);
        }
        else if(i == 1) {
            if(x < 0 || x >= (_binSize[1] * _grid[1])) {
                cout<<"get Bin coords y"<<endl;
                throw OutOfBoundsException();
            }

            index += int(x / _binSize[1]) * _grid[0];
        }
        else {
            if(x < 0 || x >= (_binSize[2] * _grid[2])) {
                cout<<"get Bin coords z"<<endl;
                throw OutOfBoundsException();
            }

            index += int(x / _binSize[2]) * _grid[0] * _grid[1];
        }
        i++;
    }

    try {
        return _binGrid->getBin(index);
    }
    catch (exception& e){
        cout << "Bad bin access at..." << endl;
        cout << "Bin index = " << index << endl;
        cout << "Coords = " << coords[0] << " " << coords[1] << " " << coords[2] << endl;
        throw NaNCoordinateException();
    }
}

Bin* HybridCylinderCylinderNL::getBin(const vector<size_t> &indices) {
    size_t index = 0;
    size_t i = 0;
    for(auto x: indices)
    {
        //Flatten the indices to 1D
        if(i == 0) {
            if(x >= _grid[0]) {
                cout<<"get Bin x"<<endl;
                throw OutOfBoundsException();
            }

            index += x;
        }
        else if(i == 1) {
            if(x >= _grid[1]) {
                cout<<"get Bin y"<<endl;
                throw OutOfBoundsException();
            }

            index += x * _grid[0];
        }
        else {
            if(x >= _grid[2]) {
                cout << "get Bin z" << endl;
                throw OutOfBoundsException();
            }

            index += x * _grid[0] * _grid[1];
        }

        i++;
    }
    try {
        return _binGrid->getBin(index);
    }
    catch (exception& e){
        cout << "Bad Bin access at..." << endl;
        cout << "Bin index = " << index << endl;
        cout << "Indices = " << indices[0] << " " << indices[1] << " " << indices[2] << endl;
        throw NaNCoordinateException();
    }
}

void HybridCylinderCylinderNL::updateNeighborsbin(Cylinder* currcylinder, bool runtime){
    //clear existing neighbors of currcylinder from all neighborlists
    for(int idx = 0; idx < totaluniquefIDpairs; idx++) {
        int countbounds = _rMaxsqvec[idx].size();
        for (int idx2 = 0; idx2 < countbounds; idx2++) {
            auto HNLID = HNLIDvec[idx][idx2];
            _list4mbinvec[HNLID][currcylinder].clear();
        }
    }
    //get necessary variables
    auto binvec = currcylinder->_hbinvec;//The different hybrid bins that this cylinder
    // belongs to.
    //Check if the cylinder has been assigned a bin. If not, assign.
    if(binvec.size()<=_ID)
        assignbin(currcylinder);
    binvec = currcylinder->_hbinvec;
    //get parent bin corresponding to this hybrid neighbor list.
    auto parentbin =  binvec.at(_ID);
    //get neighboring bins
    vector<Bin*> _neighboringBins = binvec.at(_ID)//Get the bin that belongs to the
                    // current binGrid of interest for this NL.
                                                    ->getNeighbours();
    double *coord = CUDAcommon::getSERLvars().coord;
    auto cylindervec = CUDAcommon::getSERLvars().cylindervec;
    auto cylinderpointervec = CUDAcommon::getSERLvars().cylinderpointervec;
    int cindex = currcylinder->_dcIndex;
    cylinder c = cylindervec[cindex];

    //
    int ncyls2 = 0;
    int tcyl2 = 0;
    int nbincount = 0;
    auto nbinstencil = parentbin->stencilID;// A standard templated numbering of
    // neighboring bins is implemented i.e. based on position w.r.t. bin of interest,
    // neighboring bins are given a particular ID.nbinstencil stores the set of such
    // neighbors that is close to bin of interest. Bins close to the boundary will have
    // < 27 elements in the stencilID vector.
    short ftype1 = c.type; //cylinder type and filament type is one and the
    // same.
    float _largestrMax = sqrt(_largestrMaxsq);
    for (auto &bin : _neighboringBins) {
            bool isbinneeded = _binGrid->iswithincutoff(c.coord,
                                                        parentbin->coordinates(),
                                                        nbinstencil.at(nbincount),
                                                        _largestrMax);
            nbincount++;
            if (isbinneeded) {
                auto cindicesvec = bin->getcindices();
                int numneighbors = cindicesvec.size();
                for (int iter = 0; iter < numneighbors; iter++) {
                    int ncindex = cindicesvec[iter];
                    cylinder ncylinder = cylindervec[ncindex];
                    short ftype2 = ncylinder.type;
//                    //Don't add the same cylinder
//                    if (c.ID == ncylinder.ID) continue;
                    // Testing if a half neighborlist will be stable
                    if(c.ID <= ncylinder.ID) continue;
                    //Don't add if belonging to same parent
                    if (c.filamentID == ncylinder.filamentID) {
                        auto distsep = fabs(c.filamentposition - ncylinder.filamentposition);
                        //if not cross filament, check if not neighboring
                        if (distsep <= 2) continue;
                    }

                    //Loop through all the distance bounds and add to neighborlist
                    for (int idx = 0; idx < totaluniquefIDpairs; idx++) {
                        int countbounds = _rMaxsqvec[idx].size();
                        auto fpairs = _filamentIDvec[idx].data();
                        //Check for cylinder filament types
                        if (ftype1 < ftype2) {
                            if (ftype1 != fpairs[0] || ftype2 != fpairs[1])continue;
                        }
                        else if (ftype1 != fpairs[1] || ftype2 != fpairs[0]) continue;
                        double dist = twoPointDistancesquared(c.coord, ncylinder.coord);
                        if (dist < _smallestrMinsq || dist > _largestrMaxsq) continue;
                        for (int idx2 = 0; idx2 < countbounds; idx2++) {
                            //Dont add if ID is more than cylinder for half-list
                            //if (!_fullstatusvec[idx][idx2] && c.ID <= ncylinder.ID) continue;
                            //Dont add if not within range
                            if (dist > _rMaxsqvec[idx][idx2] ||
                                dist < _rMinsqvec[idx][idx2])
                                continue;
                            short HNLID = HNLIDvec[idx][idx2];
                            //If we got through all of this, add it!
                            Cylinder *Ncylinder = cylinderpointervec[ncindex];
                            _list4mbinvec[HNLID][currcylinder].push_back(Ncylinder);

                            //if runtime, add to other list as well if full
                            /* if (runtime && _fullstatusvec[idx][idx2]) {
                                _list4mbinvec[HNLID][Ncylinder].push_back(currcylinder);
                            }*/
                        }
                    }
                }
            }
    }
}

vector<Cylinder*> HybridCylinderCylinderNL::getNeighborsstencil(short HNLID, Cylinder*
                                                                cylinder) {

    return _list4mbinvec[HNLID][cylinder];
}

void HybridCylinderCylinderNL::addNeighbor(Neighbor* n) {

    //return if not a cylinder!
    Cylinder* cylinder;
    if(!(cylinder = dynamic_cast<Cylinder*>(n))) return;

    //update neighbors
    updateNeighborsbin(cylinder, true);
}

void HybridCylinderCylinderNL::removeNeighbor(Neighbor* n) {

    Cylinder* cylinder;
    if(!(cylinder = dynamic_cast<Cylinder*>(n))) return;
    for(int idx = 0; idx < totaluniquefIDpairs; idx++) {
        int countbounds = _rMaxsqvec[idx].size();
        for (int idx2 = 0; idx2 < countbounds; idx2++) {
            auto HNLID = HNLIDvec[idx][idx2];
/*            std::cout << "Removing neighbors of cylinder with cindex " <<
                          cylinder->_dcIndex<<" and ID "<<cylinder->getID() << " from NL " << HNLID << endl;*/
            //Remove from NeighborList
            _list4mbinvec[HNLID].erase(cylinder);
            //Remove from bin
            Bin *bin = cylinder->_hbinvec.at(_ID);
            unassignbin(cylinder, bin);
            //remove from other lists
//            std::cout << "Removed from cylinders ";
            for (auto it = _list4mbinvec[HNLID].begin();
                 it != _list4mbinvec[HNLID].end(); it++) {
                auto cit = find(it->second.begin(), it->second.end(), cylinder);
                {
                    if (cit != it->second.end()) {
                        it->second.erase(cit);
//                        std::cout << it->first->getID() << " ";
                    }
                }
            }
//            std::cout<<endl;
        }
    }

}

void HybridCylinderCylinderNL::reset() {

    //loop through all neighbor keys
    for(int idx = 0; idx < totalhybridNL; idx++) {
        _list4mbinvec[idx].clear();
//        std::cout<<"Hybrid rmin rmax "<<_rMinsqvec[idx]<<" "<<_rMaxsqvec[idx]<<endl;
    }

    /*chrono::high_resolution_clock::time_point mins, mine;
    mins = chrono::high_resolution_clock::now();*/
    //check and reassign cylinders to different bins if needed.
    updateallcylinderstobin();
    _binGrid->updatecindices();
    for(auto cylinder: Cylinder::getCylinders()) {
        updateNeighborsbin(cylinder);
//        for (int idx = 0; idx < totalhybridNL; idx++) {
//            tot[idx] += _list4mbinvec[idx][cylinder].size();
//        }
    }
//    std::cout<<endl;
//    for(int idx = 0; idx < totalhybridNL; idx++)
//        std::cout<<"reset HybridNLSTENCILLIST size "<<" "<<tot[idx]<<endl;
/*    mine= chrono::high_resolution_clock::now();
    chrono::duration<double> elapsed_sten(mine - mins);
    std::cout<<"Hybrid NLSTEN reset time "<<elapsed_sten.count()<<endl;*/

    //Check if HNLID = 1 is symmetric
/*    short HNLID = 1;
    int idx = 0; int idx2 = _rMaxsqvec[idx].size()-1;
    auto _list4mbin = _list4mbinvec[HNLID];
    std::cout<<"map size = " << _list4mbin.size()<<endl;
    std::cout << "max_size = " << _list4mbin.max_size() <<endl;
    for(auto cylinder: Cylinder::getCylinders()) {
        auto neighbors = _list4mbin[cylinder];
        auto cylinderbin = cylinder->_hbinvec[0];
        auto cylbincoord = cylinderbin->coordinates();
        for(auto ncylinder:neighbors){
            auto ncylinderbin = ncylinder->_hbinvec[0];
            auto ncylbincoord = ncylinderbin->coordinates();
            //look for cylinder in the neighbor list of ncylinder
            auto ncylinderneighbors = _list4mbin[ncylinder];
            std::cout<<"neighborvec size "<<ncylinderneighbors.size()<<" capacity "
                     <<ncylinderneighbors.capacity()<<" max_size "<<ncylinderneighbors
                    .max_size()<<endl;
            if(find(ncylinderneighbors.begin(),ncylinderneighbors.end(),cylinder) ==
                    ncylinderneighbors.end()){
                std::cout<<" cylinder "<<cylinder->getID()<<" from bin "
                        ""<<cylinderbin<<" "
                        "coordinates "<<cylbincoord[0]<<" "<<cylbincoord[1]<<" "
                        ""<<cylbincoord[2]<<" has neighbor cylinder "<<ncylinder->getID()
                         <<" from bin "<<ncylinderbin<<" coordinates "
                        ""<<ncylbincoord[0]<<" "<<ncylbincoord[1]<<" "
                        ""<<ncylbincoord[2]<<endl;
                std::cout<<"But neighbor cylinder does not have cylinder in it's "
                        "neighbors list. Check algorithm."<<endl;
            }
        }
    }*/
}

void HybridCylinderCylinderNL::updateSIMDbindingsites(){

    //check and reassign cylinders to different bins if needed.
    chrono::high_resolution_clock::time_point minscreate, minecreate;
    minscreate = chrono::high_resolution_clock::now();
    updateallcylinderstobin();
    _binGrid->updatecindices();
    _binGrid->createSIMDcoordinates();
    minecreate = chrono::high_resolution_clock::now();
    chrono::duration<double> elapsed_create(minecreate - minscreate);
    cout<<"SIMD NL create time "<<elapsed_create.count()<<endl;
    short idvecL[2] = {0,0};
    short idvecM[2] = {0,1};
    minscreate = chrono::high_resolution_clock::now();
    for(auto bin:_binGrid->getBins()){
    calculatebspairsLMself<1,true, true>(bin, bspairslinkerself, idvecL);
    calculatebspairsLMenclosed<1,false, true>(bin, bspairslinker,idvecL);
    }
    minecreate = chrono::high_resolution_clock::now();
    chrono::duration<double> elapsed_calculate(minecreate - minscreate);
    cout<<"SIMD NL calculate time L "<<elapsed_calculate.count()<<endl;
    minscreate = chrono::high_resolution_clock::now();
    for(auto bin:_binGrid->getBins()){
            calculatebspairsLMself<1,true, false>(bin, bspairsmotorself, idvecM);
            calculatebspairsLMenclosed<1,false, false>(bin, bspairsmotor,idvecM);
    }
    minecreate = chrono::high_resolution_clock::now();
    chrono::duration<double> elapsed_calculateM(minecreate - minscreate);
    cout<<"SIMD NL calculate time M "<<elapsed_calculateM.count()<<endl;
}

template <uint D, bool SELF, bool LinkerorMotor>
void HybridCylinderCylinderNL::calculatebspairsLMself(Bin* bin, dist::dOut<D, SELF>&
bspairsoutSself, short idvec[2]){

    auto boundstate = SysParams::Mechanics().speciesboundvec;
    CCylinder **ccylvec = CUDAcommon::getSERLvars().ccylindervec;
    auto cylcmp1 = bin->Cyldcindexvec;

    minsfind = chrono::high_resolution_clock::now();
    if(bin->getSIMDcoords<LinkerorMotor>().size()) {
        bspairsoutSself.reset_counters();
        dist::find_distances(bspairsoutSself, bin->getSIMDcoords<LinkerorMotor>(),
                             t_avx_par);
    }
    minefind = chrono::high_resolution_clock::now();
    chrono::duration<double> elapsed_runfind(minefind - minsfind);
    findtimeV2 += elapsed_runfind.count();
/*    cout<<bin->getSIMDcoords<LinkerorMotor>().size()<<" "<<bspairsoutSself
    .counter[D-1]<<endl;*/
    //@{
    /*if(false) {

        uint N = bspairsoutSself.counter[D-1];
        uint prev_size = getfilID_fpospairs<LinkerorMotor>().size();
        getpairsLinkerorMotor<LinkerorMotor>().resize
                (getpairsLinkerorMotor<LinkerorMotor>().size() + 2*N);
        getfilID_fpospairs<LinkerorMotor>().resize(getfilID_fpospairs<LinkerorMotor>()
                                                           .size() + N);

        minsfind = chrono::high_resolution_clock::now();
        std::vector<std::thread> threads_avx;
        uint nt = nthreads;
        threads_avx.reserve(nt);
        uint prev = 0;
        uint frac = N / nt;
        uint next = frac + N % nt;
        for (uint i = 0; i < nt; ++i) {
            threads_avx.push_back(std::thread(
                    &HybridBindingSearchManager::gatherCylinderIDfromcIndex<D, SELF,
                            LinkerorMotor>, this, std::ref(bspairsoutSself), prev,
                    next, prev_size, _compartment));
            prev = next;
            next = min(N, prev + frac);
        }

        //Join
        for (auto &t : threads_avx)
            t.join();
        threads_avx.clear();

        minefind = chrono::high_resolution_clock::now();
        chrono::duration<double> elapsed_append(minefind - minsfind);
        appendtime += elapsed_append.count();
    }*/
    //@}
}

template<uint D, bool SELF, bool LinkerorMotor>
void HybridCylinderCylinderNL::calculatebspairsLMenclosed (Bin* bin, dist::dOut<D,SELF>&
bspairsoutS, short idvec[2]){
    auto boundstate = SysParams::Mechanics().speciesboundvec;
    CCylinder **ccylvec = CUDAcommon::getSERLvars().ccylindervec;
    int maxnbs = SysParams::Chemistry().maxbindingsitespercylinder;
    auto cylcmp1 = bin->Cyldcindexvec;

    for(auto nbin: bin->getuniquepermutationNeighbours()){

        minsfind = chrono::high_resolution_clock::now();

        if(bin->getSIMDcoords<LinkerorMotor>().size() > 0 &&
        nbin->getSIMDcoords<LinkerorMotor>().size() > 0) {
            bspairsoutS.reset_counters();
            dist::find_distances(bspairsoutS, bin->getSIMDcoords<LinkerorMotor>(),
                                 nbin->getSIMDcoords<LinkerorMotor>(), t_avx_par);
        }
        minefind = chrono::high_resolution_clock::now();
        chrono::duration<double> elapsed_runfind(minefind - minsfind);
        findtimeV2 += elapsed_runfind.count();
        /*cout<<bin->getSIMDcoords<LinkerorMotor>().size()<<" "
            <<nbin->getSIMDcoords<LinkerorMotor>().size()<<" "
            <<bspairsoutS.counter[D-1]<<endl;*/
        //MERGE INTO single vector
        //@{
        /*if(false) {
            minsfind = chrono::high_resolution_clock::now();
            short dim = 0;
            uint N = bspairsoutS.counter[dim];
            uint prev_size = getfilID_fpospairs<LinkerorMotor>().size();
            getpairsLinkerorMotor<LinkerorMotor>().resize
                    (getpairsLinkerorMotor<LinkerorMotor>().size() + 2*N);
            getfilID_fpospairs<LinkerorMotor>().resize(getfilID_fpospairs<LinkerorMotor>()
                                                               .size() + N);

            std::vector<std::thread> threads_avx;
            uint nt = nthreads;
            threads_avx.reserve(nt);
            uint prev = 0;
            uint frac = N / nt;
            uint next = frac + N % nt;
            for (uint i = 0; i < nt; ++i) {
                threads_avx.push_back(std::thread
                                              (&HybridBindingSearchManager::gatherCylinderIDfromcIndex<D, SELF,
                                                       LinkerorMotor>, this,
                                               std::ref(bspairsoutS), prev,
                                               next, prev_size, ncmp));
                prev = next;
                next = min(N, prev + frac);
            }

            //Join
            for (auto &t : threads_avx)
                t.join();
            threads_avx.clear();

            minefind = chrono::high_resolution_clock::now();
            chrono::duration<double> elapsed_append(minefind - minsfind);
            appendtime += elapsed_append.count();
        }*/
        //@}
    }
}
double HybridCylinderCylinderNL::SIMDtime = 0.0;
double HybridCylinderCylinderNL::HYBDtime = 0.0;
double HybridCylinderCylinderNL::findtime = 0.0;
double HybridCylinderCylinderNL::appendtime = 0.0;
double HybridCylinderCylinderNL::findtimeV2 = 0.0;
double HybridCylinderCylinderNL::SIMDparse1 = 0.0;
double HybridCylinderCylinderNL::SIMDparse2 = 0.0;
double HybridCylinderCylinderNL::SIMDparse3 = 0.0;
double HybridCylinderCylinderNL::SIMDcountbs = 0.0;

dist::dOut<1U,false> HybridCylinderCylinderNL::bspairslinker;
dist::dOut<1U,true> HybridCylinderCylinderNL::bspairslinkerself;
dist::dOut<1U,false> HybridCylinderCylinderNL::bspairsmotor;
dist::dOut<1U,true> HybridCylinderCylinderNL::bspairsmotorself;

#endif